#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/particle/flip_common.cuh"
#include "flip.h"

texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;

#include "particle_advection.cuh"

// =============================================================================

// Active particles should be consecutive, but could be freed during the
// routine.
template <typename AdvectionImpl>
__global__ void AdvectParticlesKernel(uint16_t* pos_x, uint16_t* pos_y,
                                      uint16_t* pos_z, uint16_t* density,
                                      uint16_t* life, int num_of_particles,
                                      float3 bounds,
                                      float time_step_over_cell_size,
                                      bool outflow, AdvectionImpl advect)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_of_particles)
        return;

    float x = __half2float(pos_x[i]);
    float y = __half2float(pos_y[i]);
    float z = __half2float(pos_z[i]);

    // The fluid looks less bumpy with the re-sampled velocity. Don't know
    // the exact reason yet.
    float v_x = tex3D(tex_x, x + 0.5f, y,        z);
    float v_y = tex3D(tex_y, x,        y + 0.5f, z);
    float v_z = tex3D(tex_z, x,        y,        z + 0.5f);

    if (IsStopped(v_x, v_y, v_z)) {
        // Don't eliminate the particle. It may contains density/temperature
        // information.
        //
        // We don't need the number of active particles until the sorting is
        // done.
        return;
    }

    float3 result = advect.Advect(make_float3(x, y, z),
                                  make_float3(v_x, v_y, v_z),
                                  time_step_over_cell_size);

    if (result.x < 0.0f || result.x > bounds.x)
        life[i] = 0.0f;

    if (result.y < 0.0f || result.y > bounds.y)
        life[i] = 0.0f;

    if (result.z < 0.0f || result.z > bounds.z)
        life[i] = 0.0f;

    float3 pos = clamp(result, make_float3(0.0f), bounds);

    pos_x[i] = __float2half_rn(pos.x);
    pos_y[i] = __float2half_rn(pos.y);
    pos_z[i] = __float2half_rn(pos.z);
}

// =============================================================================

namespace kern_launcher
{
void AdvectParticles(uint16_t* pos_x, uint16_t* pos_y, uint16_t* pos_z,
                     uint16_t* density, uint16_t* life, int num_of_particles,
                     hipArray* vel_x, hipArray* vel_y, hipArray* vel_z,
                     float time_step, float cell_size, bool outflow,
                     uint3 volume_size, BlockArrangement* ba)
{
    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, num_of_particles);

    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    AdvectionEuler adv_fe;
    AdvectionMidPoint adv_mp;
    AdvectionBogackiShampine adv_bs;
    AdvectionRK4 adv_rk4;

    float3 bounds = make_float3(volume_size) - 1.0f;
    int order = 3;
    switch (order) {
        case 1:
            AdvectParticlesKernel<<<grid, block>>>(pos_x, pos_y, pos_z, density,
                                                   life, num_of_particles,
                                                   bounds,
                                                   time_step / cell_size,
                                                   outflow, adv_fe);
            break;
        case 2:
            AdvectParticlesKernel<<<grid, block>>>(pos_x, pos_y, pos_z, density,
                                                   life, num_of_particles,
                                                   bounds,
                                                   time_step / cell_size,
                                                   outflow, adv_mp);
            break;
        case 3:
            AdvectParticlesKernel<<<grid, block>>>(pos_x, pos_y, pos_z, density,
                                                   life, num_of_particles,
                                                   bounds,
                                                   time_step / cell_size,
                                                   outflow, adv_bs);
            break;
        case 4:
            AdvectParticlesKernel<<<grid, block>>>(pos_x, pos_y, pos_z, density,
                                                   life, num_of_particles,
                                                   bounds,
                                                   time_step / cell_size,
                                                   outflow, adv_rk4);
            break;
    }

    DCHECK_KERNEL();
}
}
