#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "cuda/aux_buffer_manager.h"
#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/fluid_impulse.h"
#include "cuda/particle/flip_common.cuh"
#include "flip.h"
#include "random.cuh"

texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_d;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_t;

namespace
{
__device__ bool IsCellActive(float v_x, float v_y, float v_z, float density,
                             float temperature)
{
    const float kEpsilon = 0.0001f;
    return !IsStopped(v_x, v_y, v_z) || density > kEpsilon ||
            temperature > kEpsilon;
}

__device__ bool IsThereEnoughFreeParticles(FlipParticles& particles, int needed)
{
    // As we don't need to compact the particles, the number of active particles
    // becomes less concerned by us.
    bool limited_particles = false;
    if (!limited_particles)
        return true;

    int total_count = atomicAdd(particles.num_of_actives_, needed);
    if (total_count + needed > particles.num_of_particles_) {
        atomicAdd(particles.num_of_actives_, -needed);
        return false;
    }

    return true;
}

struct HorizontalEmission
{
    __device__ static bool OutsideVolume(uint x, uint y, uint z,
                                         const uint3& volume_size)
    {
        return y >= volume_size.y || z >= volume_size.z;
    }
    __device__ static float CalculateRadius(const float3& coord,
                                            const float3& center,
                                            const float3& hotspot)
    {
        float2 diff =
            make_float2(coord.y, coord.z) - make_float2(center.y, center.z);
        return hypotf(diff.x, diff.y);
    }
    __device__ static void SetVelX(uint16_t* vel_x, const float3 velocity)
    {
        *vel_x = __float2half_rn(velocity.x);
    }
};

struct VerticalEmission
{
    __device__ static bool OutsideVolume(uint x, uint y, uint z,
                                         const uint3& volume_size)
    {
        return x >= volume_size.x || z >= volume_size.z;
    }
    __device__ static float CalculateRadius(const float3& coord,
                                            const float3& center,
                                            const float3& hotspot)
    {
        float2 diff =
            make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
        return hypotf(diff.x, diff.y);
    }
    __device__ static void SetVelX(uint16_t* vel_x, const float3 velocity)
    {
    }
};

// NOTE: Assuming never overflows/underflows.
template <int Increment>
__device__ uint8_t AtomicIncrementUint8(uint8_t* addr)
{
    uint r = 0;
    uint* base_addr =
        reinterpret_cast<uint*>(reinterpret_cast<size_t>(addr) & ~3);
    switch (reinterpret_cast<size_t>(addr) & 3) {
        case 0:
            r = atomicAdd(base_addr, static_cast<uint>(Increment));
            return static_cast<uint8_t>(r & 0xFF);
        case 1:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 8);
            return static_cast<uint8_t>((r >> 8) & 0xFF);
        case 2:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 16);
            return static_cast<uint8_t>((r >> 16) & 0xFF);
        case 3:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 24);
            return static_cast<uint8_t>((r >> 24) & 0xFF);
    }

    return 0;
}

// =============================================================================

// Fields should be available: particle_count.
template <typename Emission>
__global__ void EmitFlipParticlesKernel(FlipParticles particles, float3 center,
                                        float3 hotspot, float radius,
                                        float density, float temperature,
                                        float3 velocity, uint random_seed,
                                        uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (Emission::OutsideVolume(x, y, z, volume_size))
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float d = Emission::CalculateRadius(coord, center, hotspot);
    if (d >= radius)
        return;

    uint cell_index = LinearIndexVolume(x, y, z, volume_size);
    int count = particles.particle_count_[cell_index];
    if (!count) {
        int new_particles = kMaxNumSamplesForOneTime;
        if (!IsThereEnoughFreeParticles(particles, new_particles))
            return;

        particles.particle_count_[cell_index] = new_particles;
        uint seed = random_seed + cell_index;
        for (int i = 0; i < new_particles; i++) {
            float3 pos = coord + RandomCoordCube(&seed);

            int index = cell_index * kMaxNumParticlesPerCell + i;

            particles.position_x_ [index] = __float2half_rn(pos.x);
            particles.position_y_ [index] = __float2half_rn(pos.y);
            particles.position_z_ [index] = __float2half_rn(pos.z);
            particles.velocity_x_ [index] = 0;
            particles.velocity_y_ [index] = 0;
            particles.velocity_z_ [index] = 0;
            particles.density_    [index] = __float2half_rn(density);
            particles.temperature_[index] = __float2half_rn(temperature);

            Emission::SetVelX(&particles.velocity_x_[index], velocity);
        }
    } else {
        uint p_index = cell_index * kMaxNumParticlesPerCell;
        for (int i = 0; i < count; i++) {
            particles.density_    [p_index + i] = __float2half_rn(density);
            particles.temperature_[p_index + i] = __float2half_rn(temperature);

            Emission::SetVelX(&particles.velocity_x_[p_index + i], velocity);
        }
    }
}

// Fields should be available: particle_count.
__global__ void EmitFlipParticlesFromSphereKernel(FlipParticles particles,
                                                  float3 center, float radius,
                                                  float density,
                                                  float temperature,
                                                  float velocity,
                                                  uint random_seed,
                                                  uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = coord - center;
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d >= radius)
        return;

    uint cell_index = LinearIndexVolume(x, y, z, volume_size);
    int count = particles.particle_count_[cell_index];
    if (!count) {
        int new_particles = kMaxNumSamplesForOneTime;
        if (!IsThereEnoughFreeParticles(particles, new_particles))
            return;

        particles.particle_count_[cell_index] = new_particles;
        uint seed = random_seed + cell_index;
        for (int i = 0; i < new_particles; i++) {
            float3 pos = coord + RandomCoordCube(&seed);

            int index = cell_index * kMaxNumParticlesPerCell + i;

            float3 dir = pos - center;
            float3 vel = normalize(dir) * velocity;

            // Not necessary to initialize the in_cell_index field.
            // Particle-cell mapping will be done in the binding kernel.

            particles.position_x_ [index] = __float2half_rn(pos.x);
            particles.position_y_ [index] = __float2half_rn(pos.y);
            particles.position_z_ [index] = __float2half_rn(pos.z);
            particles.velocity_x_ [index] = __float2half_rn(vel.x);
            particles.velocity_y_ [index] = __float2half_rn(vel.y);
            particles.velocity_z_ [index] = __float2half_rn(vel.z);
            particles.density_    [index] = __float2half_rn(density);
            particles.temperature_[index] = __float2half_rn(temperature);
        }
    } else {
        uint p_index = cell_index * kMaxNumParticlesPerCell;
        for (int i = 0; i < count; i++) {
            float pos_x = __half2float(particles.position_x_[p_index + i]);
            float pos_y = __half2float(particles.position_y_[p_index + i]);
            float pos_z = __half2float(particles.position_z_[p_index + i]);
            float3 pos = make_float3(pos_x, pos_y, pos_z);

            float3 dir = pos - center;
            float3 vel = normalize(dir) * velocity;

            particles.velocity_x_ [p_index + i] = __float2half_rn(vel.x);
            particles.velocity_y_ [p_index + i] = __float2half_rn(vel.y);
            particles.velocity_z_ [p_index + i] = __float2half_rn(vel.z);
            particles.density_    [p_index + i] = __float2half_rn(density);
            particles.temperature_[p_index + i] = __float2half_rn(temperature);
        }
    }
}

// The new particles sample the velocity of the last step. Please see the
// comments of FLIP particle advection kernel.
//
// One should be very careful designing the mechanism of re-sampling: an
// important difference between particles and grid is that once a particle is
// created, its density and temperature are not gonna change during its life
// time(except decaying).
//
// Fields should be available: particle_count.
// Active particles are always *NOT* consecutive.
__global__ void ResampleKernel(FlipParticles particles, uint random_seed,
                               uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    uint cell_index = LinearIndexVolume(x, y, z, volume_size);
    int count = particles.particle_count_[cell_index];

    // Scan for all undersampled cells, and try to insert new particles.
    if (count > kMinNumParticlesPerCell)
        return;

    // CAUTION: All the physics variables, except velocity, should always be
    //          updated directly to the particles, or these changes might never
    //          get a chance to be applied to the particles, since the re-sample
    //          kernel only concerns about the cells that not having sufficient
    //          particles.
    int needed = min(kMaxNumParticlesPerCell - count, kMaxNumSamplesForOneTime);
    if (needed <= 0)
        return;

    // Used to be: IsCellActive(center_vel, center_density, center_temperature).
    // But that is not correct. We should not judge all the particles only by
    // the state right at the center of the cell.

    if (!IsThereEnoughFreeParticles(particles, needed))
        return;

    // FIXME: Rectify particle count.

    float3 coord = make_float3(x, y, z) + 0.5f;

    // Reseed particles.
    uint seed = random_seed + cell_index;
    for (int i = 0; i < needed; i++) {
        float3 pos = coord + RandomCoordCube(&seed);

        // TODO: Accelerate with shared memory.
        float3 v           = LoadVel(tex_x, tex_y, tex_z, pos);
        float  density     = tex3D  (tex_d, pos.x, pos.y, pos.z);
        float  temperature = tex3D  (tex_t, pos.x, pos.y, pos.z);

        int index = cell_index * kMaxNumParticlesPerCell + count + i;

        particles.position_x_ [index] = __float2half_rn(pos.x);
        particles.position_y_ [index] = __float2half_rn(pos.y);
        particles.position_z_ [index] = __float2half_rn(pos.z);
        particles.velocity_x_ [index] = __float2half_rn(v.x);
        particles.velocity_y_ [index] = __float2half_rn(v.y);
        particles.velocity_z_ [index] = __float2half_rn(v.z);
        particles.density_    [index] = __float2half_rn(density);
        particles.temperature_[index] = __float2half_rn(temperature);
    }
}

__global__ void ResetParticlesKernel(FlipParticles particles)
{
    uint i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= particles.num_of_particles_)
        return;

    particles.velocity_x_ [i] = 0;
    particles.velocity_y_ [i] = 0;
    particles.velocity_z_ [i] = 0;
    particles.position_x_ [i] = 0;
    particles.position_y_ [i] = 0;
    particles.position_z_ [i] = 0;
    particles.density_    [i] = 0;
    particles.temperature_[i] = 0;
    FreeParticle(particles, i);

    if (i == 0)
        *particles.num_of_actives_ = 0;
}

// Fields should be reset: particle_count
// Active particles may *NOT* be consecutive.
__global__ void SortParticlesKernel(FlipParticles p_aux, FlipParticles p_src,
                                    float time_step, float velocity_dissipation,
                                    float density_dissipation,
                                    float temperature_dissipation,
                                    uint3 volume_size)
{
    uint i = LinearIndex();
    if (i >= p_src.num_of_particles_)
        return;

    uint16_t xh = p_src.position_x_[i];
    uint16_t yh = p_src.position_y_[i];
    uint16_t zh = p_src.position_z_[i];

    if (IsCellUndefined(xh))
        return;

    int cell_index = CellIndex(xh, yh, zh, volume_size);
    uint* p_count = p_src.particle_count_;
    if (p_count[cell_index] < kMaxNumParticlesPerCell) {
        uint old_count = atomicAdd(p_count + cell_index, 1);
        if (old_count >= kMaxNumParticlesPerCell) {
            // TODO: Could leave the count to be larger than
            //       kMaxNumParticlesPerCell, and clamp it in the mapping
            //       kernel. But the active number calculation is a problem.
            atomicAdd(p_count + cell_index, static_cast<uint>(-1));
        } else {
            uint sort_index = cell_index * kMaxNumParticlesPerCell + old_count;

            p_aux.position_x_ [sort_index] = xh;
            p_aux.position_y_ [sort_index] = yh;
            p_aux.position_z_ [sort_index] = zh;
            p_aux.velocity_x_ [sort_index] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(p_src.velocity_x_[i]));
            p_aux.velocity_y_ [sort_index] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(p_src.velocity_y_[i]));
            p_aux.velocity_z_ [sort_index] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(p_src.velocity_z_[i]));
            p_aux.density_    [sort_index] = __float2half_rn((1.0f - density_dissipation     * time_step) * __half2float(p_src.density_[i]));
            p_aux.temperature_[sort_index] = __float2half_rn((1.0f - temperature_dissipation * time_step) * __half2float(p_src.temperature_[i]));
        }
    }
}
} // Anonymous namespace.

struct SchemeDefault
{
    __host__ SchemeDefault(uint32_t* p_count)
        : p_count_(p_count)
    {
    }
    __device__ int Load(uint i, uint row_stride, uint slice_stride)
    {
        return p_count_[i];
    }
    __device__ void Save(int* dest, int result)
    {
        *dest = result;
    }

    uint32_t* p_count_;
};

#include "../volume_reduction.cuh"

// =============================================================================

namespace kern_launcher
{
void EmitFlipParticles(const FlipParticles& particles, float3 center,
                       float3 hotspot, float radius, float density,
                       float temperature, float3 velocity, FluidImpulse impulse,
                       uint random_seed, uint3 volume_size,
                       BlockArrangement* ba)
{

    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            const float kHeatLayerThickness = 0.025f * volume_size.y;
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(std::ceil(kHeatLayerThickness));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesKernel<VerticalEmission><<<grid, block>>>(
                particles, center, hotspot, radius, density, temperature,
                velocity, random_seed, volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(std::ceil(radius + center.y));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesFromSphereKernel<<<grid, block>>>(
                particles, center, radius, density, temperature, velocity.x,
                random_seed, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            const float kHeatLayerThickness = 0.02f * volume_size.x;
            uint3 actual_size = volume_size;
            actual_size.x = static_cast<uint>(std::ceil(kHeatLayerThickness));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesKernel<HorizontalEmission><<<grid, block>>>(
                particles, center, hotspot, radius, density, temperature,
                velocity, random_seed, volume_size);
            break;
        }
    }
    
    DCHECK_KERNEL();
}

void Resample(const FlipParticles& particles, hipArray* vel_x,
              hipArray* vel_y, hipArray* vel_z, hipArray* density,
              hipArray* temperature, uint random_seed, uint3 volume_size,
              BlockArrangement* ba)
{
    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    auto bound_d = BindHelper::Bind(&tex_d, density, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_d.error() != hipSuccess)
        return;

    auto bound_t = BindHelper::Bind(&tex_t, temperature, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_t.error() != hipSuccess)
        return;

    dim3 grid;
    dim3 block;
    ba->ArrangePrefer3dLocality(&grid, &block, volume_size);
    ResampleKernel<<<grid, block>>>(particles, random_seed, volume_size);
    DCHECK_KERNEL();
}

void ResetParticles(const FlipParticles& particles, uint3 volume_size,
                    BlockArrangement* ba)
{
    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    ResetParticlesKernel<<<grid, block>>>(particles);

    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipMemsetAsync(particles.particle_count_, 0,
                    num_of_cells * sizeof(*particles.particle_count_));
    DCHECK_KERNEL();
}

void SortParticles(FlipParticles particles, int* num_active_particles,
                   FlipParticles aux, float time_step,
                   float velocity_dissipation, float density_dissipation,
                   float temperature_dissipation, uint3 volume_size,
                   BlockArrangement* ba, AuxBufferManager* bm)
{
    // Reset all particles in |aux| to undefined.
    thrust::device_ptr<uint16_t> v(aux.position_x_);
    thrust::fill(v, v + aux.num_of_particles_, kInvalidPos);

    FlipParticles& p_src = particles;
    FlipParticles& p_aux = aux;

    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipError_t e = hipMemsetAsync(
        p_src.particle_count_, 0,
        num_of_cells * sizeof(*p_src.particle_count_));
    assert(e == hipSuccess);
    if (e != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, p_src.num_of_particles_);
    SortParticlesKernel<<<grid, block>>>(p_aux, p_src, time_step,
                                         velocity_dissipation,
                                         density_dissipation,
                                         temperature_dissipation, volume_size);
    DCHECK_KERNEL();

    SchemeDefault scheme(p_src.particle_count_);
    ReduceVolume<int>(p_src.num_of_actives_, scheme, volume_size, ba, bm);
    DCHECK_KERNEL();

    hipMemcpyAsync(num_active_particles, particles.num_of_actives_,
                    sizeof(*num_active_particles), hipMemcpyDeviceToHost);
}
}
