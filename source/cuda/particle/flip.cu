#include "hip/hip_runtime.h"
//
// Fluid3d - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Fluid3d license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/aux_buffer_manager.h"
#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/particle/flip_common.cuh"
#include "flip.h"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
surface<void, cudaSurfaceType3D> surf_d;
surface<void, cudaSurfaceType3D> surf_t;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_xp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_yp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_zp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_d;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_t;

const uint32_t kMaxNumParticlesPerCell = 4;
const uint32_t kMinNumParticlesPerCell = 2;
const uint32_t kMaxNumSamplesForOneTime = 3;

__device__ bool IsCellActive(float v_x, float v_y, float v_z, float density,
                             float temperature)
{
    const float kEpsilon = 0.0001f;
    return !IsStopped(v_x, v_y, v_z) || density > kEpsilon ||
            temperature > kEpsilon;
}

// NOTE: Assuming never overflows/underflows.
template <int Increment>
__device__ uint8_t AtomicIncrementUint8(uint8_t* addr)
{
    uint r = 0;
    uint* base_addr =
        reinterpret_cast<uint*>(reinterpret_cast<size_t>(addr) & ~3);
    switch (reinterpret_cast<size_t>(addr) & 3) {
        case 0:
            r = atomicAdd(base_addr, static_cast<uint>(Increment));
            return static_cast<uint8_t>(r & 0xFF);
        case 1:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 8);
            return static_cast<uint8_t>((r >> 8) & 0xFF);
        case 2:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 16);
            return static_cast<uint8_t>((r >> 16) & 0xFF);
        case 3:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 24);
            return static_cast<uint8_t>((r >> 24) & 0xFF);
    }

    return 0;
}

__device__ uint Tausworthe(uint z, int s1, int s2, int s3, uint M)
{
    uint b = (((z << s1) ^ z) >> s2);
    return (((z & M) << s3) ^ b);
}

__device__ float3 RandomCoord(uint* random_seed)
{
    uint seed = *random_seed;
    uint seed0 = Tausworthe(seed,  (blockIdx.x  + 1) & 0xF, (blockIdx.y  + 2) & 0xF, (blockIdx.z  + 3) & 0xF, 0xFFFFFFFE);
    uint seed1 = Tausworthe(seed0, (threadIdx.x + 1) & 0xF, (threadIdx.y + 2) & 0xF, (threadIdx.z + 3) & 0xF, 0xFFFFFFF8);
    uint seed2 = Tausworthe(seed1, (threadIdx.y + 1) & 0xF, (threadIdx.z + 2) & 0xF, (threadIdx.x + 3) & 0xF, 0xFFFFFFF0);
    uint seed3 = Tausworthe(seed2, (threadIdx.z + 1) & 0xF, (threadIdx.x + 2) & 0xF, (threadIdx.y + 3) & 0xF, 0xFFFFFFE0);

    float rand_x = (seed1 & 127) / 129.5918f - 0.49f;
    float rand_y = (seed2 & 127) / 129.5918f - 0.49f;
    float rand_z = (seed3 & 127) / 129.5918f - 0.49f;

    *random_seed = seed3;
    return make_float3(rand_x, rand_y, rand_z);
}

// =============================================================================

// Fields should be reset: particle_count, in_cell_index
// Fields should be available: cell_index.
// Active particles may *NOT* be consecutive.
__global__ void BindParticlesToCellsKernel(FlipParticles particles,
                                           uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= particles.num_of_particles_)
        return;

    uint cell_index = particles.cell_index_[i];
    if (IsCellUndefined(cell_index))
        return;

    // TODO: Free particles in resample kernel?
    uint* p_count = particles.particle_count_;
    if (p_count[cell_index] >= kMaxNumParticlesPerCell) {
        FreeParticle(particles, i);
    } else {
        uint old_count = atomicAdd(p_count + cell_index, 1);
        if (old_count >= kMaxNumParticlesPerCell) {
            atomicAdd(p_count + cell_index, static_cast<uint>(-1));
            FreeParticle(particles, i);
        } else {
            particles.in_cell_index_[i] = old_count;
        }
    }
}

__global__ void CalculateNumberOfActiveParticles(FlipParticles particles,
                                                 int last_cell_index)
{
    *particles.num_of_actives_ =
        particles.particle_index_[last_cell_index] +
        particles.particle_count_[last_cell_index];
}

// Fields should be available: cell_index, particle_count, particle_index.
__global__ void EmitParticlesKernel(FlipParticles particles,
                                    float3 center_point, float3 hotspot,
                                    float radius, float density,
                                    float temperature, uint random_seed,
                                    uint3 volume_size)
{
    uint x = VolumeX();
    uint y = 1 + threadIdx.y;
    uint z = VolumeZ();

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d >= radius)
        return;

    uint cell_index = (z * volume_size.y + y) * volume_size.x + x;
    int count = particles.particle_count_[cell_index];
    if (!count) {
        int new_particles = kMaxNumSamplesForOneTime;
        int base_index = atomicAdd(particles.num_of_actives_, new_particles);
        if (base_index + new_particles > particles.num_of_particles_) {
            atomicAdd(particles.num_of_actives_, -new_particles);
            return; // Not enough free particles.
        }

        particles.particle_count_[cell_index] += new_particles;
        uint seed = random_seed;
        for (int i = 0; i < new_particles; i++) {
            float3 pos = coord + RandomCoord(&seed);

            int index = base_index + i;

            // Not necessary to initialize the in_cell_index field.
            // Particle-cell mapping will be done in the binding kernel.

            // Assign a valid value to |cell_index_| to activate this particle.
            particles.cell_index_ [index] = cell_index;
            particles.position_x_ [index] = __float2half_rn(pos.x);
            particles.position_y_ [index] = __float2half_rn(pos.y);
            particles.position_z_ [index] = __float2half_rn(pos.z);
            particles.velocity_x_ [index] = 0;
            particles.velocity_y_ [index] = 0;
            particles.velocity_z_ [index] = 0;
            particles.density_    [index] = __float2half_rn(density);
            particles.temperature_[index] = __float2half_rn(temperature);
        }
    } else {
        uint p_index = particles.particle_index_[cell_index];
        for (int i = 0; i < count; i++) {
            // TOOD: Reset velocity to 0?
            particles.density_    [p_index + i] = __float2half_rn(density);
            particles.temperature_[p_index + i] = __float2half_rn(temperature);
        }
    }
}

// Should be invoked *BEFORE* resample kernel. Please read the comments of
// ResampleKernel().
// Active particles should be consecutive.
__global__ void InterpolateDeltaVelocityKernel(uint16_t* vel_x, uint16_t* vel_y,
                                               uint16_t* vel_z,
                                               const uint16_t* pos_x,
                                               const uint16_t* pos_y,
                                               const uint16_t* pos_z,
                                               int* num_of_active_particles)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= *num_of_active_particles) // Maybe dynamic parallelism is a better
                                       // choice.
        return;

    // Already constrained by |num_of_active_particles|.
    //
    //if (IsCellUndefined(cell_index[i]))
    //    return;

    float x = __half2float(pos_x[i]);
    float y = __half2float(pos_y[i]);
    float z = __half2float(pos_z[i]);

    float v_x =  tex3D(tex_x,  x + 0.5f, y,        z);
    float v_y =  tex3D(tex_y,  x,        y + 0.5f, z);
    float v_z =  tex3D(tex_z,  x,        y,        z + 0.5f);

    float v_xp = tex3D(tex_xp, x + 0.5f, y,        z);
    float v_yp = tex3D(tex_yp, x,        y + 0.5f, z);
    float v_zp = tex3D(tex_zp, x,        y,        z + 0.5f);

    float ��_x = v_xp - v_x;
    float ��_y = v_yp - v_y;
    float ��_z = v_zp - v_z;

    // v_np1 = (1 - ��) * v_n_pic + �� * v_n_flip.
    // We are using �� = 1.
    vel_x[i] = __float2half_rn(__half2float(vel_x[i]) + ��_x);
    vel_y[i] = __float2half_rn(__half2float(vel_y[i]) + ��_y);
    vel_z[i] = __float2half_rn(__half2float(vel_z[i]) + ��_z);
}

// Should be invoked *AFTER* interpolation kernel. Since the newly inserted
// particles sample the new velocity filed, they don't need any correction.
//
// One should be very careful designing the mechanism of re-sampling: an
// important difference between particles and grid is that once a particle is
// created, its density and temperature are not gonna change during its life
// time(except decaying).
//
// Fields should be available: cell_index, particle_count.
// Active particles should be consecutive.
__global__ void ResampleKernel(FlipParticles particles, uint random_seed,
                               uint3 volume_size)
{
    int free_particles =
        particles.num_of_particles_ - *particles.num_of_actives_;
    if (free_particles < kMaxNumSamplesForOneTime)
        return; // No more free particles.

    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    uint cell_index = (z * volume_size.y + y) * volume_size.x + x;
    int count = particles.particle_count_[cell_index];

    // Scan for all undersampled cells, and try to insert new particles.
    if (count > kMinNumParticlesPerCell)
        return;

    // CAUTION: All the physics variables, except velocity, should always be
    //          updated directly to the particles, or these changes might never
    //          get a chance to be applied to the particles, since the re-sample
    //          kernel only concerns about the cells that not having sufficient
    //          particles.
    int needed = min(kMaxNumParticlesPerCell - count, kMaxNumSamplesForOneTime);
    if (needed <= 0)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float v_x =         tex3D(tex_x, coord.x + 0.5f, coord.y,        coord.z);
    float v_y =         tex3D(tex_y, coord.x,        coord.y + 0.5f, coord.z);
    float v_z =         tex3D(tex_z, coord.x,        coord.y,        coord.z + 0.5f);
    float density =     tex3D(tex_d, coord.x,        coord.y,        coord.z);
    float temperature = tex3D(tex_t, coord.x,        coord.y,        coord.z);

    if (!IsCellActive(v_x, v_y, v_z, density, temperature)) {
        // FIXME: Recycle inactive particles.
        return;
    }

    int base_index = atomicAdd(particles.num_of_actives_, needed);
    if (base_index + needed > particles.num_of_particles_) {
        atomicAdd(particles.num_of_actives_, -needed);
        return; // Not enough free particles.
    }

    // Reseed particles.
    uint seed = random_seed;
    for (int i = 0; i < needed; i++) {
        float3 pos = coord + RandomCoord(&seed);

        // TODO: Accelerate with shared memory.
        v_x         = tex3D(tex_x, pos.x + 0.5f, pos.y,        pos.z);
        v_y         = tex3D(tex_y, pos.x,        pos.y + 0.5f, pos.z);
        v_z         = tex3D(tex_z, pos.x,        pos.y,        pos.z + 0.5f);
        density     = tex3D(tex_d, pos.x,        pos.y,        pos.z);
        temperature = tex3D(tex_t, pos.x,        pos.y,        pos.z);

        int index = base_index + i;

        // Not necessary to initialize the in_cell_index field.
        // Particle-cell mapping will be done in the binding kernel.

        // Assign a valid value to |cell_index_| to activate this particle.
        particles.cell_index_ [index] = cell_index;
        particles.position_x_ [index] = __float2half_rn(pos.x);
        particles.position_y_ [index] = __float2half_rn(pos.y);
        particles.position_z_ [index] = __float2half_rn(pos.z);
        particles.velocity_x_ [index] = __float2half_rn(v_x);
        particles.velocity_y_ [index] = __float2half_rn(v_y);
        particles.velocity_z_ [index] = __float2half_rn(v_z);
        particles.density_    [index] = __float2half_rn(density);
        particles.temperature_[index] = __float2half_rn(temperature);
    }
}

__global__ void ResetParticlesKernel(FlipParticles particles)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= particles.num_of_particles_)
        return;

    FreeParticle(particles, i);
    particles.in_cell_index_ = 0;
    particles.velocity_x_ = 0;
    particles.velocity_y_ = 0;
    particles.velocity_z_ = 0;
    particles.position_x_ = 0;
    particles.position_y_ = 0;
    particles.position_z_ = 0;
    particles.density_ = 0;
    particles.temperature_ = 0;

    if (i == 0)
        *particles.num_of_actives_ = 0;
}

// Fields should be available: cell_index, in_cell_index
// Active particles may *NOT* be consecutive.
template <typename Type>
__global__ void SortFieldKernel(Type* field_np1, Type* field,
                                uint32_t* cell_index, uint8_t* in_cell_index,
                                uint32_t* particle_index, uint num_of_particles,
                                uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_of_particles)
        return;

    if (!IsCellUndefined(cell_index[i])) {
        uint sort_index = particle_index[cell_index[i]] + in_cell_index[i];
        field_np1[sort_index] = field[i];
    }
}

// Fields should be available: cell_index, in_cell_index
// Active particles may *NOT* be consecutive.
__global__ void SortParticlesKernel(FlipParticles p_aux, FlipParticles p_src,
                                    int last_cell_index, uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= p_src.num_of_particles_)
        return;

    if (i == 0) {
        // We need the number of active particles for allocation in the next
        // frame.
        *p_src.num_of_actives_ =
            p_src.particle_index_[last_cell_index] +
            p_src.particle_count_[last_cell_index];
    }

    uint cell_index = p_src.cell_index_[i];
    uint in_cell    = p_src.in_cell_index_[i];
    if (!IsCellUndefined(cell_index)) {
        uint sort_index = p_src.particle_index_[cell_index] + in_cell;

        p_aux.cell_index_   [sort_index] = p_src.cell_index_[i];
        p_aux.in_cell_index_[sort_index] = p_src.in_cell_index_[i];
        p_aux.position_x_   [sort_index] = p_src.position_x_[i];
        p_aux.position_y_   [sort_index] = p_src.position_y_[i];
        p_aux.position_z_   [sort_index] = p_src.position_z_[i];
        p_aux.velocity_x_   [sort_index] = p_src.velocity_x_[i];
        p_aux.velocity_y_   [sort_index] = p_src.velocity_y_[i];
        p_aux.velocity_z_   [sort_index] = p_src.velocity_z_[i];
        p_aux.density_      [sort_index] = p_src.density_[i];
        p_aux.temperature_  [sort_index] = p_src.temperature_[i];
    }
}

// =============================================================================

namespace kern_launcher
{
void BindParticlesToCells(const FlipParticles& particles, uint3 volume_size,
                          BlockArrangement* ba)
{
    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipError_t e = hipMemsetAsync(
        particles.in_cell_index_, 0,
        particles.num_of_particles_ * sizeof(*particles.in_cell_index_));
    assert(e == hipSuccess);
    if (e != hipSuccess)
        return;
    
    e = hipMemsetAsync(particles.particle_count_, 0,
                        num_of_cells * sizeof(*particles.particle_count_));
    assert(e == hipSuccess);
    if (e != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    BindParticlesToCellsKernel<<<grid, block>>>(particles, volume_size);
    DCHECK_KERNEL();
}

void EmitParticles(const FlipParticles& particles, float3 center_point,
                   float3 hotspot, float radius, float density,
                   float temperature, uint random_seed, uint3 volume_size,
                   BlockArrangement* ba)
{
    const int kHeatLayerThickness = 2;
    dim3 block(volume_size.x, kHeatLayerThickness, 1);
    dim3 grid;
    ba->ArrangeGrid(&grid, block, volume_size);
    grid.y = 1;
    EmitParticlesKernel<<<grid, block>>>(particles, center_point, hotspot,
                                         radius, density, temperature,
                                         random_seed, volume_size);
    DCHECK_KERNEL();
}

void InterpolateDeltaVelocity(const FlipParticles& particles, hipArray* vnp1_x,
                              hipArray* vnp1_y, hipArray* vnp1_z,
                              hipArray* vn_x, hipArray* vn_y, hipArray* vn_z,
                              BlockArrangement* ba)
{
    auto bound_xp = BindHelper::Bind(&tex_xp, vnp1_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_xp.error() != hipSuccess)
        return;

    auto bound_yp = BindHelper::Bind(&tex_yp, vnp1_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_yp.error() != hipSuccess)
        return;

    auto bound_zp = BindHelper::Bind(&tex_zp, vnp1_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_zp.error() != hipSuccess)
        return;

    auto bound_x = BindHelper::Bind(&tex_x, vn_x, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vn_y, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vn_z, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    InterpolateDeltaVelocityKernel<<<grid, block>>>(particles.velocity_x_,
                                                    particles.velocity_y_,
                                                    particles.velocity_z_,
                                                    particles.position_x_,
                                                    particles.position_y_,
                                                    particles.position_z_,
                                                    particles.num_of_actives_);
    DCHECK_KERNEL();
}

void Resample(const FlipParticles& particles, hipArray* vel_x,
              hipArray* vel_y, hipArray* vel_z, hipArray* density,
              hipArray* temperature, uint random_seed, uint3 volume_size,
              BlockArrangement* ba)
{
    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    auto bound_d = BindHelper::Bind(&tex_d, density, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_d.error() != hipSuccess)
        return;

    auto bound_t = BindHelper::Bind(&tex_t, temperature, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_t.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    ResampleKernel<<<grid, block>>>(particles, random_seed, volume_size);
    DCHECK_KERNEL();
}

void ResetParticles(const FlipParticles& particles, uint3 volume_size,
                    BlockArrangement* ba)
{
    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    ResetParticlesKernel<<<grid, block>>>(particles);

    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipMemsetAsync(particles.particle_index_, 0,
                    num_of_cells * sizeof(*particles.particle_index_));
    hipMemsetAsync(particles.particle_count_, 0,
                    num_of_cells * sizeof(*particles.particle_count_));
    DCHECK_KERNEL();
}

void FastSort(FlipParticles particles, FlipParticles aux,
              uint3 volume_size, BlockArrangement* ba)
{
    FlipParticles& p_src = particles;
    FlipParticles& p_aux = aux;
    int last_cell_index = volume_size.x * volume_size.y * volume_size.z - 1;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, p_src.num_of_particles_);
    SortParticlesKernel<<<grid, block>>>(p_aux, p_src, last_cell_index,
                                         volume_size);
    DCHECK_KERNEL();
}

void SortParticles(FlipParticles particles, int* num_active_particles,
                   FlipParticles aux, uint3 volume_size,
                   BlockArrangement* ba)
{
    if (aux.velocity_x_) {
        FastSort(particles, aux, volume_size, ba);
    } else {
        dim3 block;
        dim3 grid;
        ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);

        uint16_t* fields[] = {
            particles.position_x_,
            particles.position_y_,
            particles.position_z_,
            particles.velocity_x_,
            particles.velocity_y_,
            particles.velocity_z_,
            particles.density_,
            particles.temperature_
        };

        for (int i = 0; i < sizeof(fields) / sizeof(*fields); i++) {
            SortFieldKernel<<<grid, block>>>(aux.position_x_, fields[i],
                                             particles.cell_index_,
                                             particles.in_cell_index_,
                                             particles.particle_index_,
                                             particles.num_of_particles_,
                                             volume_size);
            DCHECK_KERNEL();

            hipError_t e = hipMemcpyAsync(
                fields[i], aux.position_x_,
                particles.num_of_particles_ * sizeof(*fields[i]),
                hipMemcpyDeviceToDevice);
            assert(e == hipSuccess);
            if (e != hipSuccess)
                return;
        }

        // Sort index fields.
        SortFieldKernel<<<grid, block>>>(aux.cell_index_, particles.cell_index_,
                                         particles.cell_index_,
                                         particles.in_cell_index_,
                                         particles.particle_index_,
                                         particles.num_of_particles_,
                                         volume_size);
        DCHECK_KERNEL();

        SortFieldKernel<<<grid, block>>>(aux.in_cell_index_,
                                         particles.in_cell_index_,
                                         particles.cell_index_,
                                         particles.in_cell_index_,
                                         particles.particle_index_,
                                         particles.num_of_particles_,
                                         volume_size);
        DCHECK_KERNEL();

        hipError_t e = hipMemcpyAsync(
            particles.cell_index_, aux.cell_index_,
            particles.num_of_particles_ * sizeof(*particles.cell_index_),
            hipMemcpyDeviceToDevice);
        assert(e == hipSuccess);
        if (e != hipSuccess)
            return;

        e = hipMemcpyAsync(
            particles.in_cell_index_, aux.in_cell_index_,
            particles.num_of_particles_ * sizeof(*particles.in_cell_index_),
            hipMemcpyDeviceToDevice);
        assert(e == hipSuccess);
        if (e != hipSuccess)
            return;

        int last_cell_index = volume_size.x * volume_size.y * volume_size.z - 1;
        CalculateNumberOfActiveParticles<<<1, 1>>>(particles, last_cell_index);
        DCHECK_KERNEL();
    }

    hipMemcpyAsync(num_active_particles, particles.num_of_actives_,
                    sizeof(*num_active_particles), hipMemcpyDeviceToHost);
}
}
