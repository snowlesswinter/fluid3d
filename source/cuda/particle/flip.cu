#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/aux_buffer_manager.h"
#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/fluid_impulse.h"
#include "cuda/particle/flip_common.cuh"
#include "flip.h"
#include "random.cuh"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
surface<void, cudaSurfaceType3D> surf_d;
surface<void, cudaSurfaceType3D> surf_t;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_xp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_yp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_zp;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_d;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_t;

__device__ bool IsCellActive(float v_x, float v_y, float v_z, float density,
                             float temperature)
{
    const float kEpsilon = 0.0001f;
    return !IsStopped(v_x, v_y, v_z) || density > kEpsilon ||
            temperature > kEpsilon;
}

struct HorizontalEmission
{
    __device__ static bool OutsideVolume(uint x, uint y, uint z,
                                         const uint3& volume_size)
    {
        return y >= volume_size.y || z >= volume_size.z;
    }
    __device__ static float CalculateRadius(const float3& coord,
                                            const float3& center,
                                            const float3& hotspot)
    {
        float2 diff =
            make_float2(coord.y, coord.z) - make_float2(center.y, center.z);
        return hypotf(diff.x, diff.y);
    }
    __device__ static void SetVelX(uint16_t* vel_x, const float3 velocity)
    {
        *vel_x = __float2half_rn(velocity.x);
    }
};

struct VerticalEmission
{
    __device__ static bool OutsideVolume(uint x, uint y, uint z,
                                         const uint3& volume_size)
    {
        return x >= volume_size.x || z >= volume_size.z;
    }
    __device__ static float CalculateRadius(const float3& coord,
                                            const float3& center,
                                            const float3& hotspot)
    {
        float2 diff =
            make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
        return hypotf(diff.x, diff.y);
    }
    __device__ static void SetVelX(uint16_t* vel_x, const float3 velocity)
    {
    }
};

// NOTE: Assuming never overflows/underflows.
template <int Increment>
__device__ uint8_t AtomicIncrementUint8(uint8_t* addr)
{
    uint r = 0;
    uint* base_addr =
        reinterpret_cast<uint*>(reinterpret_cast<size_t>(addr) & ~3);
    switch (reinterpret_cast<size_t>(addr) & 3) {
        case 0:
            r = atomicAdd(base_addr, static_cast<uint>(Increment));
            return static_cast<uint8_t>(r & 0xFF);
        case 1:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 8);
            return static_cast<uint8_t>((r >> 8) & 0xFF);
        case 2:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 16);
            return static_cast<uint8_t>((r >> 16) & 0xFF);
        case 3:
            r = atomicAdd(base_addr, static_cast<uint>(Increment) << 24);
            return static_cast<uint8_t>((r >> 24) & 0xFF);
    }

    return 0;
}

// =============================================================================

// Fields should be reset: particle_count, in_cell_index
// Fields should be available: cell_index.
// Active particles may *NOT* be consecutive.
__global__ void BindParticlesToCellsKernel(FlipParticles particles,
                                           uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= particles.num_of_particles_)
        return;

    uint cell_index = particles.cell_index_[i];
    if (IsCellUndefined(cell_index))
        return;

    // TODO: Free particles in resample kernel?
    uint* p_count = particles.particle_count_;
    if (p_count[cell_index] >= kMaxNumParticlesPerCell) {
        FreeParticle(particles, i);
    } else {
        uint old_count = atomicAdd(p_count + cell_index, 1);
        if (old_count >= kMaxNumParticlesPerCell) {
            atomicAdd(p_count + cell_index, static_cast<uint>(-1));
            FreeParticle(particles, i);
        } else {
            particles.in_cell_index_[i] = old_count;
        }
    }
}

__global__ void CalculateNumberOfActiveParticles(FlipParticles particles,
                                                 int last_cell_index)
{
    *particles.num_of_actives_ =
        particles.particle_index_[last_cell_index] +
        particles.particle_count_[last_cell_index];
}

__global__ void DiffuseAndDecayKernel(FlipParticles particles, float time_step,
                                      float velocity_dissipation,
                                      float density_dissipation,
                                      float temperature_dissipation)
{
    FlipParticles& p = particles;

    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= *p.num_of_actives_)
        return;

    particles.velocity_x_ [i] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(particles.velocity_x_[i]));
    particles.velocity_y_ [i] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(particles.velocity_y_[i]));
    particles.velocity_z_ [i] = __float2half_rn((1.0f - velocity_dissipation    * time_step) * __half2float(particles.velocity_z_[i]));
    particles.density_    [i] = __float2half_rn((1.0f - density_dissipation     * time_step) * __half2float(particles.density_[i]));
    particles.temperature_[i] = __float2half_rn((1.0f - temperature_dissipation * time_step) * __half2float(particles.temperature_[i]));
}

// Fields should be available: cell_index, particle_count, particle_index.
template <typename Emission>
__global__ void EmitFlipParticlesKernel(FlipParticles particles, float3 center,
                                        float3 hotspot, float radius,
                                        float density, float temperature,
                                        float3 velocity, uint random_seed,
                                        uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (Emission::OutsideVolume(x, y, z, volume_size))
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float d = Emission::CalculateRadius(coord, center, hotspot);
    if (d >= radius)
        return;

    uint cell_index = (z * volume_size.y + y) * volume_size.x + x;
    int count = particles.particle_count_[cell_index];
    if (!count) {
        int new_particles = kMaxNumSamplesForOneTime;
        int base_index = atomicAdd(particles.num_of_actives_, new_particles);
        if (base_index + new_particles > particles.num_of_particles_) {
            atomicAdd(particles.num_of_actives_, -new_particles);
            return; // Not enough free particles.
        }

        particles.particle_count_[cell_index] += new_particles;
        uint seed = random_seed;
        for (int i = 0; i < new_particles; i++) {
            float3 pos = coord + RandomCoord(&seed);

            int index = base_index + i;

            // Not necessary to initialize the in_cell_index field.
            // Particle-cell mapping will be done in the binding kernel.

            // Assign a valid value to |cell_index_| to activate this particle.
            particles.cell_index_ [index] = cell_index;
            particles.position_x_ [index] = __float2half_rn(pos.x);
            particles.position_y_ [index] = __float2half_rn(pos.y);
            particles.position_z_ [index] = __float2half_rn(pos.z);
            particles.velocity_x_ [index] = 0;
            particles.velocity_y_ [index] = 0;
            particles.velocity_z_ [index] = 0;
            particles.density_    [index] = __float2half_rn(density);
            particles.temperature_[index] = __float2half_rn(temperature);

            Emission::SetVelX(&particles.velocity_x_[index], velocity);
        }
    } else {
        uint p_index = particles.particle_index_[cell_index];
        for (int i = 0; i < count; i++) {
            particles.density_    [p_index + i] = __float2half_rn(density);
            particles.temperature_[p_index + i] = __float2half_rn(temperature);

            Emission::SetVelX(&particles.velocity_x_[p_index + i], velocity);
        }
    }
}

// Fields should be available: cell_index, particle_count, particle_index.
__global__ void EmitFlipParticlesFromSphereKernel(FlipParticles particles,
                                                  float3 center, float radius,
                                                  float density,
                                                  float temperature,
                                                  float velocity,
                                                  uint random_seed,
                                                  uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = coord - center;
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d >= radius)
        return;

    uint cell_index = (z * volume_size.y + y) * volume_size.x + x;
    int count = particles.particle_count_[cell_index];
    if (!count) {
        int new_particles = kMaxNumSamplesForOneTime;
        int base_index = atomicAdd(particles.num_of_actives_, new_particles);
        if (base_index + new_particles > particles.num_of_particles_) {
            atomicAdd(particles.num_of_actives_, -new_particles);
            return; // Not enough free particles.
        }

        particles.particle_count_[cell_index] += new_particles;
        uint seed = random_seed;
        for (int i = 0; i < new_particles; i++) {
            float3 pos = coord + RandomCoord(&seed);

            int index = base_index + i;

            float3 dir = pos - center;
            float3 vel = normalize(dir) * velocity;

            // Not necessary to initialize the in_cell_index field.
            // Particle-cell mapping will be done in the binding kernel.

            // Assign a valid value to |cell_index_| to activate this particle.
            particles.cell_index_ [index] = cell_index;
            particles.position_x_ [index] = __float2half_rn(pos.x);
            particles.position_y_ [index] = __float2half_rn(pos.y);
            particles.position_z_ [index] = __float2half_rn(pos.z);
            particles.velocity_x_ [index] = __float2half_rn(vel.x);
            particles.velocity_y_ [index] = __float2half_rn(vel.y);
            particles.velocity_z_ [index] = __float2half_rn(vel.z);
            particles.density_    [index] = __float2half_rn(density);
            particles.temperature_[index] = __float2half_rn(temperature);
        }
    } else {
        uint p_index = particles.particle_index_[cell_index];
        for (int i = 0; i < count; i++) {
            float pos_x = __half2float(particles.position_x_[p_index + i]);
            float pos_y = __half2float(particles.position_y_[p_index + i]);
            float pos_z = __half2float(particles.position_z_[p_index + i]);
            float3 pos = make_float3(pos_x, pos_y, pos_z);

            float3 dir = pos - center;
            float3 vel = normalize(dir) * velocity;

            particles.velocity_x_ [p_index + i] = __float2half_rn(vel.x);
            particles.velocity_y_ [p_index + i] = __float2half_rn(vel.y);
            particles.velocity_z_ [p_index + i] = __float2half_rn(vel.z);
            particles.density_    [p_index + i] = __float2half_rn(density);
            particles.temperature_[p_index + i] = __float2half_rn(temperature);
        }
    }
}

// Should be invoked *BEFORE* resample kernel. Please read the comments of
// ResampleKernel().
// Active particles should be consecutive.
__global__ void InterpolateDeltaVelocityKernel(uint16_t* vel_x, uint16_t* vel_y,
                                               uint16_t* vel_z,
                                               const uint16_t* pos_x,
                                               const uint16_t* pos_y,
                                               const uint16_t* pos_z,
                                               int* num_of_active_particles)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= *num_of_active_particles) // Maybe dynamic parallelism is a better
                                       // choice.
        return;

    // Already constrained by |num_of_active_particles|.
    //
    //if (IsCellUndefined(cell_index[i]))
    //    return;

    float x = __half2float(pos_x[i]);
    float y = __half2float(pos_y[i]);
    float z = __half2float(pos_z[i]);

    float v_x =  tex3D(tex_x,  x + 0.5f, y,        z);
    float v_y =  tex3D(tex_y,  x,        y + 0.5f, z);
    float v_z =  tex3D(tex_z,  x,        y,        z + 0.5f);

    float v_xp = tex3D(tex_xp, x + 0.5f, y,        z);
    float v_yp = tex3D(tex_yp, x,        y + 0.5f, z);
    float v_zp = tex3D(tex_zp, x,        y,        z + 0.5f);

    float ��_x = v_xp - v_x;
    float ��_y = v_yp - v_y;
    float ��_z = v_zp - v_z;

    // v_np1 = (1 - ��) * v_n_pic + �� * v_n_flip.
    // We are using �� = 1.
    vel_x[i] = __float2half_rn(__half2float(vel_x[i]) + ��_x);
    vel_y[i] = __float2half_rn(__half2float(vel_y[i]) + ��_y);
    vel_z[i] = __float2half_rn(__half2float(vel_z[i]) + ��_z);
}

// Should be invoked *AFTER* interpolation kernel. Since the newly inserted
// particles sample the new velocity filed, they don't need any correction.
//
// One should be very careful designing the mechanism of re-sampling: an
// important difference between particles and grid is that once a particle is
// created, its density and temperature are not gonna change during its life
// time(except decaying).
//
// Fields should be available: cell_index, particle_count.
// Active particles should be consecutive.
__global__ void ResampleKernel(FlipParticles particles, uint random_seed,
                               uint3 volume_size)
{
    int free_particles =
        particles.num_of_particles_ - *particles.num_of_actives_;
    if (free_particles < kMaxNumSamplesForOneTime)
        return; // No more free particles.

    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    uint cell_index = (z * volume_size.y + y) * volume_size.x + x;
    int count = particles.particle_count_[cell_index];

    // Scan for all undersampled cells, and try to insert new particles.
    if (count > kMinNumParticlesPerCell)
        return;

    // CAUTION: All the physics variables, except velocity, should always be
    //          updated directly to the particles, or these changes might never
    //          get a chance to be applied to the particles, since the re-sample
    //          kernel only concerns about the cells that not having sufficient
    //          particles.
    int needed = min(kMaxNumParticlesPerCell - count, kMaxNumSamplesForOneTime);
    if (needed <= 0)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float v_x =         tex3D(tex_x, coord.x + 0.5f, coord.y,        coord.z);
    float v_y =         tex3D(tex_y, coord.x,        coord.y + 0.5f, coord.z);
    float v_z =         tex3D(tex_z, coord.x,        coord.y,        coord.z + 0.5f);
    float density =     tex3D(tex_d, coord.x,        coord.y,        coord.z);
    float temperature = tex3D(tex_t, coord.x,        coord.y,        coord.z);

    if (!IsCellActive(v_x, v_y, v_z, density, temperature)) {
        // FIXME: Recycle inactive particles.
        return;
    }

    int base_index = atomicAdd(particles.num_of_actives_, needed);
    if (base_index + needed > particles.num_of_particles_) {
        atomicAdd(particles.num_of_actives_, -needed);
        return; // Not enough free particles.
    }

    // Reseed particles.
    uint seed = random_seed;
    for (int i = 0; i < needed; i++) {
        float3 pos = coord + RandomCoord(&seed);

        // TODO: Accelerate with shared memory.
        v_x         = tex3D(tex_x, pos.x + 0.5f, pos.y,        pos.z);
        v_y         = tex3D(tex_y, pos.x,        pos.y + 0.5f, pos.z);
        v_z         = tex3D(tex_z, pos.x,        pos.y,        pos.z + 0.5f);
        density     = tex3D(tex_d, pos.x,        pos.y,        pos.z);
        temperature = tex3D(tex_t, pos.x,        pos.y,        pos.z);

        int index = base_index + i;

        // Not necessary to initialize the in_cell_index field.
        // Particle-cell mapping will be done in the binding kernel.

        // Assign a valid value to |cell_index_| to activate this particle.
        particles.cell_index_ [index] = cell_index;
        particles.position_x_ [index] = __float2half_rn(pos.x);
        particles.position_y_ [index] = __float2half_rn(pos.y);
        particles.position_z_ [index] = __float2half_rn(pos.z);
        particles.velocity_x_ [index] = __float2half_rn(v_x);
        particles.velocity_y_ [index] = __float2half_rn(v_y);
        particles.velocity_z_ [index] = __float2half_rn(v_z);
        particles.density_    [index] = __float2half_rn(density);
        particles.temperature_[index] = __float2half_rn(temperature);
    }
}

__global__ void ResetParticlesKernel(FlipParticles particles)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= particles.num_of_particles_)
        return;

    FreeParticle(particles, i);
    particles.in_cell_index_ = 0;
    particles.velocity_x_ = 0;
    particles.velocity_y_ = 0;
    particles.velocity_z_ = 0;
    particles.position_x_ = 0;
    particles.position_y_ = 0;
    particles.position_z_ = 0;
    particles.density_ = 0;
    particles.temperature_ = 0;

    if (i == 0)
        *particles.num_of_actives_ = 0;
}

// Fields should be available: cell_index, in_cell_index
// Active particles may *NOT* be consecutive.
template <typename Type>
__global__ void SortFieldKernel(Type* field_np1, Type* field,
                                uint32_t* cell_index, uint8_t* in_cell_index,
                                uint32_t* particle_index, uint num_of_particles,
                                uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_of_particles)
        return;

    if (!IsCellUndefined(cell_index[i])) {
        uint sort_index = particle_index[cell_index[i]] + in_cell_index[i];
        field_np1[sort_index] = field[i];
    }
}

// Fields should be available: cell_index, in_cell_index, particle_index
// Active particles may *NOT* be consecutive.
__global__ void SortParticlesKernel(FlipParticles p_aux, FlipParticles p_src,
                                    int last_cell_index, uint3 volume_size)
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= p_src.num_of_particles_)
        return;

    if (i == 0) {
        // We need the number of active particles for allocation in the next
        // frame.
        *p_src.num_of_actives_ =
            p_src.particle_index_[last_cell_index] +
            p_src.particle_count_[last_cell_index];
    }

    uint cell_index = p_src.cell_index_[i];
    uint in_cell    = p_src.in_cell_index_[i];
    if (!IsCellUndefined(cell_index)) {
        uint sort_index = p_src.particle_index_[cell_index] + in_cell;

        p_aux.cell_index_   [sort_index] = p_src.cell_index_[i];
        p_aux.in_cell_index_[sort_index] = p_src.in_cell_index_[i];
        p_aux.position_x_   [sort_index] = p_src.position_x_[i];
        p_aux.position_y_   [sort_index] = p_src.position_y_[i];
        p_aux.position_z_   [sort_index] = p_src.position_z_[i];
        p_aux.velocity_x_   [sort_index] = p_src.velocity_x_[i];
        p_aux.velocity_y_   [sort_index] = p_src.velocity_y_[i];
        p_aux.velocity_z_   [sort_index] = p_src.velocity_z_[i];
        p_aux.density_      [sort_index] = p_src.density_[i];
        p_aux.temperature_  [sort_index] = p_src.temperature_[i];
    }
}

// =============================================================================

namespace kern_launcher
{
void BindParticlesToCells(const FlipParticles& particles, uint3 volume_size,
                          BlockArrangement* ba)
{
    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipError_t e = hipMemsetAsync(
        particles.in_cell_index_, 0,
        particles.num_of_particles_ * sizeof(*particles.in_cell_index_));
    assert(e == hipSuccess);
    if (e != hipSuccess)
        return;
    
    e = hipMemsetAsync(particles.particle_count_, 0,
                        num_of_cells * sizeof(*particles.particle_count_));
    assert(e == hipSuccess);
    if (e != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    BindParticlesToCellsKernel<<<grid, block>>>(particles, volume_size);
    DCHECK_KERNEL();
}

void DiffuseAndDecay(const FlipParticles& particles, float time_step,
                     float velocity_dissipation, float density_dissipation,
                     float temperature_dissipation, BlockArrangement* ba)
{
    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);

    DiffuseAndDecayKernel<<<grid, block>>>(particles, time_step,
                                           velocity_dissipation,
                                           density_dissipation,
                                           temperature_dissipation);
}

void EmitFlipParticles(const FlipParticles& particles, float3 center,
                       float3 hotspot, float radius, float density,
                       float temperature, float3 velocity, FluidImpulse impulse,
                       uint random_seed, uint3 volume_size,
                       BlockArrangement* ba)
{

    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            const float kHeatLayerThickness = 0.025f * volume_size.y;
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(std::ceil(kHeatLayerThickness));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesKernel<VerticalEmission><<<grid, block>>>(
                particles, center, hotspot, radius, density, temperature,
                velocity, random_seed, volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(std::ceil(radius + center.y));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesFromSphereKernel<<<grid, block>>>(
                particles, center, radius, density, temperature, velocity.x,
                random_seed, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            const float kHeatLayerThickness = 0.02f * volume_size.x;
            uint3 actual_size = volume_size;
            actual_size.x = static_cast<uint>(std::ceil(kHeatLayerThickness));

            dim3 grid;
            dim3 block;
            ba->ArrangeRowScan(&grid, &block, actual_size);
            EmitFlipParticlesKernel<HorizontalEmission><<<grid, block>>>(
                particles, center, hotspot, radius, density, temperature,
                velocity, random_seed, volume_size);
            break;
        }
    }
    
    DCHECK_KERNEL();
}

void InterpolateDeltaVelocity(const FlipParticles& particles, hipArray* vnp1_x,
                              hipArray* vnp1_y, hipArray* vnp1_z,
                              hipArray* vn_x, hipArray* vn_y, hipArray* vn_z,
                              BlockArrangement* ba)
{
    auto bound_xp = BindHelper::Bind(&tex_xp, vnp1_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_xp.error() != hipSuccess)
        return;

    auto bound_yp = BindHelper::Bind(&tex_yp, vnp1_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_yp.error() != hipSuccess)
        return;

    auto bound_zp = BindHelper::Bind(&tex_zp, vnp1_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_zp.error() != hipSuccess)
        return;

    auto bound_x = BindHelper::Bind(&tex_x, vn_x, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vn_y, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vn_z, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    InterpolateDeltaVelocityKernel<<<grid, block>>>(particles.velocity_x_,
                                                    particles.velocity_y_,
                                                    particles.velocity_z_,
                                                    particles.position_x_,
                                                    particles.position_y_,
                                                    particles.position_z_,
                                                    particles.num_of_actives_);
    DCHECK_KERNEL();
}

void Resample(const FlipParticles& particles, hipArray* vel_x,
              hipArray* vel_y, hipArray* vel_z, hipArray* density,
              hipArray* temperature, uint random_seed, uint3 volume_size,
              BlockArrangement* ba)
{
    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    auto bound_d = BindHelper::Bind(&tex_d, density, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_d.error() != hipSuccess)
        return;

    auto bound_t = BindHelper::Bind(&tex_t, temperature, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_t.error() != hipSuccess)
        return;

    dim3 grid;
    dim3 block;
    ba->ArrangePrefer3dLocality(&grid, &block, volume_size);
    ResampleKernel<<<grid, block>>>(particles, random_seed, volume_size);
    DCHECK_KERNEL();
}

void ResetParticles(const FlipParticles& particles, uint3 volume_size,
                    BlockArrangement* ba)
{
    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);
    ResetParticlesKernel<<<grid, block>>>(particles);

    uint num_of_cells = volume_size.x * volume_size.y * volume_size.z;
    hipMemsetAsync(particles.particle_index_, 0,
                    num_of_cells * sizeof(*particles.particle_index_));
    hipMemsetAsync(particles.particle_count_, 0,
                    num_of_cells * sizeof(*particles.particle_count_));
    DCHECK_KERNEL();
}

void FastSort(FlipParticles particles, FlipParticles aux,
              uint3 volume_size, BlockArrangement* ba)
{
    FlipParticles& p_src = particles;
    FlipParticles& p_aux = aux;
    int last_cell_index = volume_size.x * volume_size.y * volume_size.z - 1;

    dim3 block;
    dim3 grid;
    ba->ArrangeLinear(&grid, &block, p_src.num_of_particles_);
    SortParticlesKernel<<<grid, block>>>(p_aux, p_src, last_cell_index,
                                         volume_size);
    DCHECK_KERNEL();
}

void SortParticles(FlipParticles particles, int* num_active_particles,
                   FlipParticles aux, uint3 volume_size,
                   BlockArrangement* ba)
{
    if (aux.velocity_x_) {
        FastSort(particles, aux, volume_size, ba);
    } else {
        dim3 block;
        dim3 grid;
        ba->ArrangeLinear(&grid, &block, particles.num_of_particles_);

        uint16_t* fields[] = {
            particles.position_x_,
            particles.position_y_,
            particles.position_z_,
            particles.velocity_x_,
            particles.velocity_y_,
            particles.velocity_z_,
            particles.density_,
            particles.temperature_
        };

        for (int i = 0; i < sizeof(fields) / sizeof(*fields); i++) {
            SortFieldKernel<<<grid, block>>>(aux.position_x_, fields[i],
                                             particles.cell_index_,
                                             particles.in_cell_index_,
                                             particles.particle_index_,
                                             particles.num_of_particles_,
                                             volume_size);
            DCHECK_KERNEL();

            hipError_t e = hipMemcpyAsync(
                fields[i], aux.position_x_,
                particles.num_of_particles_ * sizeof(*fields[i]),
                hipMemcpyDeviceToDevice);
            assert(e == hipSuccess);
            if (e != hipSuccess)
                return;
        }

        // Sort index fields.
        SortFieldKernel<<<grid, block>>>(aux.cell_index_, particles.cell_index_,
                                         particles.cell_index_,
                                         particles.in_cell_index_,
                                         particles.particle_index_,
                                         particles.num_of_particles_,
                                         volume_size);
        DCHECK_KERNEL();

        SortFieldKernel<<<grid, block>>>(aux.in_cell_index_,
                                         particles.in_cell_index_,
                                         particles.cell_index_,
                                         particles.in_cell_index_,
                                         particles.particle_index_,
                                         particles.num_of_particles_,
                                         volume_size);
        DCHECK_KERNEL();

        hipError_t e = hipMemcpyAsync(
            particles.cell_index_, aux.cell_index_,
            particles.num_of_particles_ * sizeof(*particles.cell_index_),
            hipMemcpyDeviceToDevice);
        assert(e == hipSuccess);
        if (e != hipSuccess)
            return;

        e = hipMemcpyAsync(
            particles.in_cell_index_, aux.in_cell_index_,
            particles.num_of_particles_ * sizeof(*particles.in_cell_index_),
            hipMemcpyDeviceToDevice);
        assert(e == hipSuccess);
        if (e != hipSuccess)
            return;

        int last_cell_index = volume_size.x * volume_size.y * volume_size.z - 1;
        CalculateNumberOfActiveParticles<<<1, 1>>>(particles, last_cell_index);
        DCHECK_KERNEL();
    }

    hipMemcpyAsync(num_active_particles, particles.num_of_actives_,
                    sizeof(*num_active_particles), hipMemcpyDeviceToHost);
}
}
