#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> tex_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_div;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vort_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vort_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vort_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_conf_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_conf_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_conf_z;

__global__ void ApplyVorticityConfinementStaggeredKernel()
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x == 0 || y == 0 || z == 0)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float conf_x = tex3D(tex_conf_x, coord.x - 0.5f, coord.y + 0.5f, coord.z + 0.5f);
    float conf_y = tex3D(tex_conf_y, coord.x + 0.5f, coord.y - 0.5f, coord.z + 0.5f);
    float conf_z = tex3D(tex_conf_z, coord.x + 0.5f, coord.y + 0.5f, coord.z - 0.5f);
    float4 velocity = tex3D(tex_velocity, coord.x, coord.y, coord.z);
    ushort4 result = make_ushort4(__float2half_rn(velocity.x + conf_x),
                                  __float2half_rn(velocity.y + conf_y),
                                  __float2half_rn(velocity.z + conf_z),
                                  0);
    surf3Dwrite(result, surf, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void BuildVorticityConfinementStaggeredKernel(
    float coeff, float cell_size, float half_inverse_cell_size,
    uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    // Calculate the gradient of vorticity.
    float near_vort_x =   tex3D(tex_vort_x, coord.x,        coord.y + 0.5f, coord.z - 0.5f);
    float near_vort_y =   tex3D(tex_vort_y, coord.x + 0.5f, coord.y,        coord.z - 0.5f);
    float near_vort_z =   tex3D(tex_vort_z, coord.x + 0.5f, coord.y + 0.5f, coord.z - 1.0f);
    float near_vort = sqrtf(near_vort_x * near_vort_x + near_vort_y * near_vort_y + near_vort_z * near_vort_z);

    float south_vort_x =  tex3D(tex_vort_x, coord.x,        coord.y - 0.5f, coord.z + 0.5f);
    float south_vort_y =  tex3D(tex_vort_y, coord.x + 0.5f, coord.y - 1.0f, coord.z + 0.5f);
    float south_vort_z =  tex3D(tex_vort_z, coord.x + 0.5f, coord.y - 0.5f, coord.z);
    float south_vort = sqrtf(south_vort_x * south_vort_x + south_vort_y * south_vort_y + south_vort_z * south_vort_z);

    float west_vort_x =   tex3D(tex_vort_x, coord.x - 1.0f, coord.y + 0.5f, coord.z + 0.5f);
    float west_vort_y =   tex3D(tex_vort_y, coord.x - 0.5f, coord.y,        coord.z + 0.5f);
    float west_vort_z =   tex3D(tex_vort_z, coord.x - 0.5f, coord.y + 0.5f, coord.z);
    float west_vort = sqrtf(west_vort_x * west_vort_x + west_vort_y * west_vort_y + west_vort_z * west_vort_z);

    float center_vort_x = tex3D(tex_vort_x, coord.x,        coord.y,        coord.z);
    float center_vort_y = tex3D(tex_vort_y, coord.x,        coord.y,        coord.z);
    float center_vort_z = tex3D(tex_vort_z, coord.x,        coord.y,        coord.z);

    float east_vort_x =   tex3D(tex_vort_x, coord.x + 1.0f, coord.y + 0.5f, coord.z + 0.5f);
    float east_vort_y =   tex3D(tex_vort_y, coord.x + 1.5f, coord.y,        coord.z + 0.5f);
    float east_vort_z =   tex3D(tex_vort_z, coord.x + 1.5f, coord.y + 0.5f, coord.z);
    float east_vort = sqrtf(east_vort_x * east_vort_x + east_vort_y * east_vort_y + east_vort_z * east_vort_z);

    float north_vort_x =  tex3D(tex_vort_x, coord.x,        coord.y + 1.5f, coord.z + 0.5f);
    float north_vort_y =  tex3D(tex_vort_y, coord.x + 0.5f, coord.y + 1.0f, coord.z + 0.5f);
    float north_vort_z =  tex3D(tex_vort_z, coord.x + 0.5f, coord.y + 1.5f, coord.z);
    float north_vort = sqrtf(north_vort_x * north_vort_x + north_vort_y * north_vort_y + north_vort_z * north_vort_z);

    float far_vort_x =    tex3D(tex_vort_x, coord.x,        coord.y + 0.5f, coord.z + 1.5f);
    float far_vort_y =    tex3D(tex_vort_y, coord.x + 0.5f, coord.y,        coord.z + 1.5f);
    float far_vort_z =    tex3D(tex_vort_z, coord.x + 0.5f, coord.y + 0.5f, coord.z + 1.0f);
    float far_vort = sqrtf(far_vort_x * far_vort_x + far_vort_y * far_vort_y + far_vort_z * far_vort_z);

    // Calculate normalized ��.
    float ��_x = half_inverse_cell_size * (east_vort - west_vort);
    float ��_y = half_inverse_cell_size * (north_vort - south_vort);
    float ��_z = half_inverse_cell_size * (far_vort - near_vort);

    float ��_mag = sqrtf(��_x * ��_x + ��_y * ��_y + ��_z * ��_z + 0.00001f);
    ��_x /= ��_mag;
    ��_y /= ��_mag;
    ��_z /= ��_mag;

    // Vorticity confinement at the center of the grid.
    float tex_conf_x = coeff * cell_size * (��_y * center_vort_z - ��_z * center_vort_y);
    float tex_conf_y = coeff * cell_size * (��_z * center_vort_x - ��_x * center_vort_z);
    float tex_conf_z = coeff * cell_size * (��_x * center_vort_y - ��_y * center_vort_x);

    ushort result_x = __float2half_rn(tex_conf_x);
    surf3Dwrite(result_x, surf_x, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);

    ushort result_y = __float2half_rn(tex_conf_y);
    surf3Dwrite(result_y, surf_y, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);

    ushort result_z = __float2half_rn(tex_conf_z);
    surf3Dwrite(result_z, surf_z, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void ComputeCurlStaggeredKernel(uint3 volume_size,
                                           float inverse_cell_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 result;
    if (x < volume_size.x - 1 && y < volume_size.y - 1 && z < volume_size.z - 1) {
        if (x > 0 && y > 0 && z > 0) {
            float4 v_near =  tex3D(tex_velocity, coord.x, coord.y, coord.z - 1.0f);
            float4 v_west =  tex3D(tex_velocity, coord.x - 1.0f, coord.y, coord.z);
            float4 v_south = tex3D(tex_velocity, coord.x, coord.y - 1.0f, coord.z);
            float4 v =       tex3D(tex_velocity, coord.x, coord.y, coord.z);

            result.x = inverse_cell_size * (v.z - v_south.z - v.y + v_near.y);
            result.y = inverse_cell_size * (v.x - v_near.x  - v.z + v_west.z);
            result.z = inverse_cell_size * (v.y - v_west.y  - v.x + v_south.x);
        } else if (x == 0) {
            result.x = tex3D(tex_vort_x, coord.x + 1.0f, coord.y, coord.z);
            result.y = tex3D(tex_vort_y, coord.x + 1.0f, coord.y, coord.z);
            result.z = tex3D(tex_vort_z, coord.x + 1.0f, coord.y, coord.z);
        } else if (y == 0) {
            result.x = tex3D(tex_vort_x, coord.x, coord.y + 1.0f, coord.z);
            result.y = tex3D(tex_vort_y, coord.x, coord.y + 1.0f, coord.z);
            result.z = tex3D(tex_vort_z, coord.x, coord.y + 1.0f, coord.z);
        } else {
            result.x = tex3D(tex_vort_x, coord.x, coord.y, coord.z + 1.0f);
            result.y = tex3D(tex_vort_y, coord.x, coord.y, coord.z + 1.0f);
            result.z = tex3D(tex_vort_z, coord.x, coord.y, coord.z + 1.0f);
        }
    } else if (x == volume_size.x - 1) {
        result.x = tex3D(tex_vort_x, coord.x - 1.0f, coord.y, coord.z);
        result.y = tex3D(tex_vort_y, coord.x - 1.0f, coord.y, coord.z);
        result.z = tex3D(tex_vort_z, coord.x - 1.0f, coord.y, coord.z);
    } else if (y == volume_size.y - 1) {
        result.x = tex3D(tex_vort_x, coord.x, coord.y - 1.0f, coord.z);
        result.y = tex3D(tex_vort_y, coord.x, coord.y - 1.0f, coord.z);
        result.z = tex3D(tex_vort_z, coord.x, coord.y - 1.0f, coord.z);
    } else {
        result.x = tex3D(tex_vort_x, coord.x, coord.y, coord.z - 1.0f);
        result.y = tex3D(tex_vort_y, coord.x, coord.y, coord.z - 1.0f);
        result.z = tex3D(tex_vort_z, coord.x, coord.y, coord.z - 1.0f);
    }

    ushort raw_x = __float2half_rn(result.x);
    surf3Dwrite(raw_x, surf_x, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    ushort raw_y = __float2half_rn(result.y);
    surf3Dwrite(raw_y, surf_y, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    ushort raw_z = __float2half_rn(result.z);
    surf3Dwrite(raw_z, surf_z, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void ComputeDivergenceStaggeredKernelForVort(float inverse_cell_size,
                                                        uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float4 base =   tex3D(tex_velocity, coord.x,        coord.y,        coord.z);
    float  east =   tex3D(tex_velocity, coord.x + 1.0f, coord.y,        coord.z).x;
    float  north =  tex3D(tex_velocity, coord.x,        coord.y + 1.0f, coord.z).y;
    float  far =    tex3D(tex_velocity, coord.x,        coord.y,        coord.z + 1.0f).z;

    float diff_ew = east  - base.x;
    float diff_ns = north - base.y;
    float diff_fn = far   - base.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -base.x;

    if (y >= volume_size.y - 1)
        diff_ns = -base.y;

    if (z >= volume_size.z - 1)
        diff_fn = -base.z;

    float div = inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    ushort result = __float2half_rn(div);
    surf3Dwrite(result, surf, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void DecayVorticesStaggeredKernel(float time_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float div_x = tex3D(tex_div, coord.x, coord.y - 0.5f, coord.z - 0.5f);
    float coef_x = fminf(0.0f, -div_x * time_step);

    float vort_x = tex3D(tex_vort_x, coord.x, coord.y, coord.z);
    ushort result_x = __float2half_rn(vort_x * __expf(coef_x));
    surf3Dwrite(result_x, surf, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    float div_y = tex3D(tex_div, coord.x - 0.5f, coord.y, coord.z - 0.5f);
    float coef_y = fminf(0.0f, -div_y * time_step);

    float vort_y = tex3D(tex_vort_y, coord.x, coord.y, coord.z);
    ushort result_y = __float2half_rn(vort_y * __expf(coef_y));
    surf3Dwrite(result_y, surf, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    float div_z = tex3D(tex_div, coord.x - 0.5f, coord.y - 0.5f, coord.z);
    float coef_z = fminf(0.0f, -div_z * time_step);

    float vort_z = tex3D(tex_vort_z, coord.x, coord.y, coord.z);
    ushort result_z = __float2half_rn(vort_z * __expf(coef_z));
    surf3Dwrite(result_z, surf, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void StretchVortexStaggeredKernel(float scale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float ��_xx = tex3D(tex_vort_x, coord.x, coord.y, coord.z);
    float ��_xy = tex3D(tex_vort_y, coord.x + 0.5f, coord.y - 0.5f, coord.z);
    float ��_xz = tex3D(tex_vort_z, coord.x + 0.5f, coord.y, coord.z - 0.5f);

    float mag_x = sqrtf(��_xx * ��_xx + ��_xy * ��_xy + ��_xz * ��_xz + 0.00001f);
    float dx_x = ��_xx / mag_x;
    float dy_x = ��_xy / mag_x;
    float dz_x = ��_xz / mag_x;

    float v_x0 = tex3D(tex_velocity, coord.x + dx_x + 0.5f, coord.y + dy_x - 0.5f, coord.z + dz_x - 0.5f).x;
    float v_x1 = tex3D(tex_velocity, coord.x - dx_x + 0.5f, coord.y - dy_x - 0.5f, coord.z - dz_x - 0.5f).x;

    ushort result_x = __float2half_rn(scale * (v_x0 - v_x1) * mag_x + ��_xx);
    surf3Dwrite(result_x, surf_x, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    float ��_yx = tex3D(tex_vort_x, coord.x - 0.5f, coord.y + 0.5f, coord.z);
    float ��_yy = tex3D(tex_vort_y, coord.x, coord.y, coord.z);
    float ��_yz = tex3D(tex_vort_z, coord.x, coord.y + 0.5f, coord.z - 0.5f);

    float mag_y = sqrtf(��_yx * ��_yx + ��_yy * ��_yy + ��_yz * ��_yz + 0.00001f);
    float dx_y = ��_yx / mag_y;
    float dy_y = ��_yy / mag_y;
    float dz_y = ��_yz / mag_y;

    float v_y0 = tex3D(tex_velocity, coord.x + dx_y - 0.5f, coord.y + dy_y + 0.5f, coord.z + dz_y - 0.5f).y;
    float v_y1 = tex3D(tex_velocity, coord.x - dx_y - 0.5f, coord.y - dy_y + 0.5f, coord.z - dz_y - 0.5f).y;

    ushort result_y = __float2half_rn(scale * (v_y0 - v_y1) * mag_y + ��_yy);
    surf3Dwrite(result_y, surf_y, x * sizeof(ushort), y, z, hipBoundaryModeTrap);

    float ��_zx = tex3D(tex_vort_x, coord.x - 0.5f, coord.y, coord.z + 0.5f);
    float ��_zy = tex3D(tex_vort_y, coord.x, coord.y - 0.5f, coord.z + 0.5f);
    float ��_zz = tex3D(tex_vort_z, coord.x, coord.y, coord.z);

    float mag_z = sqrtf(��_zx * ��_zx + ��_zy * ��_zy + ��_zz * ��_zz + 0.00001f);
    float dx_z = ��_zx / mag_z;
    float dy_z = ��_zy / mag_z;
    float dz_z = ��_zz / mag_z;

    float v_z0 = tex3D(tex_velocity, coord.x + dx_z - 0.5f, coord.y + dy_z - 0.5f, coord.z + dz_z + 0.5f).z;
    float v_z1 = tex3D(tex_velocity, coord.x - dx_z - 0.5f, coord.y - dy_z - 0.5f, coord.z - dz_z + 0.5f).z;

    ushort result_z = __float2half_rn(scale * (v_z0 - v_z1) * mag_z + ��_zz);
    surf3Dwrite(result_z, surf_z, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

// =============================================================================

void LaunchApplyVorticityConfinementStaggered(hipArray* dest,
                                              hipArray* velocity,
                                              hipArray* conf_x,
                                              hipArray* conf_y,
                                              hipArray* conf_z,
                                              uint3 volume_size,
                                              BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&tex_velocity, velocity, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_conf_x = BindHelper::Bind(&tex_conf_x, conf_x, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_conf_x.error() != hipSuccess)
        return;

    auto bound_conf_y = BindHelper::Bind(&tex_conf_y, conf_y, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_conf_y.error() != hipSuccess)
        return;

    auto bound_conf_z = BindHelper::Bind(&tex_conf_z, conf_z, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_conf_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    ApplyVorticityConfinementStaggeredKernel<<<grid, block>>>();
}

void LaunchBuildVorticityConfinementStaggered(hipArray* dest_x,
                                              hipArray* dest_y,
                                              hipArray* dest_z,
                                              hipArray* curl_x,
                                              hipArray* curl_y,
                                              hipArray* curl_z,
                                              float coeff, float cell_size,
                                              uint3 volume_size,
                                              BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf_x, dest_x) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_y, dest_y) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_z, dest_z) != hipSuccess)
        return;

    auto bound_curl_x = BindHelper::Bind(&tex_vort_x, curl_x, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_x.error() != hipSuccess)
        return;

    auto bound_curl_y = BindHelper::Bind(&tex_vort_y, curl_y, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_y.error() != hipSuccess)
        return;

    auto bound_curl_z = BindHelper::Bind(&tex_vort_z, curl_z, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    BuildVorticityConfinementStaggeredKernel<<<grid, block>>>(coeff, cell_size,
                                                              0.5f / cell_size,
                                                              volume_size);
}


void LaunchComputeDivergenceStaggeredForVort(hipArray* dest,
                                             hipArray* velocity,
                                             float cell_size, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&tex_velocity, velocity, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceStaggeredKernelForVort<<<grid, block>>>(1.0f / cell_size,
                                                             volume_size);
}


void LaunchComputeCurlStaggered(hipArray* dest_x, hipArray* dest_y,
                                hipArray* dest_z, hipArray* velocity,
                                hipArray* curl_x, hipArray* curl_y,
                                hipArray* curl_z, float inverse_cell_size,
                                uint3 volume_size, BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf_x, dest_x) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_y, dest_y) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_z, dest_z) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&tex_velocity, velocity, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_curl_x = BindHelper::Bind(&tex_vort_x, curl_x, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_x.error() != hipSuccess)
        return;

    auto bound_curl_y = BindHelper::Bind(&tex_vort_y, curl_y, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_y.error() != hipSuccess)
        return;

    auto bound_curl_z = BindHelper::Bind(&tex_vort_z, curl_z, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_curl_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    ComputeCurlStaggeredKernel<<<grid, block>>>(volume_size, inverse_cell_size);
}
