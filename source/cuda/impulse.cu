#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"
#include "cuda_common.h"
#include "fluid_impulse.h"

surface<void, cudaSurfaceType3D> surf;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;

__global__ void ApplyImpulse1Kernel(float3 center_point, float3 hotspot,
                                    float radius, float value,
                                    uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        surf3Dwrite(__float2half_rn(scale * value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void HotFloorKernel(float3 center_point, float3 hotspot,
                               float radius, float value, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf, x * sizeof(ushort), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensityKernel(float3 center_point, float radius,
                                     float value, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
    }
}

__global__ void GenerateHeatSphereKernel(float3 center_point, float radius,
                                         float value, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = make_float3(coord.x, coord.y, coord.z) -
        make_float3(center_point.x, center_point.y, center_point.z);
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d < radius && d > radius * 0.5f) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensitySphereKernel(float3 center_point, float radius,
                                           float value, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = make_float3(coord.x, coord.y, coord.z) -
        make_float3(center_point.x, center_point.y, center_point.z);
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d < radius && d > radius * 0.5f) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void BuoyantJetKernel(float3 hotspot, float radius, float value,
                                 uint3 volume_size)
{
    int x = 1 + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.y, coord.z) - make_float2(hotspot.y, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf, x * sizeof(ushort), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

// =============================================================================

void LaunchImpulseScalar(hipArray* dest, hipArray* original,
                         float3 center_point, float3 hotspot, float radius,
                         float value, FluidImpulse impulse, uint3 volume_size,
                         BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    const int heat_layer_thickness = 3;
    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            dim3 block(volume_size.x, heat_layer_thickness, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.y = 1;
            HotFloorKernel<<<grid, block>>>(center_point, hotspot, radius,
                                            value, volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(radius + center_point.y) + 1;

            dim3 block;
            dim3 grid;
            ba->ArrangeRowScan(&block, &grid, actual_size);
            GenerateHeatSphereKernel<<<grid, block>>>(center_point, radius,
                                                      value, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            dim3 block(heat_layer_thickness, volume_size.y, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.x = 1;
            BuoyantJetKernel<<<grid, block>>>(center_point, radius, value,
                                              volume_size);
            break;
        }
    }
}

void LaunchImpulseDensity(hipArray* dest, hipArray* original,
                          float3 center_point, float radius, float value,
                          FluidImpulse impulse, uint3 volume_size,
                          BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            dim3 block(volume_size.x, 2, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.y = 1;
            ImpulseDensityKernel<<<grid, block>>>(center_point, radius, value,
                                                  volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(radius + center_point.y) + 1;

            dim3 block;
            dim3 grid;
            ba->ArrangeRowScan(&block, &grid, actual_size);
            grid.y = 1;
            ImpulseDensitySphereKernel<<<grid, block>>>(center_point, radius,
                                                        value, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            dim3 block(2, volume_size.y, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.x = 1;
            BuoyantJetKernel<<<grid, block>>>(center_point, radius, value,
                                              volume_size);
            break;
        }
    }
}