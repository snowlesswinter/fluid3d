#include "hip/hip_runtime.h"
//
// Fluid3d - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Fluid3d license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"
#include "cuda_common_host.h"
#include "cuda_common_kern.h"
#include "fluid_impulse.h"

surface<void, cudaSurfaceType3D> surf;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;

__global__ void ApplyImpulse1Kernel(float3 center_point, float3 hotspot,
                                    float radius, float value,
                                    uint3 volume_size)
{
    int x = VolumeX();
    int y = 1 + threadIdx.y;
    int z = VolumeZ();

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        surf3Dwrite(__float2half_rn(scale * value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void HotFloorKernel(float3 center_point, float3 hotspot,
                               float radius, float value, uint3 volume_size)
{
    int x = VolumeX();
    int y = 1 + threadIdx.y;
    int z = VolumeZ();

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf, x * sizeof(ushort), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensityKernel(float3 center_point, float radius,
                                     float value, uint3 volume_size)
{
    int x = VolumeX();
    int y = 1 + threadIdx.y;
    int z = VolumeZ();

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
    }
}

__global__ void GenerateHeatSphereKernel(float3 center_point, float radius,
                                         float value, uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = make_float3(coord.x, coord.y, coord.z) -
        make_float3(center_point.x, center_point.y, center_point.z);
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d < radius && d > radius * 0.5f) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensitySphereKernel(float3 center_point, float radius,
                                           float value, uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 diff = make_float3(coord.x, coord.y, coord.z) -
        make_float3(center_point.x, center_point.y, center_point.z);
    float d = norm3df(diff.x, diff.y, diff.z);
    if (d < radius && d > radius * 0.5f) {
        surf3Dwrite(__float2half_rn(value), surf,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void BuoyantJetKernel(float3 hotspot, float radius, float value,
                                 uint3 volume_size)
{
    int x = 1 + threadIdx.x;
    int y = VolumeY();
    int z = VolumeZ();

    if (y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.y, coord.z) - make_float2(hotspot.y, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), surf, x * sizeof(ushort), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

// =============================================================================

void LaunchImpulseScalar(hipArray* dest, hipArray* original,
                         float3 center_point, float3 hotspot, float radius,
                         float value, FluidImpulse impulse, uint3 volume_size,
                         BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    const int kHeatLayerThickness = 8;
    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            dim3 block(volume_size.x, kHeatLayerThickness, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.y = 1;
            HotFloorKernel<<<grid, block>>>(center_point, hotspot, radius,
                                            value, volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(radius + center_point.y) + 1;

            dim3 block;
            dim3 grid;
            ba->ArrangeRowScan(&block, &grid, actual_size);
            GenerateHeatSphereKernel<<<grid, block>>>(center_point, radius,
                                                      value, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            dim3 block(kHeatLayerThickness, volume_size.y, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.x = 1;
            BuoyantJetKernel<<<grid, block>>>(center_point, radius, value,
                                              volume_size);
            break;
        }
    }
}

void LaunchImpulseDensity(hipArray* dest, hipArray* original,
                          float3 center_point, float radius, float value,
                          FluidImpulse impulse, uint3 volume_size,
                          BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    switch (impulse) {
        case IMPULSE_HOT_FLOOR: {
            dim3 block(volume_size.x, 8, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.y = 1;
            ImpulseDensityKernel<<<grid, block>>>(center_point, radius, value,
                                                  volume_size);
            break;
        }
        case IMPULSE_SPHERE: {
            uint3 actual_size = volume_size;
            actual_size.y = static_cast<uint>(radius + center_point.y) + 1;

            dim3 block;
            dim3 grid;
            ba->ArrangeRowScan(&block, &grid, actual_size);
            grid.y = 1;
            ImpulseDensitySphereKernel<<<grid, block>>>(center_point, radius,
                                                        value, volume_size);
            break;
        }
        case IMPULSE_BUOYANT_JET: {
            dim3 block(2, volume_size.y, 1);
            dim3 grid;
            ba->ArrangeGrid(&grid, block, volume_size);
            grid.x = 1;
            BuoyantJetKernel<<<grid, block>>>(center_point, radius, value,
                                              volume_size);
            break;
        }
    }
}