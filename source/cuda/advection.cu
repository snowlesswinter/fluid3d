#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "advection_method.h"
#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> surf;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vx;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vy;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vz;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_aux;

__device__ float TrilinearInterpolationSingle(float x0y0z0, float x1y0z0,
                                              float x0y1z0, float x0y0z1,
                                              float x1y1z0, float x0y1z1,
                                              float x1y0z1, float x1y1z1,
                                              float ��, float ��, float ��)
{
    float xy0z0 = (1 - ��) * x0y0z0 + �� * x1y0z0;
    float xy1z0 = (1 - ��) * x0y1z0 + �� * x1y1z0;
    float xy0z1 = (1 - ��) * x0y0z1 + �� * x1y0z1;
    float xy1z1 = (1 - ��) * x0y1z1 + �� * x1y1z1;

    float yz0 = (1 - ��) * xy0z0 + �� * xy1z0;
    float yz1 = (1 - ��) * xy0z1 + �� * xy1z1;

    return (1 - ��) * yz0 + �� * yz1;
}

__device__ float3 TrilinearInterpolation(float3* cache, float3 coord,
                                         int slice_stride, int row_stride)
{
    float int_x = floorf(coord.x);
    float int_y = floorf(coord.y);
    float int_z = floorf(coord.z);

    float �� = fracf(coord.x);
    float �� = fracf(coord.y);
    float �� = fracf(coord.z);

    int index = int_z * slice_stride + int_y * row_stride + int_x;
    float3 x0y0z0 = cache[index];
    float3 x1y0z0 = cache[index + 1];
    float3 x0y1z0 = cache[index + row_stride];
    float3 x0y0z1 = cache[index + slice_stride];
    float3 x1y1z0 = cache[index + row_stride + 1];
    float3 x0y1z1 = cache[index + slice_stride + row_stride];
    float3 x1y0z1 = cache[index + slice_stride + 1];
    float3 x1y1z1 = cache[index + slice_stride + row_stride + 1];

    float x = TrilinearInterpolationSingle(x0y0z0.x, x1y0z0.x, x0y1z0.x, x0y0z1.x, x1y1z0.x, x0y1z1.x, x1y0z1.x, x1y1z1.x, ��, ��, ��);
    float y = TrilinearInterpolationSingle(x0y0z0.y, x1y0z0.y, x0y1z0.y, x0y0z1.y, x1y1z0.y, x0y1z1.y, x1y0z1.y, x1y1z1.y, ��, ��, ��);
    float z = TrilinearInterpolationSingle(x0y0z0.z, x1y0z0.z, x0y1z0.z, x0y0z1.z, x1y1z0.z, x0y1z1.z, x1y0z1.z, x1y1z1.z, ��, ��, ��);
    return make_float3(x, y, z);
}

// Only ~45% hit rate, serious block effect, deprecated.
__global__ void AdvectVelocityKernel_smem(float time_step, float dissipation)
{
    __shared__ float3 cached_block[600];

    int base_x = blockIdx.x * blockDim.x;
    int base_y = blockIdx.y * blockDim.y;
    int base_z = blockIdx.z * blockDim.z;

    int x = base_x + threadIdx.x;
    int y = base_y + threadIdx.y;
    int z = base_z + threadIdx.z;

    int bw = blockDim.x;
    int bh = blockDim.y;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;

    int index = threadIdx.z * bw * bh + threadIdx.y * bw + threadIdx.x;
    cached_block[index] =
        make_float3(tex3D(tex, coord.x, coord.y, coord.z));
    float3 velocity = cached_block[index];
    __syncthreads();

    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity;
    if (back_traced.x >= base_x + 0.5f && back_traced.x < base_x + blockDim.x + 0.5f &&
            back_traced.y >= base_y + 0.5f && back_traced.y < base_y + blockDim.y + 0.5f &&
            back_traced.z >= base_z + 0.5f && back_traced.z < base_z + blockDim.z + 0.5f) {

        new_velocity = TrilinearInterpolation(
            cached_block, back_traced - make_float3(base_x + 0.5f, base_y + 0.5f, base_z + 0.5f),
            bw * bh, bw);
    } else {
        new_velocity =
            make_float3( 
                tex3D(tex, back_traced.x, back_traced.y,
                      back_traced.z));
    }
    new_velocity *= 1.0f - dissipation * time_step;
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, surf, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

__device__ float3 GetVelocity(float3 pos)
{
    float v_x = tex3D(tex_vx, pos.x, pos.y, pos.z);
    float v_y = tex3D(tex_vy, pos.x, pos.y, pos.z);
    float v_z = tex3D(tex_vz, pos.x, pos.y, pos.z);
    return make_float3(v_x, v_y, v_z);
}

template <bool MidPoint>
__device__ inline float3 AdvectImpl(float3 vel, float3 pos, float time_step_over_cell_size)
{
    return pos - vel * time_step_over_cell_size;
}

template <>
__device__ inline float3 AdvectImpl<true>(float3 vel, float3 pos, float time_step_over_cell_size)
{
    float3 mid_point = pos - vel * 0.5f * time_step_over_cell_size;
    vel = GetVelocity(mid_point);
    return pos - vel * time_step_over_cell_size;
}

template <bool MidPoint>
__global__ void AdvectFieldBfeccKernel(float time_step_over_cell_size, float dissipation, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocity(coord);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step_over_cell_size);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_new = tex3D(tex_aux, back_traced.x, back_traced.y, back_traced.z);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);

    auto r = __float2half_rn(dissipation * ��_new);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldMacCormackKernel(float time_step_over_cell_size, float dissipation, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocity(coord);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step_over_cell_size);

    float ��_n = tex3D(tex, coord.x, coord.y, coord.z);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_np1_hat = tex3D(tex_aux, coord.x, coord.y, coord.z);

    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord, -time_step_over_cell_size);
    float ��_n_hat = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);

    float ��_new = ��_np1_hat + 0.5f * (��_n - ��_n_hat);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new)
        ��_new = ��_np1_hat;

    auto r = __float2half_rn(��_new * dissipation);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldSemiLagrangianKernel(float time_step_over_cell_size,
                                                float dissipation,
                                                uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocity(coord);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord,
                                              time_step_over_cell_size);

    float �� = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);
    auto r = __float2half_rn(�� * dissipation);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void BfeccRemoveErrorKernel(float time_step_over_cell_size,
                                       uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocity(coord);
    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord,
                                                -time_step_over_cell_size);

    float �� = tex3D(tex, coord.x, coord.y, coord.z);
    float r = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);
    r = 0.5f * (3.0f * �� - r);
    surf3Dwrite(__float2half_rn(r), surf, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void AdvectFieldsBfecc(hipArray** fnp1, hipArray** fn, int num_of_fields,
                       hipArray* vel_x, hipArray* vel_y, hipArray* vel_z,
                       hipArray* aux, float cell_size, float time_step,
                       float dissipation, uint3 volume_size, bool mid_point,
                       BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        // Pass 1: Calculate ��_n_plus_1_hat, and store in |fnp1[i]|.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianKernel<true><<<grid, block>>>(
                time_step / cell_size, 1.0f, volume_size);
        else
            AdvectFieldSemiLagrangianKernel<false><<<grid, block>>>(
                time_step / cell_size, 1.0f, volume_size);

        // Pass 2: Calculate ��_n_hat, and store in |aux|.
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        {
            auto bound_a = BindHelper::Bind(&tex_aux, fnp1[i], false,
                                            hipFilterModeLinear,
                                            hipAddressModeClamp);
            if (bound_a.error() != hipSuccess)
                return;

            if (mid_point)
                BfeccRemoveErrorKernel<true><<<grid, block>>>(
                    time_step / cell_size, volume_size);
            else
                BfeccRemoveErrorKernel<false><<<grid, block>>>(
                    time_step / cell_size, volume_size);
        }

        // Pass 3: Calculate the final result.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound_a = BindHelper::Bind(&tex_aux, aux, false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp);
        if (bound_a.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldBfeccKernel<true><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
        else
            AdvectFieldBfeccKernel<false><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
    }
}

void AdvectFieldsMacCormack(hipArray** fnp1, hipArray** fn,
                            int num_of_fields, hipArray* vel_x,
                            hipArray* vel_y, hipArray* vel_z, hipArray* aux,
                            float cell_size, float time_step, float dissipation,
                            uint3 volume_size, bool mid_point,
                            BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    auto bound_a = BindHelper::Bind(&tex_aux, aux, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_a.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianKernel<true><<<grid, block>>>(
                time_step / cell_size, 1.0f, volume_size);
        else
            AdvectFieldSemiLagrangianKernel<false><<<grid, block>>>(
                time_step / cell_size, 1.0f, volume_size);

        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldMacCormackKernel<true><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
        else
            AdvectFieldMacCormackKernel<false><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
    }
}

void AdvectFieldsSemiLagrangian(hipArray** fnp1, hipArray** fn,
                                int num_of_fields, hipArray* vel_x,
                                hipArray* vel_y, hipArray* vel_z,
                                float cell_size, float time_step,
                                float dissipation, uint3 volume_size,
                                bool mid_point, BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianKernel<true><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
        else
            AdvectFieldSemiLagrangianKernel<false><<<grid, block>>>(
                time_step / cell_size, 1.0f - dissipation * time_step,
                volume_size);
    }
}

void LaunchAdvectScalarField(hipArray* fnp1, hipArray* fn,
                             hipArray* vel_x, hipArray* vel_y,
                             hipArray* vel_z, hipArray* aux, float cell_size,
                             float time_step, float dissipation,
                             AdvectionMethod method,
                             uint3 volume_size, bool mid_point,
                             BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1};
    hipArray* fns[] = {fn};
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormack(fnp1s, fns, num_of_fields, vel_x, vel_y, vel_z,
                               aux, cell_size, time_step, dissipation,
                               volume_size, mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfecc(fnp1s, fns, num_of_fields, vel_x, vel_y, vel_z, aux,
                          cell_size, time_step, dissipation, volume_size,
                          mid_point, ba);
    } else {
        AdvectFieldsSemiLagrangian(fnp1s, fns, num_of_fields, vel_x, vel_y,
                                   vel_z, cell_size, time_step, dissipation,
                                   volume_size, mid_point, ba);
    }
}

void LaunchAdvectVectorField(hipArray* fnp1_x, hipArray* fnp1_y,
                             hipArray* fnp1_z, hipArray* fn_x,
                             hipArray* fn_y, hipArray* fn_z,
                             hipArray* vel_x, hipArray* vel_y,
                             hipArray* vel_z, hipArray* aux, float cell_size,
                             float time_step, float dissipation,
                             AdvectionMethod method,
                             uint3 volume_size, bool mid_point,
                             BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1_x, fnp1_y, fnp1_z};
    hipArray* fns[] = {fn_x, fn_y, fn_z};
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormack(fnp1s, fns, num_of_fields, vel_x, vel_y, vel_z,
                               aux, cell_size, time_step, dissipation,
                               volume_size, mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfecc(fnp1s, fns, num_of_fields, vel_x, vel_y, vel_z, aux,
                          cell_size, time_step, dissipation, volume_size,
                          mid_point, ba);
    } else {
        AdvectFieldsSemiLagrangian(fnp1s, fns, num_of_fields, vel_x, vel_y,
                                   vel_z, cell_size, time_step, dissipation,
                                   volume_size, mid_point, ba);
    }
}
