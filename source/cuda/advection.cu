#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "advection_method.h"
#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> advect_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_velocity;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_intermediate;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_intermediate1;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_source;

__global__ void AdvectScalarBfeccKernel(float time_step, float dissipation,
                                        bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float ��0 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_new = tex3D(advect_intermediate1, back_traced.x, back_traced.y, back_traced.z);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = tex3D(advect_source, back_traced.x, back_traced.y, back_traced.z);

    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��_new)) * ��_new :
        (1.0f - dissipation * time_step) * ��_new;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectScalarBfeccRemoveErrorKernel(float time_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced = coord - time_step * make_float3(velocity);

    float �� = tex3D(advect_source, coord.x, coord.y, coord.z);
    float result = tex3D(advect_intermediate1, back_traced.x, back_traced.y,
                         back_traced.z);
    result = 0.5f * (3.0f * �� - result);
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectScalarMacCormackKernel(float time_step, float dissipation,
                                             bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;
    float �� = tex3D(advect_source, coord.x, coord.y, coord.z);

    float ��0 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_n_plus_1_hat = tex3D(advect_intermediate1, coord.x, coord.y, coord.z);
    float3 forward_traced = coord + time_step * velocity;
    float ��_n_hat = tex3D(advect_intermediate1, forward_traced.x, forward_traced.y, forward_traced.z);

    float ��_new = (��_n_plus_1_hat + 0.5f * (�� - ��_n_hat));
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = ��_n_plus_1_hat;

    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��_new)) * ��_new :
        (1.0f - dissipation * time_step) * ��_new;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectScalarSemiLagrangianKernel(float time_step,
                                                 float dissipation,
                                                 bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced = coord - time_step * make_float3(velocity);

    float �� = tex3D(advect_source, back_traced.x, back_traced.y, back_traced.z);
    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��)) * �� :
        (1.0f - dissipation * time_step) * ��;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectVelocityBfeccKernel(float time_step, float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 v_n = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * v_n;

    float3 v0 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v1 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v2 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v3 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));
    float3 v4 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v5 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v6 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v7 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));

    float3 v_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0, v1), v2), v3), v4), v5), v6), v7);
    float3 v_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0, v1), v2), v3), v4), v5), v6), v7);

    float3 v_new = make_float3(tex3D(advect_intermediate, back_traced.x, back_traced.y, back_traced.z));
    float3 clamped = fmaxf(fminf(v_new, v_max), v_min);
    if (clamped.x != v_new.x || clamped.y != v_new.y || clamped.z != v_new.z)
        v_new = make_float3(tex3D(advect_velocity, back_traced.x, back_traced.y, back_traced.z));

    v_new = (1.0f - dissipation * time_step) * v_new;
    ushort4 result = make_ushort4(__float2half_rn(v_new.x), __float2half_rn(v_new.y), __float2half_rn(v_new.z), 0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectVelocityBfeccRemoveErrorKernel(float time_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity =
        make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity =
        make_float3(
            tex3D(advect_intermediate, back_traced.x, back_traced.y,
                  back_traced.z));
    new_velocity = 0.5f * (3.0f * velocity - new_velocity);
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectVelocityMacCormackKernel(float time_step,
                                               float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 v_n = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * v_n;

    float3 v0 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v1 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v2 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v3 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));
    float3 v4 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v5 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v6 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v7 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));

    float3 v_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0, v1), v2), v3), v4), v5), v6), v7);
    float3 v_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0, v1), v2), v3), v4), v5), v6), v7);

    float3 v_n_plus_1_hat = make_float3(tex3D(advect_intermediate, coord.x, coord.y, coord.z));
    float3 forward_trace = coord + time_step * v_n;
    float3 v_n_hat = make_float3(tex3D(advect_intermediate, forward_trace.x, forward_trace.y, forward_trace.z));

    float3 v_new = (v_n_plus_1_hat + 0.5f * (v_n - v_n_hat));
    float3 clamped = fmaxf(fminf(v_new, v_max), v_min);
    if (clamped.x != v_new.x || clamped.y != v_new.y || clamped.z != v_new.z)
        v_new = v_n_plus_1_hat;

    v_new = (1.0f - dissipation * time_step) * v_new;
    ushort4 result = make_ushort4(__float2half_rn(v_new.x), __float2half_rn(v_new.y), __float2half_rn(v_new.z), 0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectVelocitySemiLagrangianKernel(float time_step,
                                                   float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity =
        make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity =
        (1.0f - dissipation * time_step) * 
            make_float3(
                tex3D(advect_velocity, back_traced.x, back_traced.y,
                      back_traced.z));
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__device__ float TrilinearInterpolationSingle(float x0y0z0, float x1y0z0,
                                              float x0y1z0, float x0y0z1,
                                              float x1y1z0, float x0y1z1,
                                              float x1y0z1, float x1y1z1,
                                              float ��, float ��, float ��)
{
    float xy0z0 = (1 - ��) * x0y0z0 + �� * x1y0z0;
    float xy1z0 = (1 - ��) * x0y1z0 + �� * x1y1z0;
    float xy0z1 = (1 - ��) * x0y0z1 + �� * x1y0z1;
    float xy1z1 = (1 - ��) * x0y1z1 + �� * x1y1z1;

    float yz0 = (1 - ��) * xy0z0 + �� * xy1z0;
    float yz1 = (1 - ��) * xy0z1 + �� * xy1z1;

    return (1 - ��) * yz0 + �� * yz1;
}

__device__ float3 TrilinearInterpolation(float3* cache, float3 coord,
                                         int slice_stride, int row_stride)
{
    float int_x = floorf(coord.x);
    float int_y = floorf(coord.y);
    float int_z = floorf(coord.z);

    float �� = fracf(coord.x);
    float �� = fracf(coord.y);
    float �� = fracf(coord.z);

    int index = int_z * slice_stride + int_y * row_stride + int_x;
    float3 x0y0z0 = cache[index];
    float3 x1y0z0 = cache[index + 1];
    float3 x0y1z0 = cache[index + row_stride];
    float3 x0y0z1 = cache[index + slice_stride];
    float3 x1y1z0 = cache[index + row_stride + 1];
    float3 x0y1z1 = cache[index + slice_stride + row_stride];
    float3 x1y0z1 = cache[index + slice_stride + 1];
    float3 x1y1z1 = cache[index + slice_stride + row_stride + 1];

    float x = TrilinearInterpolationSingle(x0y0z0.x, x1y0z0.x, x0y1z0.x, x0y0z1.x, x1y1z0.x, x0y1z1.x, x1y0z1.x, x1y1z1.x, ��, ��, ��);
    float y = TrilinearInterpolationSingle(x0y0z0.y, x1y0z0.y, x0y1z0.y, x0y0z1.y, x1y1z0.y, x0y1z1.y, x1y0z1.y, x1y1z1.y, ��, ��, ��);
    float z = TrilinearInterpolationSingle(x0y0z0.z, x1y0z0.z, x0y1z0.z, x0y0z1.z, x1y1z0.z, x0y1z1.z, x1y0z1.z, x1y1z1.z, ��, ��, ��);
    return make_float3(x, y, z);
}

// Only ~45% hit rate, serious block effect, deprecated.
__global__ void AdvectVelocityKernel_smem(float time_step, float dissipation)
{
    __shared__ float3 cached_block[600];

    int base_x = blockIdx.x * blockDim.x;
    int base_y = blockIdx.y * blockDim.y;
    int base_z = blockIdx.z * blockDim.z;

    int x = base_x + threadIdx.x;
    int y = base_y + threadIdx.y;
    int z = base_z + threadIdx.z;

    int bw = blockDim.x;
    int bh = blockDim.y;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;

    int index = threadIdx.z * bw * bh + threadIdx.y * bw + threadIdx.x;
    cached_block[index] =
        make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 velocity = cached_block[index];
    __syncthreads();

    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity;
    if (back_traced.x >= base_x + 0.5f && back_traced.x < base_x + blockDim.x + 0.5f &&
            back_traced.y >= base_y + 0.5f && back_traced.y < base_y + blockDim.y + 0.5f &&
            back_traced.z >= base_z + 0.5f && back_traced.z < base_z + blockDim.z + 0.5f) {

        new_velocity = TrilinearInterpolation(
            cached_block, back_traced - make_float3(base_x + 0.5f, base_y + 0.5f, base_z + 0.5f),
            bw * bh, bw);
    } else {
        new_velocity =
            make_float3( 
                tex3D(advect_velocity, back_traced.x, back_traced.y,
                      back_traced.z));
    }
    new_velocity *= 1.0f - dissipation * time_step;
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchAdvectScalarBfecc(hipArray_t dest_array, hipArray_t velocity_array,
                             hipArray_t source_array,
                             hipArray_t intermediate_array, float time_step,
                             float dissipation, bool quadratic_dissipation,
                             uint3 volume_size)
{
    // Pass 1: Calculate ��_n_plus_1_hat, and store in |dest_array|.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array,
                                      false, hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array,
                                         false, hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianKernel<<<grid, block>>>(time_step, 0.0f,
                                                      quadratic_dissipation);

    // Pass 2: Calculate ��_n_hat, and store in |intermediate_array|.
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_intermediate1 = BindHelper::Bind(&advect_intermediate1,
                                                dest_array, false,
                                                hipFilterModeLinear,
                                                hipAddressModeClamp);
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarBfeccRemoveErrorKernel<<<grid, block>>>(-time_step);

    // Pass 3: Calculate the final result.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    bound_intermediate1.Take(
        BindHelper::Bind(&advect_intermediate1, intermediate_array, false,
                         hipFilterModeLinear, hipAddressModeClamp));
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarBfeccKernel<<<grid, block>>>(time_step, dissipation,
                                             quadratic_dissipation);
}

void LaunchAdvectScalarMacCormack(hipArray_t dest_array,
                                  hipArray_t velocity_array,
                                  hipArray_t source_array,
                                  hipArray_t intermediate_array,
                                  float time_step, float dissipation,
                                  bool quadratic_dissipation, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianKernel<<<grid, block>>>(time_step, 0.0f,
                                                      quadratic_dissipation);

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_intermediate1 = BindHelper::Bind(&advect_intermediate1,
                                                intermediate_array, false,
                                                hipFilterModeLinear,
                                                hipAddressModeClamp);
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarMacCormackKernel<<<grid, block>>>(time_step, dissipation,
                                                  quadratic_dissipation);
}

void LaunchAdvectScalar(hipArray_t dest_array, hipArray_t velocity_array,
                        hipArray_t source_array,
                        hipArray_t intermediate_array, float time_step,
                        float dissipation, bool quadratic_dissipation,
                        uint3 volume_size, AdvectionMethod method)
{
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        LaunchAdvectScalarMacCormack(dest_array, velocity_array, source_array,
                                     intermediate_array, time_step, dissipation,
                                     false, volume_size);
        return;
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        LaunchAdvectScalarBfecc(dest_array, velocity_array, source_array,
                                intermediate_array, time_step, dissipation,
                                false, volume_size);
        return;
    }

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array, false,
                                         hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianKernel<<<grid, block>>>(time_step, dissipation,
                                                      quadratic_dissipation);
}

void LaunchAdvectVelocityBfecc(hipArray_t dest_array,
                               hipArray_t velocity_array,
                               hipArray_t intermediate_array, float time_step,
                               float time_step_prev, float dissipation,
                               uint3 volume_size)
{
    // Pass 1: Calculate ��_n_plus_1_hat, and store in |dest_array|.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianKernel<<<grid, block>>>(time_step, 0.0f);

    // Pass 2: Calculate ��_n_hat, and store in |intermediate_array|.
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_intermediate = BindHelper::Bind(&advect_intermediate, dest_array,
                                               false, hipFilterModeLinear,
                                               hipAddressModeClamp);
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityBfeccRemoveErrorKernel<<<grid, block>>>(-time_step);

    // Pass 3: Calculate the final result.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    bound_intermediate.Take(
        BindHelper::Bind(&advect_intermediate, intermediate_array, false,
                         hipFilterModeLinear, hipAddressModeClamp));
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityBfeccKernel<<<grid, block>>>(time_step, dissipation);
}

void LaunchAdvectVelocityMacCormack(hipArray_t dest_array,
                                    hipArray_t velocity_array,
                                    hipArray_t intermediate_array,
                                    float time_step, float time_step_prev,
                                    float dissipation, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianKernel<<<grid, block>>>(time_step, 0.0f);

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_intermediate = BindHelper::Bind(&advect_intermediate,
                                               intermediate_array, false,
                                               hipFilterModeLinear,
                                               hipAddressModeClamp);
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityMacCormackKernel<<<grid, block>>>(time_step, dissipation);
}

void LaunchAdvectVelocity(hipArray_t dest_array, hipArray_t velocity_array,
                          hipArray_t intermediate_array, float time_step,
                          float time_step_prev, float dissipation,
                          uint3 volume_size, AdvectionMethod method)
{
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        LaunchAdvectVelocityMacCormack(dest_array, velocity_array,
                                       intermediate_array, time_step,
                                       time_step_prev, dissipation,
                                       volume_size);
        return;
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        LaunchAdvectVelocityBfecc(dest_array, velocity_array,
                                  intermediate_array, time_step, time_step_prev,
                                  dissipation, volume_size);
        return;
    }

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianKernel<<<grid, block>>>(time_step,
                                                        dissipation);
}
