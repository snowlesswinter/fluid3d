#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

surface<void, hipTextureType3D> advect_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_source;
surface<void, hipTextureType3D> buoyancy_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_temperature;
surface<void, hipTextureType3D> impulse_dest;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> impulse_original;
surface<void, hipTextureType3D> divergence_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> divergence_velocity;
surface<void, hipTextureType3D> gradient_velocity;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> gradient_packed;
surface<void, hipTextureType3D> jacobi;
texture<ushort2, hipTextureType3D, hipReadModeElementType> jacobi_raw;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> jacobi_packed;
surface<void, hipTextureType3D> diagnosis;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> diagnosis_source;

__global__ void AdvectPureKernel(float time_step, float dissipation,
                                 int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced =
        coord - time_step * make_float3(velocity.x, velocity.y, velocity.z);

    float result = dissipation * tex3D(advect_source, back_traced.x,
                                       back_traced.y, back_traced.z);
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectVelocityPureKernel(float time_step, float dissipation,
                                         int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced =
        coord - time_step * make_float3(velocity.x, velocity.y, velocity.z);

    float4 new_velocity = dissipation * tex3D(advect_velocity, back_traced.x,
                                              back_traced.y, back_traced.z);
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ApplyBuoyancyPureKernel(float time_step,
                                        float ambient_temperature,
                                        float accel_factor, float gravity,
                                        int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float4 velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);

    ushort4 result = make_ushort4(__float2half_rn(velocity.x),
                                  __float2half_rn(velocity.y),
                                  __float2half_rn(velocity.z),
                                  0);
    if (t > ambient_temperature) {
        float accel = time_step * ((t - ambient_temperature) * accel_factor -
                                   gravity);
        result.y = __float2half_rn(velocity.y + accel);
    }
    surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ApplyImpulsePureKernel(float3 center_point, float3 hotspot,
                                       float radius, float value,
                                       int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float original = tex3D(impulse_original, coord.x, coord.y, coord.z);

    if (coord.x > 1.0f && coord.y < 3.0f) {
        float2 diff = make_float2(coord.x, coord.z) -
            make_float2(center_point.x, center_point.z);
        float d = hypotf(diff.x, diff.y);
        if (d < radius) {
            diff = make_float2(coord.x, coord.z) -
                make_float2(hotspot.x, hotspot.z);
            float scale = (radius - hypotf(diff.x, diff.y)) / radius;
            scale = max(scale, 0.5f);
            surf3Dwrite(__float2half_rn(scale * value), impulse_dest,
                        x * sizeof(ushort), y, z, hipBoundaryModeTrap);
            return;
        }
    }

    surf3Dwrite(__float2half_rn(original), impulse_dest, x * sizeof(ushort), y,
                z, hipBoundaryModeTrap);
}

__global__ void ComputeDivergencePureKernel(float half_inverse_cell_size,
                                            int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float4 near =   tex3D(divergence_velocity, coord.x, coord.y, coord.z - 1.0f);
    float4 south =  tex3D(divergence_velocity, coord.x, coord.y - 1.0f, coord.z);
    float4 west =   tex3D(divergence_velocity, coord.x - 1.0f, coord.y, coord.z);
    float4 center = tex3D(divergence_velocity, coord.x, coord.y, coord.z);
    float4 east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y, coord.z);
    float4 north =  tex3D(divergence_velocity, coord.x, coord.y + 1.0f, coord.z);
    float4 far =    tex3D(divergence_velocity, coord.x, coord.y, coord.z + 1.0f);

    float diff_ew = east.x - west.x;
    float diff_ns = north.y - south.y;
    float diff_fn = far.z - near.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -center.x - west.x;

    if (x <= 0)
        diff_ew = east.x + center.x;

    if (y >= volume_size.y - 1)
        diff_ns = -center.y - south.y;

    if (y <= 0)
        diff_ns = north.y + center.y;

    if (z >= volume_size.z - 1)
        diff_fn = -center.z - near.z;

    if (z <= 0)
        diff_fn = far.z + center.z;

    float div = half_inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    ushort2 result = make_ushort2(0, __float2half_rn(div));
    surf3Dwrite(result, divergence_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ComputeResidualPackedDiagnosisKernel(float inverse_h_square,
                                                     int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float near =    tex3D(diagnosis_source, coord.x, coord.y, coord.z - 1.0f).x;
    float south =   tex3D(diagnosis_source, coord.x, coord.y - 1.0f, coord.z).x;
    float west =    tex3D(diagnosis_source, coord.x - 1.0f, coord.y, coord.z).x;
    float4 center = tex3D(diagnosis_source, coord.x, coord.y, coord.z);
    float east =    tex3D(diagnosis_source, coord.x + 1.0f, coord.y, coord.z).x;
    float north =   tex3D(diagnosis_source, coord.x, coord.y + 1.0f, coord.z).x;
    float far =     tex3D(diagnosis_source, coord.x, coord.y, coord.z + 1.0f).x;
    float b_center = center.y;

    if (coord.y == volume_size.y - 1)
        north = center.x;

    if (coord.y == 0)
        south = center.x;

    if (coord.x == volume_size.x - 1)
        east = center.x;

    if (coord.x == 0)
        west = center.x;

    if (coord.z == volume_size.z - 1)
        far = center.x;

    if (coord.z == 0)
        near = center.x;

    float v = b_center -
        (north + south + east + west + far + near - 6.0 * center.x) *
        inverse_h_square;
    surf3Dwrite(fabsf(v), diagnosis, x * sizeof(float), y, z,
                hipBoundaryModeTrap);
}

__device__ float2 xys(ushort4 v)
{
    return make_float2(__half2float(v.x), __half2float(v.y));
}

__device__ ushort2 xyt(ushort4 v)
{
    return make_ushort2(v.x, v.y);
}

__device__ ushort2 xyu(float2 v)
{
    return make_ushort2(__float2half_rn(v.x), __float2half_rn(v.y));
}

__device__ float2 xyi(ushort2 v)
{
    return make_float2(__half2float(v.x), __half2float(v.y));
}


__global__ void DampedJacobiPureKernel(float minus_square_cell_size,
                                       float omega_over_beta, int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float near =              tex3D(jacobi_packed, x, y, z - 1.0f).x;
    float south =             tex3D(jacobi_packed, x, y - 1.0f, z).x;
    float west =              tex3D(jacobi_packed, x - 1.0f, y, z).x;
    float2 packed_center =    tex3D(jacobi_packed, x, y, z);
    float east =              tex3D(jacobi_packed, x + 1.0f, y, z).x;
    float north =             tex3D(jacobi_packed, x, y + 1.0f, z).x;
    float far =               tex3D(jacobi_packed, x, y, z + 1.0f).x;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        east = packed_center.x;

    if (x <= 0)
        west = packed_center.x;

    if (y >= volume_size.y - 1)
        north = packed_center.x;

    if (y <= 0)
        south = packed_center.x;

    if (z >= volume_size.z - 1)
        far = packed_center.x;

    if (z <= 0)
        near = packed_center.x;

    float u = omega_over_beta * 3.0f * packed_center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        packed_center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u),
                               __float2half_rn(packed_center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiPureKernel_smem_full(float minus_square_cell_size,
                                        float omega_over_beta, int3 volume_size)
{
    __shared__ ushort2 cached_block[1000];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = (threadIdx.z + 1) * (blockDim.x + 2) * (blockDim.y + 2) + (threadIdx.y + 1) * (blockDim.x + 2) +
        threadIdx.x + 1;
    cached_block[index] = xyu(tex3D(jacobi_packed, x, y, z));

    if (threadIdx.x == 0)
        cached_block[index - 1] = x == 0 ? cached_block[index] : xyu(tex3D(jacobi_packed, x - 1, y, z));
    
    if (threadIdx.x == blockDim.x - 1)
        cached_block[index + 1] = x == volume_size.x - 1 ? cached_block[index] : xyu(tex3D(jacobi_packed, x + 1, y, z));
    
    if (threadIdx.y == 0)
        cached_block[index - (blockDim.x + 2)] = y == 0 ? cached_block[index] : xyu(tex3D(jacobi_packed, x, y - 1, z));
    
    if (threadIdx.y == blockDim.y - 1)
        cached_block[index + (blockDim.x + 2)] = y == volume_size.y - 1 ? cached_block[index] : xyu(tex3D(jacobi_packed, x, y + 1, z));
    
    if (threadIdx.z == 0)
        cached_block[index - (blockDim.x + 2) * (blockDim.y + 2)] = z == 0 ? cached_block[index] : xyu(tex3D(jacobi_packed, x, y, z - 1));
    
    if (threadIdx.z == blockDim.z - 1)
        cached_block[index + (blockDim.x + 2) * (blockDim.y + 2)] = z == volume_size.z - 1 ? cached_block[index] : xyu(tex3D(jacobi_packed, x, y, z + 1));

    __syncthreads();

    float  near =   __half2float(cached_block[index - (blockDim.x + 2) * (blockDim.y + 2) ].x);
    float  south =  __half2float(cached_block[index - (blockDim.x + 2)].x);
    float  west =   __half2float(cached_block[index - 1].x);
    float2 center = xyi(cached_block[index]);
    float  east =   __half2float(cached_block[index + 1].x);
    float  north =  __half2float(cached_block[index + (blockDim.x + 2)].x);
    float  far =    __half2float(cached_block[index + (blockDim.x + 2) * (blockDim.y + 2) ].x);

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiPureKernel_smem_full_float(float minus_square_cell_size,
                                                 float omega_over_beta, int3 volume_size)
{
    __shared__ float2 cached_block[1000];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = (threadIdx.z + 1) * (blockDim.x + 2) * (blockDim.y + 2) + (threadIdx.y + 1) * (blockDim.x + 2) +
        threadIdx.x + 1;
    cached_block[index] = tex3D(jacobi_packed, x, y, z);

    if (threadIdx.x == 0)
        cached_block[index - 1] = x == 0 ? cached_block[index] : tex3D(jacobi_packed, x - 1, y, z);

    if (threadIdx.x == blockDim.x - 1)
        cached_block[index + 1] = x == volume_size.x - 1 ? cached_block[index] : tex3D(jacobi_packed, x + 1, y, z);

    if (threadIdx.y == 0)
        cached_block[index - (blockDim.x + 2)] = y == 0 ? cached_block[index] : tex3D(jacobi_packed, x, y - 1, z);

    if (threadIdx.y == blockDim.y - 1)
        cached_block[index + (blockDim.x + 2)] = y == volume_size.y - 1 ? cached_block[index] : tex3D(jacobi_packed, x, y + 1, z);

    if (threadIdx.z == 0)
        cached_block[index - (blockDim.x + 2) * (blockDim.y + 2)] = z == 0 ? cached_block[index] : tex3D(jacobi_packed, x, y, z - 1);

    if (threadIdx.z == blockDim.z - 1)
        cached_block[index + (blockDim.x + 2) * (blockDim.y + 2)] = z == volume_size.z - 1 ? cached_block[index] : tex3D(jacobi_packed, x, y, z + 1);

    __syncthreads();

    float  near = cached_block[index - (blockDim.x + 2) * (blockDim.y + 2)].x;
    float  south = cached_block[index - (blockDim.x + 2)].x;
    float  west = cached_block[index - 1].x;
    float2 center = cached_block[index];
    float  east = cached_block[index + 1].x;
    float  north = cached_block[index + (blockDim.x + 2)].x;
    float  far = cached_block[index + (blockDim.x + 2) * (blockDim.y + 2)].x;

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

/*
__global__ void DampedJacobiPureKernel3(float minus_square_cell_size,
                                       float omega_over_beta, int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float2 packed_center = xys(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y, z));
    float near = z <= 0 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y, z - 1).x);
    float south = y <= 0 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y - 1, z).x);
    float west = x <= 0 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, (x - 1) * sizeof(ushort4), y, z).x);
    float east = x >= volume_size.x - 1 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, (x + 1) * sizeof(ushort4), y, z).x);
    float north = y >= volume_size.y - 1 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y + 1, z).x);
    float far = z >= volume_size.z - 1 ? packed_center.x : __half2float(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y, z + 1).x);

    float u = omega_over_beta * 3.0f * packed_center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        packed_center.y) * omega_over_beta;
    ushort4 raw = make_ushort4(__float2half_rn(u),
                               __float2half_rn(packed_center.y), 0, 0);
    surf3Dwrite(raw, jacobi, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiPureKernel4(float minus_square_cell_size,
                                        float omega_over_beta, int3 volume_size)
{
    __shared__ ushort2 cached_block[1000];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = (threadIdx.z + 1) * (blockDim.x + 2) * (blockDim.y + 2) + (threadIdx.y + 1) * (blockDim.x + 2) +
        threadIdx.x + 1;
    cached_block[index] = xyt(surf3Dread<ushort4>(jacobi, x * sizeof(ushort4), y, z));

    if (threadIdx.x == 0)
        cached_block[index - 1] = x == 0 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x - 1) * sizeof(ushort4), y, z));

    if (threadIdx.x == blockDim.x - 1)
        cached_block[index + 1] = x == volume_size.x - 1 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x + 1) * sizeof(ushort4), y, z));

    if (threadIdx.y == 0)
        cached_block[index - (blockDim.x + 2)] = y == 0 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x ) * sizeof(ushort4), y- 1, z));

    if (threadIdx.y == blockDim.y - 1)
        cached_block[index + (blockDim.x + 2)] = y == volume_size.y - 1 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x)* sizeof(ushort4), y + 1, z));

    if (threadIdx.z == 0)
        cached_block[index - (blockDim.x + 2) * (blockDim.y + 2)] = z == 0 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x)* sizeof(ushort4), y , z- 1));

    if (threadIdx.z == blockDim.z - 1)
        cached_block[index + (blockDim.x + 2) * (blockDim.y + 2)] = z == volume_size.z - 1 ? cached_block[index] : xyt(surf3Dread<ushort4>(jacobi, (x)* sizeof(ushort4), y , z+ 1));

    __syncthreads();

    float3 coord = make_float3(x, y, z);

    float near =   __half2float(cached_block[index - (blockDim.x + 2) * (blockDim.y + 2)].x);
    float south =  __half2float(cached_block[index - (blockDim.x + 2)].x);
    float west =   __half2float(cached_block[index - 1].x);
    float center = __half2float(cached_block[index].x);
    float east =   __half2float(cached_block[index + 1].x);
    float north =  __half2float(cached_block[index + (blockDim.x + 2)].x);
    float far =    __half2float(cached_block[index + (blockDim.x + 2) * (blockDim.y + 2)].x);

    float b_center = __half2float(cached_block[index].y);


    float u = omega_over_beta * 3.0f * center +
        (west + east + south + north + far + near + minus_square_cell_size *
        b_center) * omega_over_beta;
    ushort4 raw = make_ushort4(__float2half_rn(u),
                               __float2half_rn(b_center), 0, 0);
    surf3Dwrite(raw, jacobi, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}
*/

__global__ void DampedJacobiPureKernel_smem_reduced(float minus_square_cell_size,
                                        float omega_over_beta, int3 volume_size)
{
    __shared__ ushort2 cached_block[512];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x;
    
    cached_block[index] = xyu(tex3D(jacobi_packed, x , y, z));
    __syncthreads();

    float center = __half2float(cached_block[index].x);
    float near = threadIdx.z == 0 ? (z == 0 ? center : tex3D(jacobi_packed, x, y, z - 1.0f).x) : __half2float(cached_block[index - blockDim.x * blockDim.y].x);
    float south = threadIdx.y == 0 ? (y == 0 ? center : tex3D(jacobi_packed, x, y - 1.0f, z).x) : __half2float(cached_block[index - blockDim.x].x);
    float west = threadIdx.x == 0 ? (x == 0 ? center : tex3D(jacobi_packed, x - 1.0f, y, z).x) : __half2float(cached_block[index - 1].x);
    float east = threadIdx.x == blockDim.x - 1 ? (x == volume_size.x - 1 ? center : tex3D(jacobi_packed, x + 1.0f, y, z).x) : __half2float(cached_block[index + 1].x);
    float north = threadIdx.y == blockDim.y - 1 ? (y == volume_size.y - 1 ? center : tex3D(jacobi_packed, x, y + 1.0f, z).x) : __half2float(cached_block[index + blockDim.x].x);
    float far = threadIdx.z == blockDim.z - 1 ? (z == volume_size.z - 1 ? center : tex3D(jacobi_packed, x, y, z + 1.0f).x) : __half2float(cached_block[index + blockDim.x * blockDim.y].x);

    float b_center = __half2float(cached_block[index].y);

    float u = omega_over_beta * 3.0f * center +
        (west + east + south + north + far + near + minus_square_cell_size *
        b_center) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(b_center));
    cached_block[index].x = raw.x;
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiPureKernel_smem_reduced_float(float minus_square_cell_size,
                                                    float omega_over_beta, int3 volume_size)
{
    __shared__ float2 cached_block[256];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x;

    cached_block[index] = tex3D(jacobi_packed, x, y, z);
    __syncthreads();

    float center = cached_block[index].x;
    float near = threadIdx.z == 0 ? (z == 0 ? center : tex3D(jacobi_packed, x, y, z - 1.0f).x) : cached_block[index - blockDim.x * blockDim.y].x;
    float south = threadIdx.y == 0 ? (y == 0 ? center : tex3D(jacobi_packed, x, y - 1.0f, z).x) : cached_block[index - blockDim.x].x;
    float west = threadIdx.x == 0 ? (x == 0 ? center : tex3D(jacobi_packed, x - 1.0f, y, z).x) : cached_block[index - 1].x;
    float east = threadIdx.x == blockDim.x - 1 ? (x == volume_size.x - 1 ? center : tex3D(jacobi_packed, x + 1.0f, y, z).x) : cached_block[index + 1].x;
    float north = threadIdx.y == blockDim.y - 1 ? (y == volume_size.y - 1 ? center : tex3D(jacobi_packed, x, y + 1.0f, z).x) : cached_block[index + blockDim.x].x;
    float far = threadIdx.z == blockDim.z - 1 ? (z == volume_size.z - 1 ? center : tex3D(jacobi_packed, x, y, z + 1.0f).x) : cached_block[index + blockDim.x * blockDim.y].x;

    float b_center = cached_block[index].y;

    float u = omega_over_beta * 3.0f * center +
        (west + east + south + north + far + near + minus_square_cell_size *
        b_center) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(b_center));

    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void SubstractGradientPureKernel(float gradient_scale,
                                            int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float near =   tex3D(gradient_packed, coord.x, coord.y, coord.z - 1.0f).x;
    float south =  tex3D(gradient_packed, coord.x, coord.y - 1.0f, coord.z).x;
    float west =   tex3D(gradient_packed, coord.x - 1.0f, coord.y, coord.z).x;
    float center = tex3D(gradient_packed, coord.x, coord.y, coord.z).x;
    float east =   tex3D(gradient_packed, coord.x + 1.0f, coord.y, coord.z).x;
    float north =  tex3D(gradient_packed, coord.x, coord.y + 1.0f, coord.z).x;
    float far =    tex3D(gradient_packed, coord.x, coord.y, coord.z + 1.0f).x;

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f, 1.0f, 1.0f);
    if (x >= volume_size.x - 1)
        mask.x = 0;

    if (x <= 0)
        mask.x = 0;

    if (y >= volume_size.y - 1)
        mask.y = 0;

    if (y <= 0)
        mask.y = 0;

    if (z >= volume_size.z - 1)
        mask.z = 0;

    if (z <= 0)
        mask.z = 0;

    ushort4 raw;
    surf3Dread(&raw, gradient_velocity, x * sizeof(ushort4), y, z);
    float3 old_v = make_float3(__half2float(raw.x), __half2float(raw.y),
                               __half2float(raw.z));
    float3 grad = make_float3(diff_ew, diff_ns, diff_fn) * gradient_scale;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // Velocity goes to 0 when hit ???
    raw = make_ushort4(__float2half_rn(result.x), __float2half_rn(result.y),
                       __float2half_rn(result.z), 0);
    surf3Dwrite(raw, gradient_velocity, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchAdvectPure(hipArray_t dest_array, hipArray_t velocity_array,
                      hipArray_t source_array, float time_step,
                      float dissipation, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&advect_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    advect_velocity.normalized = false;
    advect_velocity.filterMode = hipFilterModeLinear;
    advect_velocity.addressMode[0] = hipAddressModeClamp;
    advect_velocity.addressMode[1] = hipAddressModeClamp;
    advect_velocity.addressMode[2] = hipAddressModeClamp;
    advect_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&advect_velocity, velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    advect_source.normalized = false;
    advect_source.filterMode = hipFilterModeLinear;
    advect_source.addressMode[0] = hipAddressModeClamp;
    advect_source.addressMode[1] = hipAddressModeClamp;
    advect_source.addressMode[2] = hipAddressModeClamp;
    advect_source.channelDesc = desc;

    result = hipBindTextureToArray(&advect_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectPureKernel<<<grid, block>>>(time_step, dissipation, volume_size);

    hipUnbindTexture(&advect_source);
    hipUnbindTexture(&advect_velocity);
}

void LaunchAdvectVelocityPure(hipArray_t dest_array,
                              hipArray_t velocity_array,
                              float time_step, float dissipation,
                              int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&advect_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    advect_velocity.normalized = false;
    advect_velocity.filterMode = hipFilterModeLinear;
    advect_velocity.addressMode[0] = hipAddressModeClamp;
    advect_velocity.addressMode[1] = hipAddressModeClamp;
    advect_velocity.addressMode[2] = hipAddressModeClamp;
    advect_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&advect_velocity, velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocityPureKernel<<<grid, block>>>(time_step, dissipation,
                                              volume_size);

    hipUnbindTexture(&advect_velocity);
}

void LaunchApplyBuoyancyPure(hipArray* dest_array, hipArray* velocity_array,
                             hipArray* temperature_array, float time_step,
                             float ambient_temperature, float accel_factor,
                             float gravity, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&buoyancy_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    buoyancy_velocity.normalized = false;
    buoyancy_velocity.filterMode = hipFilterModePoint;
    buoyancy_velocity.addressMode[0] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[1] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[2] = hipAddressModeClamp;
    buoyancy_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&buoyancy_velocity, velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, temperature_array);
    buoyancy_temperature.normalized = false;
    buoyancy_temperature.filterMode = hipFilterModePoint;
    buoyancy_temperature.addressMode[0] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[1] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[2] = hipAddressModeClamp;
    buoyancy_temperature.channelDesc = desc;

    result = hipBindTextureToArray(&buoyancy_temperature,
                                    temperature_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyPureKernel<<<grid, block>>>(time_step, ambient_temperature,
                                             accel_factor, gravity,
                                             volume_size);

    hipUnbindTexture(&buoyancy_temperature);
    hipUnbindTexture(&buoyancy_velocity);
}

void LaunchApplyImpulsePure(hipArray* dest_array, hipArray* original_array,
                            float3 center_point, float3 hotspot, float radius,
                            float value, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&impulse_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, original_array);
    impulse_original.normalized = false;
    impulse_original.filterMode = hipFilterModeLinear;
    impulse_original.addressMode[0] = hipAddressModeClamp;
    impulse_original.addressMode[1] = hipAddressModeClamp;
    impulse_original.addressMode[2] = hipAddressModeClamp;
    impulse_original.channelDesc = desc;

    result = hipBindTextureToArray(&impulse_original, original_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyImpulsePureKernel<<<grid, block>>>(center_point, hotspot, radius,
                                            value, volume_size);

    hipUnbindTexture(&impulse_original);
}

void LaunchComputeDivergencePure(hipArray* dest_array,
                                 hipArray* velocity_array,
                                 float half_inverse_cell_size, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&divergence_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    divergence_velocity.normalized = false;
    divergence_velocity.filterMode = hipFilterModePoint;
    divergence_velocity.addressMode[0] = hipAddressModeClamp;
    divergence_velocity.addressMode[1] = hipAddressModeClamp;
    divergence_velocity.addressMode[2] = hipAddressModeClamp;
    divergence_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&divergence_velocity, velocity_array,
                                    &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergencePureKernel<<<grid, block>>>(half_inverse_cell_size,
                                                 volume_size);

    hipUnbindTexture(&divergence_velocity);
}

void LaunchComputeResidualPackedDiagnosis(hipArray* dest_array,
                                          hipArray* source_array,
                                          float inverse_h_square,
                                          int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&diagnosis, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    diagnosis_source.normalized = false;
    diagnosis_source.filterMode = hipFilterModePoint;
    diagnosis_source.addressMode[0] = hipAddressModeClamp;
    diagnosis_source.addressMode[1] = hipAddressModeClamp;
    diagnosis_source.addressMode[2] = hipAddressModeClamp;
    diagnosis_source.channelDesc = desc;

    result = hipBindTextureToArray(&diagnosis_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeResidualPackedDiagnosisKernel<<<grid, block>>>(inverse_h_square,
                                                          volume_size);

    hipUnbindTexture(&diagnosis_source);
}

void LaunchDampedJacobiPure(hipArray* packed_array, float one_minus_omega,
                            float minus_square_cell_size, float omega_over_beta,
                            int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, packed_array);
    hipError_t result = cudaBindSurfaceToArray(&jacobi, packed_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, packed_array);
    jacobi_packed.normalized = false;
    jacobi_packed.filterMode = hipFilterModePoint;
    jacobi_packed.addressMode[0] = hipAddressModeClamp;
    jacobi_packed.addressMode[1] = hipAddressModeClamp;
    jacobi_packed.addressMode[2] = hipAddressModeClamp;
    jacobi_packed.channelDesc = desc;

    result = hipBindTextureToArray(&jacobi_packed, packed_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    DampedJacobiPureKernel<<<grid, block>>>(minus_square_cell_size,
                                            omega_over_beta, volume_size);

    hipUnbindTexture(&jacobi_packed);
}

void LaunchSubstractGradientPure(hipArray* dest_array, hipArray* packed_array,
                                 float gradient_scale, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&gradient_velocity, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, packed_array);
    gradient_packed.normalized = false;
    gradient_packed.filterMode = hipFilterModePoint;
    gradient_packed.addressMode[0] = hipAddressModeClamp;
    gradient_packed.addressMode[1] = hipAddressModeClamp;
    gradient_packed.addressMode[2] = hipAddressModeClamp;
    gradient_packed.channelDesc = desc;

    result = hipBindTextureToArray(&gradient_packed, packed_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    SubstractGradientPureKernel<<<grid, block>>>(gradient_scale, volume_size);

    hipUnbindTexture(&gradient_packed);
}
