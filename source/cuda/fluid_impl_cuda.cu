#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> buoyancy_dest;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_b;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_temperature;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_density;
surface<void, cudaSurfaceType3D> impulse_dest1;
surface<void, cudaSurfaceType3D> impulse_dest4;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> divergence_velocity;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> gradient_velocity;

__global__ void ApplyBuoyancyKernel(float time_step, float ambient_temperature,
                                    float accel_factor, float gravity)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);
    float d = tex3D(buoyancy_density, coord.x, coord.y, coord.z);
    float accel =
        time_step * ((t - ambient_temperature) * accel_factor - d * gravity);

    float4 velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
    ushort4 result = make_ushort4(__float2half_rn(velocity.x),
                                  __float2half_rn(velocity.y + accel),
                                  __float2half_rn(velocity.z),
                                  0);
    surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ApplyBuoyancyStaggeredKernel(float time_step,
                                             float ambient_temperature,
                                             float accel_factor, float gravity,
                                             float3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);
    float d = tex3D(buoyancy_density, coord.x, coord.y, coord.z);
    float accel =
        time_step * ((t - ambient_temperature) * accel_factor - d * gravity);

    float4 velocity;
    ushort4 result;
    if (y > 0) {
        velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
        result = make_ushort4(__float2half_rn(velocity.x),
                              __float2half_rn(velocity.y + accel * 0.5f),
                              __float2half_rn(velocity.z),
                              0);
        surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                    hipBoundaryModeTrap);
    }
    if (y < volume_size.y - 1) {
        velocity = tex3D(buoyancy_velocity, coord.x, coord.y + 1.0f, coord.z);
        result = make_ushort4(__float2half_rn(velocity.x),
                              __float2half_rn(velocity.y + accel * 0.5f),
                              __float2half_rn(velocity.z),
                              0);
        surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y + 1, z,
                    hipBoundaryModeTrap);
    }
}

__global__ void ComputeDivergenceKernel(float half_inverse_cell_size,
                                        uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z); // Careful: Non-interpolation version.

    float  near =   tex3D(divergence_velocity, coord.x,        coord.y,        coord.z - 1.0f).z;
    float  south =  tex3D(divergence_velocity, coord.x,        coord.y - 1.0f, coord.z).y;
    float  west =   tex3D(divergence_velocity, coord.x - 1.0f, coord.y,        coord.z).x;
    float4 center = tex3D(divergence_velocity, coord.x,        coord.y,        coord.z);
    float  east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y,        coord.z).x;
    float  north =  tex3D(divergence_velocity, coord.x,        coord.y + 1.0f, coord.z).y;
    float  far =    tex3D(divergence_velocity, coord.x,        coord.y,        coord.z + 1.0f).z;

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem.
    if (x >= volume_size.x - 1)
        diff_ew = -center.x - west;

    if (x <= 0)
        diff_ew = east + center.x;

    if (y >= volume_size.y - 1)
        diff_ns = -center.y - south;

    if (y <= 0)
        diff_ns = north + center.y;

    if (z >= volume_size.z - 1)
        diff_fn = -center.z - near;

    if (z <= 0)
        diff_fn = far + center.z;

    float div = half_inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    auto r = __float2half_rn(div);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

__global__ void ComputeDivergenceStaggeredKernel(float inverse_cell_size,
                                                 uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float4 base =   tex3D(divergence_velocity, coord.x,        coord.y,        coord.z);
    float  east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y,        coord.z).x;
    float  north =  tex3D(divergence_velocity, coord.x,        coord.y + 1.0f, coord.z).y;
    float  far =    tex3D(divergence_velocity, coord.x,        coord.y,        coord.z + 1.0f).z;

    float diff_ew = east  - base.x;
    float diff_ns = north - base.y;
    float diff_fn = far   - base.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -base.x;

    if (y >= volume_size.y - 1)
        diff_ns = -base.y;

    if (z >= volume_size.z - 1)
        diff_fn = -base.z;

    float div = inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    auto r = __float2half_rn(div);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

__global__ void ComputeResidualDiagnosisKernel(float inverse_h_square,
                                               uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float near =   tex3D(tex, coord.x, coord.y, coord.z - 1.0f);
    float south =  tex3D(tex, coord.x, coord.y - 1.0f, coord.z);
    float west =   tex3D(tex, coord.x - 1.0f, coord.y, coord.z);
    float center = tex3D(tex, coord.x, coord.y, coord.z);
    float east =   tex3D(tex, coord.x + 1.0f, coord.y, coord.z);
    float north =  tex3D(tex, coord.x, coord.y + 1.0f, coord.z);
    float far =    tex3D(tex, coord.x, coord.y, coord.z + 1.0f);
    float b =      tex3D(tex_b, coord.x, coord.y, coord.z);

    if (coord.y == volume_size.y - 1)
        north = center;

    if (coord.y == 0)
        south = center;

    if (coord.x == volume_size.x - 1)
        east = center;

    if (coord.x == 0)
        west = center;

    if (coord.z == volume_size.z - 1)
        far = center;

    if (coord.z == 0)
        near = center;

    float v = b -
        (north + south + east + west + far + near - 6.0 * center) *
        inverse_h_square;
    surf3Dwrite(fabsf(v), surf, x * sizeof(float), y, z, hipBoundaryModeTrap);
}

__global__ void RoundPassedKernel(int* dest_array, int round, int x)
{
    dest_array[0] = x * x - round * round;
}

__global__ void SubtractGradientKernel(float half_inverse_cell_size,
                                       uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z); // Careful: Non-interpolation version.

    float near =   tex3D(tex, coord.x, coord.y, coord.z - 1.0f);
    float south =  tex3D(tex, coord.x, coord.y - 1.0f, coord.z);
    float west =   tex3D(tex, coord.x - 1.0f, coord.y, coord.z);
    float center = tex3D(tex, coord.x, coord.y, coord.z);
    float east =   tex3D(tex, coord.x + 1.0f, coord.y, coord.z);
    float north =  tex3D(tex, coord.x, coord.y + 1.0f, coord.z);
    float far =    tex3D(tex, coord.x, coord.y, coord.z + 1.0f);

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f);
    if (x >= volume_size.x - 1) // Careful: Non-interpolation version.
        mask.x = 0.0f;

    if (x <= 0)
        mask.x = 0.0f;

    if (y >= volume_size.y - 1)
        mask.y = 0.0f;

    if (y <= 0)
        mask.y = 0.0f;

    if (z >= volume_size.z - 1)
        mask.z = 0.0f;

    if (z <= 0)
        mask.z = 0.0f;

    float3 old_v =
        make_float3(tex3D(gradient_velocity, coord.x, coord.y, coord.z));
    float3 grad =
        make_float3(diff_ew, diff_ns, diff_fn) * half_inverse_cell_size;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // Velocity goes to 0 when hit ???
    ushort4 raw = make_ushort4(__float2half_rn(result.x),
                               __float2half_rn(result.y),
                               __float2half_rn(result.z),
                               0);
    surf3Dwrite(raw, surf, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void SubtractGradientStaggeredKernel(float inverse_cell_size,
                                                uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    
    float near =  tex3D(tex, coord.x,        coord.y,          coord.z - 1.0f);
    float south = tex3D(tex, coord.x,        coord.y - 1.0f,   coord.z);
    float west =  tex3D(tex, coord.x - 1.0f, coord.y,          coord.z);
    float base =  tex3D(tex, coord.x,        coord.y,          coord.z);

    float diff_ew = base - west;
    float diff_ns = base - south;
    float diff_fn = base - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f);
    if (x <= 0)
        mask.x = 0;

    if (y <= 0)
        mask.y = 0;

    if (z <= 0)
        mask.z = 0;

    float3 old_v =
        make_float3(tex3D(gradient_velocity, coord.x, coord.y, coord.z));
    float3 grad = make_float3(diff_ew, diff_ns, diff_fn) * inverse_cell_size;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // The mask makes sense in staggered grid.
    auto raw = make_ushort4(__float2half_rn(result.x),
                            __float2half_rn(result.y),
                            __float2half_rn(result.z),
                            0);
    surf3Dwrite(raw, surf, x * sizeof(raw), y, z, hipBoundaryModeTrap);
}

// =============================================================================

void LaunchApplyBuoyancy(hipArray* dest_array, hipArray* velocity_array,
                         hipArray* temperature_array, hipArray* density_array,
                         float time_step, float ambient_temperature,
                         float accel_factor, float gravity, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&buoyancy_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&buoyancy_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_temp = BindHelper::Bind(&buoyancy_temperature, temperature_array,
                                       false, hipFilterModeLinear,
                                       hipAddressModeClamp);
    if (bound_temp.error() != hipSuccess)
        return;

    auto bound_density = BindHelper::Bind(&buoyancy_density, density_array,
                                          false, hipFilterModeLinear,
                                          hipAddressModeClamp);
    if (bound_density.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyKernel<<<grid, block>>>(time_step, ambient_temperature,
                                         accel_factor, gravity);
}

void LaunchApplyBuoyancyStaggered(hipArray* dest_array,
                                  hipArray* velocity_array,
                                  hipArray* temperature_array,
                                  hipArray* density_array, float time_step,
                                  float ambient_temperature, float accel_factor,
                                  float gravity, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&buoyancy_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&buoyancy_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_temp = BindHelper::Bind(&buoyancy_temperature, temperature_array,
                                       false, hipFilterModeLinear,
                                       hipAddressModeClamp);
    if (bound_temp.error() != hipSuccess)
        return;

    auto bound_density = BindHelper::Bind(&buoyancy_density, density_array,
                                          false, hipFilterModeLinear,
                                          hipAddressModeClamp);
    if (bound_density.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyStaggeredKernel<<<grid, block>>>(time_step,
                                                  ambient_temperature,
                                                  accel_factor, gravity,
                                                  make_float3(volume_size));
}

void LaunchComputeDivergence(hipArray* dest_array, hipArray* velocity_array,
                             float half_inverse_cell_size, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&surf, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&divergence_velocity, velocity_array,
                                      false, hipFilterModePoint,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceKernel<<<grid, block>>>(half_inverse_cell_size,
                                             volume_size);
}

void LaunchComputeDivergenceStaggered(hipArray* dest_array,
                                      hipArray* velocity_array,
                                      float inverse_cell_size,
                                      uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&surf, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&divergence_velocity, velocity_array,
                                      false, hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceStaggeredKernel<<<grid, block>>>(inverse_cell_size,
                                                      volume_size);
}

void LaunchComputeResidualDiagnosis(hipArray* residual, hipArray* u,
                                    hipArray* b, float inverse_h_square,
                                    uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&surf, residual) != hipSuccess)
        return;

    auto bound_u = BindHelper::Bind(&tex, u, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_u.error() != hipSuccess)
        return;

    auto bound_b = BindHelper::Bind(&tex_b, b, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_b.error() != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeResidualDiagnosisKernel<<<grid, block>>>(inverse_h_square,
                                                    volume_size);
}

void LaunchRoundPassed(int* dest_array, int round, int x)
{
    RoundPassedKernel<<<1, 1>>>(dest_array, round, x);
}

void LaunchSubtractGradient(hipArray* velocity, hipArray* pressure,
                            float half_inverse_cell_size, uint3 volume_size,
                            BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, velocity) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&gradient_velocity, velocity,
                                      false, hipFilterModePoint,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound = BindHelper::Bind(&tex, pressure, false,
                                  hipFilterModePoint, hipAddressModeClamp);
    if (bound.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    SubtractGradientKernel<<<grid, block>>>(half_inverse_cell_size,
                                            volume_size);
}

void LaunchSubtractGradientStaggered(hipArray* velocity, hipArray* pressure,
                                     float inverse_cell_size, uint3 volume_size,
                                     BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, velocity) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&gradient_velocity, velocity,
                                      false, hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound = BindHelper::Bind(&tex, pressure, false, hipFilterModeLinear,
                                  hipAddressModeClamp);
    if (bound.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    SubtractGradientStaggeredKernel<<<grid, block>>>(inverse_cell_size,
                                                     volume_size);
}
