#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> buoyancy_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_temperature;
surface<void, cudaSurfaceType3D> impulse_dest1;
surface<void, cudaSurfaceType3D> impulse_dest4;
surface<void, cudaSurfaceType3D> divergence_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> divergence_velocity;
surface<void, cudaSurfaceType3D> gradient_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> gradient_velocity;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> gradient_packed;
surface<void, cudaSurfaceType3D> diagnosis;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> diagnosis_source;

__global__ void ApplyBuoyancyKernel(float time_step, float ambient_temperature,
                                    float accel_factor, float gravity)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);
    float accel = time_step * ((t - ambient_temperature) * accel_factor -
                                gravity);

    float4 velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
    ushort4 result = make_ushort4(__float2half_rn(velocity.x),
                                  __float2half_rn(velocity.y + accel),
                                  __float2half_rn(velocity.z),
                                  0);
    surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ApplyBuoyancyStaggeredKernel(float time_step,
                                             float ambient_temperature,
                                             float accel_factor, float gravity,
                                             float3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);
    float accel = time_step * ((t - ambient_temperature) * accel_factor -
                               gravity);

    float4 velocity;
    ushort4 result;
    if (y > 0) {
        velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
        result = make_ushort4(__float2half_rn(velocity.x),
                              __float2half_rn(velocity.y + accel * 0.5f),
                              __float2half_rn(velocity.z),
                              0);
        surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                    hipBoundaryModeTrap);
    }
    if (y < volume_size.y - 1) {
        velocity = tex3D(buoyancy_velocity, coord.x, coord.y + 1.0f, coord.z);
        result = make_ushort4(__float2half_rn(velocity.x),
                              __float2half_rn(velocity.y + accel * 0.5f),
                              __float2half_rn(velocity.z),
                              0);
        surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y + 1, z,
                    hipBoundaryModeTrap);
    }
}

__global__ void ApplyImpulse1Kernel(float3 center_point, float3 hotspot,
                                    float radius, float value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        surf3Dwrite(__float2half_rn(scale * value), impulse_dest1,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void ApplyImpulse1Kernel2(float3 center_point, float3 hotspot,
                                     float radius, float value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff =
        make_float2(coord.x, coord.z) - make_float2(hotspot.x, hotspot.z);
    float d = hypotf(diff.x, diff.y);
    if (d < 2.0f) {
        surf3Dwrite(__float2half_rn(value), impulse_dest1,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensityKernel(float3 center_point, float radius,
                                     float value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), impulse_dest1,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
    }
}

__global__ void ApplyImpulse3Kernel(float3 center_point, float3 hotspot,
                                    float radius, float3 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        ushort4 result = make_ushort4(__float2half_rn(scale * value.x),
                                      __float2half_rn(scale * value.y),
                                      __float2half_rn(scale * value.z),
                                      0);
        surf3Dwrite(result, impulse_dest4, x * sizeof(ushort4), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

__global__ void ComputeDivergenceKernel(float half_inverse_cell_size,
                                        uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z); // Careful: Non-interpolation version.

    float  near =   tex3D(divergence_velocity, coord.x,        coord.y,        coord.z - 1.0f).z;
    float  south =  tex3D(divergence_velocity, coord.x,        coord.y - 1.0f, coord.z).y;
    float  west =   tex3D(divergence_velocity, coord.x - 1.0f, coord.y,        coord.z).x;
    float4 center = tex3D(divergence_velocity, coord.x,        coord.y,        coord.z);
    float  east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y,        coord.z).x;
    float  north =  tex3D(divergence_velocity, coord.x,        coord.y + 1.0f, coord.z).y;
    float  far =    tex3D(divergence_velocity, coord.x,        coord.y,        coord.z + 1.0f).z;

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem.
    if (x >= volume_size.x - 1)
        diff_ew = -center.x - west;

    if (x <= 0)
        diff_ew = east + center.x;

    if (y >= volume_size.y - 1)
        diff_ns = -center.y - south;

    if (y <= 0)
        diff_ns = north + center.y;

    if (z >= volume_size.z - 1)
        diff_fn = -center.z - near;

    if (z <= 0)
        diff_fn = far + center.z;

    float div = half_inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    ushort2 result = make_ushort2(0, __float2half_rn(div));
    surf3Dwrite(result, divergence_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ComputeDivergenceStaggeredKernel(float inverse_cell_size,
                                                 uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float4 base =   tex3D(divergence_velocity, coord.x,        coord.y,        coord.z);
    float  east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y,        coord.z).x;
    float  north =  tex3D(divergence_velocity, coord.x,        coord.y + 1.0f, coord.z).y;
    float  far =    tex3D(divergence_velocity, coord.x,        coord.y,        coord.z + 1.0f).z;

    float diff_ew = east  - base.x;
    float diff_ns = north - base.y;
    float diff_fn = far   - base.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -base.x;

    if (y >= volume_size.y - 1)
        diff_ns = -base.y;

    if (z >= volume_size.z - 1)
        diff_fn = -base.z;

    float div = inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    ushort2 result = make_ushort2(0, __float2half_rn(div));
    surf3Dwrite(result, divergence_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ComputeResidualPackedDiagnosisKernel(float inverse_h_square,
                                                     uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float  near =   tex3D(diagnosis_source, coord.x, coord.y, coord.z - 1.0f).x;
    float  south =  tex3D(diagnosis_source, coord.x, coord.y - 1.0f, coord.z).x;
    float  west =   tex3D(diagnosis_source, coord.x - 1.0f, coord.y, coord.z).x;
    float2 center = tex3D(diagnosis_source, coord.x, coord.y, coord.z);
    float  east =   tex3D(diagnosis_source, coord.x + 1.0f, coord.y, coord.z).x;
    float  north =  tex3D(diagnosis_source, coord.x, coord.y + 1.0f, coord.z).x;
    float  far =    tex3D(diagnosis_source, coord.x, coord.y, coord.z + 1.0f).x;
    float  b_center = center.y;

    if (coord.y == volume_size.y - 1)
        north = center.x;

    if (coord.y == 0)
        south = center.x;

    if (coord.x == volume_size.x - 1)
        east = center.x;

    if (coord.x == 0)
        west = center.x;

    if (coord.z == volume_size.z - 1)
        far = center.x;

    if (coord.z == 0)
        near = center.x;

    float v = b_center -
        (north + south + east + west + far + near - 6.0 * center.x) *
        inverse_h_square;
    surf3Dwrite(fabsf(v), diagnosis, x * sizeof(float), y, z,
                hipBoundaryModeTrap);
}

__global__ void RoundPassedKernel(int* dest_array, int round, int x)
{
    dest_array[0] = x * x - round * round;
}

__global__ void SubtractGradientKernel(float half_inverse_cell_size,
                                       uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z); // Careful: Non-interpolation version.

    float near =   tex3D(gradient_packed, coord.x, coord.y, coord.z - 1.0f).x;
    float south =  tex3D(gradient_packed, coord.x, coord.y - 1.0f, coord.z).x;
    float west =   tex3D(gradient_packed, coord.x - 1.0f, coord.y, coord.z).x;
    float center = tex3D(gradient_packed, coord.x, coord.y, coord.z).x;
    float east =   tex3D(gradient_packed, coord.x + 1.0f, coord.y, coord.z).x;
    float north =  tex3D(gradient_packed, coord.x, coord.y + 1.0f, coord.z).x;
    float far =    tex3D(gradient_packed, coord.x, coord.y, coord.z + 1.0f).x;

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f);
    if (x >= volume_size.x - 1) // Careful: Non-interpolation version.
        mask.x = 0.0f;

    if (x <= 0)
        mask.x = 0.0f;

    if (y >= volume_size.y - 1)
        mask.y = 0.0f;

    if (y <= 0)
        mask.y = 0.0f;

    if (z >= volume_size.z - 1)
        mask.z = 0.0f;

    if (z <= 0)
        mask.z = 0.0f;

    float3 old_v =
        make_float3(tex3D(gradient_velocity, coord.x, coord.y, coord.z));
    float3 grad =
        make_float3(diff_ew, diff_ns, diff_fn) * half_inverse_cell_size;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // Velocity goes to 0 when hit ???
    ushort4 raw = make_ushort4(__float2half_rn(result.x),
                               __float2half_rn(result.y),
                               __float2half_rn(result.z),
                               0);
    surf3Dwrite(raw, gradient_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void SubtractGradientStaggeredKernel(float inverse_cell_size,
                                                uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    
    float near =  tex3D(gradient_packed, coord.x,        coord.y,          coord.z - 1.0f).x;
    float south = tex3D(gradient_packed, coord.x,        coord.y - 1.0f,   coord.z).x;
    float west =  tex3D(gradient_packed, coord.x - 1.0f, coord.y,          coord.z).x;
    float base =  tex3D(gradient_packed, coord.x,        coord.y,          coord.z).x;

    float diff_ew = base - west;
    float diff_ns = base - south;
    float diff_fn = base - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f);
    if (x <= 0)
        mask.x = 0;

    if (y <= 0)
        mask.y = 0;

    if (z <= 0)
        mask.z = 0;

    float3 old_v =
        make_float3(tex3D(gradient_velocity, coord.x, coord.y, coord.z));
    float3 grad = make_float3(diff_ew, diff_ns, diff_fn) * inverse_cell_size;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // The mask makes sense in staggered grid.
    ushort4 raw = make_ushort4(__float2half_rn(result.x),
                               __float2half_rn(result.y),
                               __float2half_rn(result.z),
                               0);
    surf3Dwrite(raw, gradient_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchApplyBuoyancy(hipArray* dest_array, hipArray* velocity_array,
                         hipArray* temperature_array, float time_step,
                         float ambient_temperature, float accel_factor,
                         float gravity, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&buoyancy_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&buoyancy_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_temp = BindHelper::Bind(&buoyancy_temperature, temperature_array,
                                       false, hipFilterModeLinear,
                                       hipAddressModeClamp);
    if (bound_temp.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyKernel<<<grid, block>>>(time_step, ambient_temperature,
                                         accel_factor, gravity);
}

void LaunchApplyBuoyancyStaggered(hipArray* dest_array,
                                  hipArray* velocity_array,
                                  hipArray* temperature_array, float time_step,
                                  float ambient_temperature, float accel_factor,
                                  float gravity, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&buoyancy_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&buoyancy_velocity, velocity_array, false,
                                      hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_temp = BindHelper::Bind(&buoyancy_temperature, temperature_array,
                                       false, hipFilterModeLinear,
                                       hipAddressModeClamp);
    if (bound_temp.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyStaggeredKernel<<<grid, block>>>(time_step,
                                                  ambient_temperature,
                                                  accel_factor, gravity,
                                                  make_float3(volume_size));
}

void LaunchApplyImpulse(hipArray* dest_array, hipArray* original_array,
                        float3 center_point, float3 hotspot, float radius,
                        float3 value, uint32_t mask, uint3 volume_size)
{
    assert(mask == 1 || mask == 7);
    if (mask != 1 && mask != 7)
        return;

    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    dim3 block(128, 2, 1);
    dim3 grid(volume_size.x / block.x, 1, volume_size.z / block.z);
    if (mask == 1) {
        if (BindCudaSurfaceToArray(&impulse_dest1, dest_array) != hipSuccess)
            return;

        ApplyImpulse1Kernel<<<grid, block>>>(center_point, hotspot, radius,
                                             value.x);
    } else if (mask == 7) {
        if (BindCudaSurfaceToArray(&impulse_dest4, dest_array) != hipSuccess)
            return;

        ApplyImpulse3Kernel<<<grid, block>>>(center_point, hotspot, radius,
                                             value);
    }
}

void LaunchComputeDivergence(hipArray* dest_array, hipArray* velocity_array,
                             float half_inverse_cell_size, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&divergence_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&divergence_velocity, velocity_array,
                                      false, hipFilterModePoint,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceKernel<<<grid, block>>>(half_inverse_cell_size,
                                             volume_size);
}

void LaunchComputeDivergenceStaggered(hipArray* dest_array,
                                      hipArray* velocity_array,
                                      float inverse_cell_size,
                                      uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&divergence_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&divergence_velocity, velocity_array,
                                      false, hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceStaggeredKernel<<<grid, block>>>(inverse_cell_size,
                                                      volume_size);
}

void LaunchComputeResidualPackedDiagnosis(hipArray* dest_array,
                                          hipArray* source_array,
                                          float inverse_h_square,
                                          uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&diagnosis, dest_array) != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&diagnosis_source, source_array,
                                         false, hipFilterModePoint,
                                         hipAddressModeClamp);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeResidualPackedDiagnosisKernel<<<grid, block>>>(inverse_h_square,
                                                          volume_size);
}

void LaunchImpulseDensity(hipArray* dest_array, hipArray* original_array,
                          float3 center_point, float radius, float3 value,
                          uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&impulse_dest1, dest_array) != hipSuccess)
        return;

    dim3 block(128, 2, 1);
    dim3 grid(volume_size.x / block.x, 1, volume_size.z / block.z);
    ImpulseDensityKernel<<<grid, block>>>(center_point, radius, value.x);
}

void LaunchRoundPassed(int* dest_array, int round, int x)
{
    RoundPassedKernel<<<1, 1>>>(dest_array, round, x);
}

void LaunchSubtractGradient(hipArray* dest_array, hipArray* packed_array,
                            float half_inverse_cell_size, uint3 volume_size,
                            BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&gradient_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&gradient_velocity, dest_array,
                                      false, hipFilterModePoint,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_packed = BindHelper::Bind(&gradient_packed, packed_array,
                                         false, hipFilterModePoint,
                                         hipAddressModeClamp);
    if (bound_packed.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    SubtractGradientKernel<<<grid, block>>>(half_inverse_cell_size,
                                            volume_size);
}

void LaunchSubtractGradientStaggered(hipArray* dest_array,
                                     hipArray* packed_array,
                                     float inverse_cell_size, uint3 volume_size,
                                     BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&gradient_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&gradient_velocity, dest_array,
                                      false, hipFilterModeLinear,
                                      hipAddressModeClamp);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_packed = BindHelper::Bind(&gradient_packed, packed_array,
                                         false, hipFilterModeLinear,
                                         hipAddressModeClamp);
    if (bound_packed.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    SubtractGradientStaggeredKernel<<<grid, block>>>(inverse_cell_size,
                                                     volume_size);
}
