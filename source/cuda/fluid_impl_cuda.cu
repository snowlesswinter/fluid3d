#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/multi_precision.cuh"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_t;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_d;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_b;
texture<float, hipTextureType3D, hipReadModeElementType> texf;
texture<float, hipTextureType3D, hipReadModeElementType> texf_b;
texture<long2, hipTextureType3D, hipReadModeElementType> texd;
texture<long2, hipTextureType3D, hipReadModeElementType> texd_b;

template <typename FPType>
struct UpperBoundaryHandlerNeumann
{
    __device__ void HandleUpperBoundary(FPType* diff_ns, FPType base_y)
    {
        *diff_ns = -base_y;
    }
};

template <typename FPType>
struct UpperBoundaryHandlerOutflow
{
    __device__ void HandleUpperBoundary(FPType* diff_ns, FPType base_y)
    {
        if (base_y < 0.0f)
            *diff_ns = -base_y;
        else
            *diff_ns = 0.0f;
    }
};

// =============================================================================

__global__ void ApplyBuoyancyKernel(float time_step, float ambient_temperature,
                                    float accel_factor, float gravity,
                                    uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float t = tex3D(tex_t, coord.x, coord.y, coord.z);
    float d = tex3D(tex_d, coord.x, coord.y, coord.z);
    float accel =
        time_step * ((t - ambient_temperature) * accel_factor - d * gravity);

    float velocity = tex3D(tex, coord.x, coord.y, coord.z);
    auto result = __float2half_rn(velocity + accel);
    surf3Dwrite(result, surf, x * sizeof(result), y, z, hipBoundaryModeTrap);
}

__global__ void ApplyBuoyancyStaggeredKernel(float time_step,
                                             float ambient_temperature,
                                             float accel_factor, float gravity,
                                             uint3 volume_size)
{
    int x = VolumeX();
    int z = VolumeZ();

    if (x >= volume_size.x || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, 0, z) + 0.5f;
    float t_prev = tex3D(tex_t, coord.x, coord.y, coord.z);
    float d_prev = tex3D(tex_d, coord.x, coord.y, coord.z);
    float accel_prev = time_step *
        ((t_prev - ambient_temperature) * accel_factor - d_prev * gravity);

    float y = 1.5f;
    for (int i = 1; i < volume_size.y; i++, y += 1.0f) {
        float t = tex3D(tex_t, coord.x, y, coord.z);
        float d = tex3D(tex_d, coord.x, y, coord.z);
        float accel = time_step *
            ((t - ambient_temperature) * accel_factor - d * gravity);

        float velocity = tex3D(tex, coord.x, y, coord.z);

        auto r = __float2half_rn(velocity + (accel_prev + accel) * 0.5f);
        surf3Dwrite(r, surf, x * sizeof(r), i, z, hipBoundaryModeTrap);

        t_prev = t;
        d_prev = d;
        accel_prev = accel;
    }
}

__global__ void DecayVelocityKernel(float velocity_dissipation,
                                    uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float vel_x = tex3D(tex_x, coord.x, coord.y, coord.z) * velocity_dissipation;
    float vel_y = tex3D(tex_y, coord.x, coord.y, coord.z) * velocity_dissipation;
    float vel_z = tex3D(tex_z, coord.x, coord.y, coord.z) * velocity_dissipation;

    auto r_x = __float2half_rn(vel_x);
    surf3Dwrite(r_x, surf_x, x * sizeof(r_x), y, z, hipBoundaryModeTrap);
    auto r_y = __float2half_rn(vel_y);
    surf3Dwrite(r_y, surf_y, x * sizeof(r_y), y, z, hipBoundaryModeTrap);
    auto r_z = __float2half_rn(vel_z);
    surf3Dwrite(r_z, surf_z, x * sizeof(r_z), y, z, hipBoundaryModeTrap);
}

template <typename UpperBoundaryHandler>
__global__ void ComputeDivergenceKernel(float half_inverse_cell_size,
                                        uint3 volume_size,
                                        UpperBoundaryHandler handler)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float west =     tex3D(tex_x, coord.x - 1.0f, coord.y,        coord.z);
    float center_x = tex3D(tex_x, coord.x,        coord.y,        coord.z);
    float east =     tex3D(tex_x, coord.x + 1.0f, coord.y,        coord.z);
    float south =    tex3D(tex_y, coord.x,        coord.y - 1.0f, coord.z);
    float center_y = tex3D(tex_y, coord.x,        coord.y,        coord.z);
    float north =    tex3D(tex_y, coord.x,        coord.y + 1.0f, coord.z);
    float near =     tex3D(tex_z, coord.x,        coord.y,        coord.z - 1.0f);
    float center_z = tex3D(tex_z, coord.x,        coord.y,        coord.z);
    float far =      tex3D(tex_z, coord.x,        coord.y,        coord.z + 1.0f);

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem.
    if (x >= volume_size.x - 1)
        diff_ew = (center_x + west) * -0.5f;

    if (x <= 0)
        diff_ew = (east + center_x) * 0.5f;

    if (y >= volume_size.y - 1)
        handler.HandleUpperBoundary(&diff_ns, (center_y + south) * 0.5f);

    if (y <= 0)
        diff_ns = (north + center_y) * 0.5f;

    if (z >= volume_size.z - 1)
        diff_fn = (center_z + near) * -0.5f;

    if (z <= 0)
        diff_fn = (far + center_z) * 0.5f;

    float div = half_inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    auto r = __float2half_rn(div);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <typename StorageType, typename UpperBoundaryHandler>
__global__ void ComputeDivergenceStaggeredKernel(float cell_size,
                                                 uint3 volume_size,
                                                 UpperBoundaryHandler handler)
{
    using FPType = typename Tex3d<StorageType>::ValType;

    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    FPType base_x = tex3D(tex_x, coord.x,        coord.y,        coord.z);
    FPType base_y = tex3D(tex_y, coord.x,        coord.y,        coord.z);
    FPType base_z = tex3D(tex_z, coord.x,        coord.y,        coord.z);
    FPType east =   tex3D(tex_x, coord.x + 1.0f, coord.y,        coord.z);
    FPType north =  tex3D(tex_y, coord.x,        coord.y + 1.0f, coord.z);
    FPType far =    tex3D(tex_z, coord.x,        coord.y,        coord.z + 1.0f);

    FPType diff_ew = east  - base_x;
    FPType diff_ns = north - base_y;
    FPType diff_fn = far   - base_z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -base_x;

    if (y >= volume_size.y - 1)
        handler.HandleUpperBoundary(&diff_ns, base_y);

    if (z >= volume_size.z - 1)
        diff_fn = -base_z;

    // NOTE: Premultiply h^2 to get a uniformed cell size at all levels
    //       of multigrid hierarchy.
    FPType div = cell_size * (diff_ew + diff_ns + diff_fn);

    Tex3d<StorageType> t3d;
    t3d.Store(div, surf, x, y, z);
}

template <typename StorageType>
__global__ void ComputeResidualDiagnosisKernel(float inverse_h_square,
                                               uint3 volume_size)
{
    using FPType = typename Tex3d<StorageType>::ValType;

    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z);

    Tex3d<StorageType> t3d;
    FPType near =   t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x, coord.y, coord.z - 1.0f);
    FPType south =  t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x, coord.y - 1.0f, coord.z);
    FPType west =   t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x - 1.0f, coord.y, coord.z);
    FPType center = t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x, coord.y, coord.z);
    FPType east =   t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x + 1.0f, coord.y, coord.z);
    FPType north =  t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x, coord.y + 1.0f, coord.z);
    FPType far =    t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x, coord.y, coord.z + 1.0f);
    FPType b =      t3d(TexSel<StorageType>::Tex(tex_b, texf_b, texd_b), coord.x, coord.y, coord.z);

    if (coord.y == volume_size.y - 1)
        north = center;

    if (coord.y == 0)
        south = center;

    if (coord.x == volume_size.x - 1)
        east = center;

    if (coord.x == 0)
        west = center;

    if (coord.z == volume_size.z - 1)
        far = center;

    if (coord.z == 0)
        near = center;

    FPType v = (b - (north + south + east + west + far + near - 6.0 * center)) *
        inverse_h_square;

    // Destination is a fp32 volume.
    surf3Dwrite(fabsf(v), surf, x * sizeof(float), y, z, hipBoundaryModeTrap);
}

__global__ void RoundPassedKernel(int* dest_array, int round, int x)
{
    dest_array[0] = x * x - round * round;
}

__global__ void SubtractGradientKernel(float half_inverse_cell_size,
                                       uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float near =   tex3D(tex, coord.x, coord.y, coord.z - 1.0f);
    float south =  tex3D(tex, coord.x, coord.y - 1.0f, coord.z);
    float west =   tex3D(tex, coord.x - 1.0f, coord.y, coord.z);
    float center = tex3D(tex, coord.x, coord.y, coord.z);
    float east =   tex3D(tex, coord.x + 1.0f, coord.y, coord.z);
    float north =  tex3D(tex, coord.x, coord.y + 1.0f, coord.z);
    float far =    tex3D(tex, coord.x, coord.y, coord.z + 1.0f);

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f);
    if (x >= volume_size.x - 1)
        mask.x = 0.0f;

    if (x <= 0)
        mask.x = 0.0f;

    if (y >= volume_size.y - 1)
        mask.y = 0.0f;

    if (y <= 0)
        mask.y = 0.0f;

    if (z >= volume_size.z - 1)
        mask.z = 0.0f;

    if (z <= 0)
        mask.z = 0.0f;

    float old_x = tex3D(tex_x, coord.x, coord.y, coord.z);
    float grad_x = diff_ew * half_inverse_cell_size;
    float new_x = old_x - grad_x;
    auto r_x = __float2half_rn(new_x * mask.x);
    surf3Dwrite(r_x, surf_x, x * sizeof(r_x), y, z, hipBoundaryModeTrap);

    float old_y = tex3D(tex_y, coord.x, coord.y, coord.z);
    float grad_y = diff_ns * half_inverse_cell_size;
    float new_y = old_y - grad_y;
    auto r_y = __float2half_rn(new_y * mask.y);
    surf3Dwrite(r_y, surf_y, x * sizeof(r_y), y, z, hipBoundaryModeTrap);

    float old_z = tex3D(tex_z, coord.x, coord.y, coord.z);
    float grad_z = diff_fn * half_inverse_cell_size;
    float new_z = old_z - grad_z;
    auto r_z = __float2half_rn(new_z * mask.z);
    surf3Dwrite(r_z, surf_z, x * sizeof(r_z), y, z, hipBoundaryModeTrap);
}

template <typename StorageType>
__global__ void SubtractGradientStaggeredKernel(float inverse_cell_size,
                                                uint3 volume_size)
{
    using FPType = typename Tex3d<StorageType>::ValType;

    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z);
    
    Tex3d<StorageType> t3d;
    FPType near =  t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x,        coord.y,          coord.z - 1.0f);
    FPType south = t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x,        coord.y - 1.0f,   coord.z);
    FPType west =  t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x - 1.0f, coord.y,          coord.z);
    FPType base =  t3d(TexSel<StorageType>::Tex(tex, texf, texd), coord.x,        coord.y,          coord.z);

    // Handle boundary problem.
    FPType mask = 1.0f;
    if (x <= 0)
        mask = 0;

    if (y <= 0)
        mask = 0;

    if (z <= 0)
        mask = 0;

    FPType old_x = tex3D(tex_x, coord.x, coord.y, coord.z);
    FPType grad_x = (base - west) * inverse_cell_size;
    FPType new_x = old_x - grad_x;
    auto r_x = __float2half_rn(new_x * mask);
    surf3Dwrite(r_x, surf_x, x * sizeof(r_x), y, z, hipBoundaryModeTrap);

    FPType old_y = tex3D(tex_y, coord.x, coord.y, coord.z);
    FPType grad_y = (base - south) * inverse_cell_size;
    FPType new_y = old_y - grad_y;
    auto r_y = __float2half_rn(new_y * mask);
    surf3Dwrite(r_y, surf_y, x * sizeof(r_y), y, z, hipBoundaryModeTrap);

    FPType old_z = tex3D(tex_z, coord.x, coord.y, coord.z);
    FPType grad_z = (base - near) * inverse_cell_size;
    FPType new_z = old_z - grad_z;
    auto r_z = __float2half_rn(new_z * mask);
    surf3Dwrite(r_z, surf_z, x * sizeof(r_z), y, z, hipBoundaryModeTrap);
}

// =============================================================================

template <typename StorageType>
struct ComputeDivergenceStaggeredKernelMeta
{
    static void Invoke(const dim3& grid, const dim3& block, float cell_size,
                       const uint3& volume_size, bool outflow)
    {
        using FPType = typename Tex3d<StorageType>::ValType;
        UpperBoundaryHandlerOutflow<FPType> outflow_handler;
        UpperBoundaryHandlerNeumann<FPType> neumann_handler;
        if (outflow)
            ComputeDivergenceStaggeredKernel<StorageType><<<grid, block>>>(
                cell_size, volume_size, outflow_handler);
        else
            ComputeDivergenceStaggeredKernel<StorageType><<<grid, block>>>(
                cell_size, volume_size, neumann_handler);
    }
};

DECLARE_KERNEL_META(
    ComputeResidualDiagnosisKernel,
    MAKE_INVOKE_DECLARATION(float inverse_h_square, const uint3& volume_size),
    inverse_h_square, volume_size);

DECLARE_KERNEL_META(
    SubtractGradientStaggeredKernel,
    MAKE_INVOKE_DECLARATION(float inverse_cell_size, const uint3& volume_size),
    inverse_cell_size, volume_size);

// =============================================================================

namespace kern_launcher
{
void ApplyBuoyancy(hipArray* vnp1_x, hipArray* vnp1_y, hipArray* vnp1_z,
                   hipArray* vn_x, hipArray* vn_y, hipArray* vn_z,
                   hipArray* temperature, hipArray* density, float time_step,
                   float ambient_temperature, float accel_factor,
                   float gravity, bool staggered, uint3 volume_size,
                   BlockArrangement* ba)
{
    if (vnp1_x != vn_x)
        CopyVolumeAsync(vnp1_x, vn_x, volume_size);

    if (vnp1_z != vn_z)
        CopyVolumeAsync(vnp1_z, vn_z, volume_size);

    if (BindCudaSurfaceToArray(&surf, vnp1_y) != hipSuccess)
        return;

    auto bound_v = BindHelper::Bind(&tex, vn_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_v.error() != hipSuccess)
        return;

    auto bound_t = BindHelper::Bind(&tex_t, temperature, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_t.error() != hipSuccess)
        return;

    auto bound_d = BindHelper::Bind(&tex_d, density, false,
                                    hipFilterModeLinear, hipAddressModeClamp);
    if (bound_d.error() != hipSuccess)
        return;

    if (staggered) {
        dim3 block(16, 1, 16);
        dim3 grid(volume_size.x / block.x, 1, volume_size.z / block.z);
        ba->ArrangeGrid(&grid, block, volume_size);
        grid.y = 1;
        ApplyBuoyancyStaggeredKernel<<<grid, block>>>(time_step,
                                                      ambient_temperature,
                                                      accel_factor, gravity,
                                                      volume_size);
    } else {
        dim3 block;
        dim3 grid;
        ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
        ApplyBuoyancyKernel<<<grid, block>>>(time_step, ambient_temperature,
                                             accel_factor, gravity,
                                             volume_size);
    }

    DCHECK_KERNEL();
}

void ComputeDivergence(hipArray* div, hipArray* vel_x, hipArray* vel_y,
                       hipArray* vel_z, float cell_size, bool outflow,
                       bool staggered, uint3 volume_size, BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, div) != hipSuccess)
        return;

    // A lazy way making selective dispatching. Hope not to hurt the
    // performance.
    auto bound = SelectiveBind(div, false, hipFilterModePoint,
                               hipAddressModeClamp, &tex, &texf, &texd);
    if (!bound.Succeeded())
        return;

    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);

    if (staggered) {
        InvokeKernel<ComputeDivergenceStaggeredKernelMeta>(
            bound, grid, block, cell_size, volume_size, outflow);
    } else {
        UpperBoundaryHandlerOutflow<float> outflow_handler;
        UpperBoundaryHandlerNeumann<float> neumann_handler;
        if (outflow) {
            ComputeDivergenceKernel<<<grid, block>>>(0.5f / cell_size,
                                                     volume_size,
                                                     outflow_handler);
        } else {
            ComputeDivergenceKernel<<<grid, block>>>(0.5f / cell_size,
                                                     volume_size,
                                                     neumann_handler);
        }
    }

    DCHECK_KERNEL();
}

void ComputeResidualDiagnosis(hipArray* residual, hipArray* u, hipArray* b,
                              float cell_size, uint3 volume_size,
                              BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, residual) != hipSuccess)
        return;

    auto bound_u = SelectiveBind(u, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex, &texf, &texd);
    if (!bound_u.Succeeded())
        return;

    auto bound_b = SelectiveBind(b, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_b, &texf_b,
                                 &texd_b);
    if (!bound_b.Succeeded())
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);

    InvokeKernel<ComputeResidualDiagnosisKernelMeta>(
        bound_u, grid, block, 1.0f / (cell_size * cell_size), volume_size);
    DCHECK_KERNEL();
}

void DecayVelocity(hipArray* vel_x, hipArray* vel_y, hipArray* vel_z,
                   float time_step, float velocity_dissipation,
                   const uint3& volume_size, BlockArrangement* ba)
{

    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_x, vel_x) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_y, vel_y) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_z, vel_z) != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeRowScan(&block, &grid, volume_size);
    DecayVelocityKernel<<<grid, block>>>(
        1.0f - velocity_dissipation * time_step, volume_size);

    DCHECK_KERNEL();
}

void RoundPassed(int* dest_array, int round, int x)
{
    RoundPassedKernel<<<1, 1>>>(dest_array, round, x);
    DCHECK_KERNEL();
}

void SubtractGradient(hipArray* vel_x, hipArray* vel_y, hipArray* vel_z,
                      hipArray* pressure, float cell_size, bool staggered,
                      uint3 volume_size, BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf_x, vel_x) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_y, vel_y) != hipSuccess)
        return;

    if (BindCudaSurfaceToArray(&surf_z, vel_z) != hipSuccess)
        return;

    auto bound_x = BindHelper::Bind(&tex_x, vel_x, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_x.error() != hipSuccess)
        return;

    auto bound_y = BindHelper::Bind(&tex_y, vel_y, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_y.error() != hipSuccess)
        return;

    auto bound_z = BindHelper::Bind(&tex_z, vel_z, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_z.error() != hipSuccess)
        return;

    auto bound = SelectiveBind(pressure, false, hipFilterModePoint,
                               hipAddressModeClamp, &tex, &texf, &texd);
    if (!bound.Succeeded())
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);

    if (staggered)
        InvokeKernel<SubtractGradientStaggeredKernelMeta>(bound, grid, block,
                                                          1.0f / cell_size,
                                                          volume_size);
    else
        SubtractGradientKernel<<<grid, block>>>(0.5f / cell_size, volume_size);

    DCHECK_KERNEL();
}
}