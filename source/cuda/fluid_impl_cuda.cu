#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"

surface<void, cudaSurfaceType3D> buoyancy_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> buoyancy_temperature;
surface<void, cudaSurfaceType3D> impulse_dest1;
surface<void, cudaSurfaceType3D> impulse_dest4;
surface<void, cudaSurfaceType3D> divergence_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> divergence_velocity;
surface<void, cudaSurfaceType3D> gradient_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> gradient_velocity;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> gradient_packed;
surface<void, cudaSurfaceType3D> diagnosis;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> diagnosis_source;

__global__ void ApplyBuoyancyKernel(float time_step, float ambient_temperature,
                                    float accel_factor, float gravity)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float4 velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);

    ushort4 result = make_ushort4(__float2half_rn(velocity.x),
                                  __float2half_rn(velocity.y),
                                  __float2half_rn(velocity.z),
                                  0);
    if (t > ambient_temperature) {
        float accel = time_step * ((t - ambient_temperature) * accel_factor -
                                   gravity);
        result.y = __float2half_rn(velocity.y + accel);
    }
    surf3Dwrite(result, buoyancy_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ApplyImpulse1Kernel(float3 center_point, float3 hotspot,
                                    float radius, float value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        surf3Dwrite(__float2half_rn(scale * value), impulse_dest1,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
        return;
    }
}

__global__ void ImpulseDensityKernel(float3 center_point, float radius,
                                     float value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        surf3Dwrite(__float2half_rn(value), impulse_dest1,
                    x * sizeof(ushort), y, z, hipBoundaryModeTrap);
    }
}

__global__ void ApplyImpulse3Kernel(float3 center_point, float3 hotspot,
                                    float radius, float3 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = 1 + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float2 diff = make_float2(coord.x, coord.z) -
        make_float2(center_point.x, center_point.z);
    float d = hypotf(diff.x, diff.y);
    if (d < radius) {
        diff = make_float2(coord.x, coord.z) -
            make_float2(hotspot.x, hotspot.z);
        float scale = (radius - hypotf(diff.x, diff.y)) / radius;
        scale = fmaxf(scale, 0.1f);
        ushort4 result = make_ushort4(__float2half_rn(scale * value.x),
                                      __float2half_rn(scale * value.y),
                                      __float2half_rn(scale * value.z),
                                      0);
        surf3Dwrite(result, impulse_dest4, x * sizeof(ushort4), y, z,
                    hipBoundaryModeTrap);
        return;
    }
}

__global__ void ComputeDivergenceKernel(float half_inverse_cell_size,
                                        uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float4 near =   tex3D(divergence_velocity, coord.x, coord.y, coord.z - 1.0f);
    float4 south =  tex3D(divergence_velocity, coord.x, coord.y - 1.0f, coord.z);
    float4 west =   tex3D(divergence_velocity, coord.x - 1.0f, coord.y, coord.z);
    float4 center = tex3D(divergence_velocity, coord.x, coord.y, coord.z);
    float4 east =   tex3D(divergence_velocity, coord.x + 1.0f, coord.y, coord.z);
    float4 north =  tex3D(divergence_velocity, coord.x, coord.y + 1.0f, coord.z);
    float4 far =    tex3D(divergence_velocity, coord.x, coord.y, coord.z + 1.0f);

    float diff_ew = east.x - west.x;
    float diff_ns = north.y - south.y;
    float diff_fn = far.z - near.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -center.x - west.x;

    if (x <= 0)
        diff_ew = east.x + center.x;

    if (y >= volume_size.y - 1)
        diff_ns = -center.y - south.y;

    if (y <= 0)
        diff_ns = north.y + center.y;

    if (z >= volume_size.z - 1)
        diff_fn = -center.z - near.z;

    if (z <= 0)
        diff_fn = far.z + center.z;

    float div = half_inverse_cell_size * (diff_ew + diff_ns + diff_fn);
    ushort2 result = make_ushort2(0, __float2half_rn(div));
    surf3Dwrite(result, divergence_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ComputeResidualPackedDiagnosisKernel(float inverse_h_square,
                                                     uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float  near =   tex3D(diagnosis_source, coord.x, coord.y, coord.z - 1.0f).x;
    float  south =  tex3D(diagnosis_source, coord.x, coord.y - 1.0f, coord.z).x;
    float  west =   tex3D(diagnosis_source, coord.x - 1.0f, coord.y, coord.z).x;
    float2 center = tex3D(diagnosis_source, coord.x, coord.y, coord.z);
    float  east =   tex3D(diagnosis_source, coord.x + 1.0f, coord.y, coord.z).x;
    float  north =  tex3D(diagnosis_source, coord.x, coord.y + 1.0f, coord.z).x;
    float  far =    tex3D(diagnosis_source, coord.x, coord.y, coord.z + 1.0f).x;
    float  b_center = center.y;

    if (coord.y == volume_size.y - 1)
        north = center.x;

    if (coord.y == 0)
        south = center.x;

    if (coord.x == volume_size.x - 1)
        east = center.x;

    if (coord.x == 0)
        west = center.x;

    if (coord.z == volume_size.z - 1)
        far = center.x;

    if (coord.z == 0)
        near = center.x;

    float v = b_center -
        (north + south + east + west + far + near - 6.0 * center.x) *
        inverse_h_square;
    surf3Dwrite(fabsf(v), diagnosis, x * sizeof(float), y, z,
                hipBoundaryModeTrap);
}

__global__ void RoundPassedKernel(int* dest_array, int round, int x)
{
    dest_array[0] = x * x - round * round;
}

__global__ void SubtractGradientKernel(float gradient_scale, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float near =   tex3D(gradient_packed, coord.x, coord.y, coord.z - 1.0f).x;
    float south =  tex3D(gradient_packed, coord.x, coord.y - 1.0f, coord.z).x;
    float west =   tex3D(gradient_packed, coord.x - 1.0f, coord.y, coord.z).x;
    float center = tex3D(gradient_packed, coord.x, coord.y, coord.z).x;
    float east =   tex3D(gradient_packed, coord.x + 1.0f, coord.y, coord.z).x;
    float north =  tex3D(gradient_packed, coord.x, coord.y + 1.0f, coord.z).x;
    float far =    tex3D(gradient_packed, coord.x, coord.y, coord.z + 1.0f).x;

    float diff_ew = east - west;
    float diff_ns = north - south;
    float diff_fn = far - near;

    // Handle boundary problem
    float3 mask = make_float3(1.0f, 1.0f, 1.0f);
    if (x >= volume_size.x - 1)
        mask.x = 0;

    if (x <= 0)
        mask.x = 0;

    if (y >= volume_size.y - 1)
        mask.y = 0;

    if (y <= 0)
        mask.y = 0;

    if (z >= volume_size.z - 1)
        mask.z = 0;

    if (z <= 0)
        mask.z = 0;

    float3 old_v =
        make_float3(tex3D(gradient_velocity, coord.x, coord.y, coord.z));
    float3 grad = make_float3(diff_ew, diff_ns, diff_fn) * gradient_scale;
    float3 new_v = old_v - grad;
    float3 result = mask * new_v; // Velocity goes to 0 when hit ???
    ushort4 raw = make_ushort4(__float2half_rn(result.x),
                               __float2half_rn(result.y),
                               __float2half_rn(result.z),
                               0);
    surf3Dwrite(raw, gradient_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchApplyBuoyancy(hipArray* dest_array, hipArray* velocity_array,
                         hipArray* temperature_array, float time_step,
                         float ambient_temperature, float accel_factor,
                         float gravity, uint3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&buoyancy_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    buoyancy_velocity.normalized = false;
    buoyancy_velocity.filterMode = hipFilterModePoint;
    buoyancy_velocity.addressMode[0] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[1] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[2] = hipAddressModeClamp;
    buoyancy_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&buoyancy_velocity, velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, temperature_array);
    buoyancy_temperature.normalized = false;
    buoyancy_temperature.filterMode = hipFilterModePoint;
    buoyancy_temperature.addressMode[0] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[1] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[2] = hipAddressModeClamp;
    buoyancy_temperature.channelDesc = desc;

    result = hipBindTextureToArray(&buoyancy_temperature,
                                    temperature_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ApplyBuoyancyKernel<<<grid, block>>>(time_step, ambient_temperature,
                                         accel_factor, gravity);

    hipUnbindTexture(&buoyancy_temperature);
    hipUnbindTexture(&buoyancy_velocity);
}

void LaunchApplyImpulse(hipArray* dest_array, hipArray* original_array,
                        float3 center_point, float3 hotspot, float radius,
                        float3 value, uint32_t mask, uint3 volume_size)
{
    assert(mask == 1 || mask == 7);
    if (mask != 1 && mask != 7)
        return;

    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    dim3 block(128, 2, 1);
    dim3 grid(volume_size.x / block.x, 1, volume_size.z / block.z);
    if (mask == 1) {
        hipError_t result = cudaBindSurfaceToArray(&impulse_dest1, dest_array,
                                                    &desc);
        assert(result == hipSuccess);
        if (result != hipSuccess)
            return;

        ApplyImpulse1Kernel<<<grid, block>>>(center_point, hotspot, radius,
                                             value.x);
    } else if (mask == 7) {
        hipError_t result = cudaBindSurfaceToArray(&impulse_dest4, dest_array,
                                                    &desc);
        assert(result == hipSuccess);
        if (result != hipSuccess)
            return;

        ApplyImpulse3Kernel<<<grid, block>>>(center_point, hotspot, radius,
                                             value);
    }
}

void LaunchComputeDivergence(hipArray* dest_array, hipArray* velocity_array,
                             float half_inverse_cell_size, uint3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&divergence_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, velocity_array);
    divergence_velocity.normalized = false;
    divergence_velocity.filterMode = hipFilterModePoint;
    divergence_velocity.addressMode[0] = hipAddressModeClamp;
    divergence_velocity.addressMode[1] = hipAddressModeClamp;
    divergence_velocity.addressMode[2] = hipAddressModeClamp;
    divergence_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&divergence_velocity, velocity_array,
                                    &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeDivergenceKernel<<<grid, block>>>(half_inverse_cell_size,
                                             volume_size);

    hipUnbindTexture(&divergence_velocity);
}

void LaunchComputeResidualPackedDiagnosis(hipArray* dest_array,
                                          hipArray* source_array,
                                          float inverse_h_square,
                                          uint3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&diagnosis, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    diagnosis_source.normalized = false;
    diagnosis_source.filterMode = hipFilterModePoint;
    diagnosis_source.addressMode[0] = hipAddressModeClamp;
    diagnosis_source.addressMode[1] = hipAddressModeClamp;
    diagnosis_source.addressMode[2] = hipAddressModeClamp;
    diagnosis_source.channelDesc = desc;

    result = hipBindTextureToArray(&diagnosis_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeResidualPackedDiagnosisKernel<<<grid, block>>>(inverse_h_square,
                                                          volume_size);

    hipUnbindTexture(&diagnosis_source);
}

void LaunchImpulseDensity(hipArray* dest_array, hipArray* original_array,
                          float3 center_point, float radius, float3 value,
                          uint3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    dim3 block(128, 2, 1);
    dim3 grid(volume_size.x / block.x, 1, volume_size.z / block.z);

    hipError_t result = cudaBindSurfaceToArray(&impulse_dest1, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    ImpulseDensityKernel<<<grid, block>>>(center_point, radius, value.x);
}

void LaunchRoundPassed(int* dest_array, int round, int x)
{
    RoundPassedKernel<<<1, 1>>>(dest_array, round, x);
}

void LaunchSubtractGradient(hipArray* dest_array, hipArray* packed_array,
                            float gradient_scale, uint3 volume_size,
                            BlockArrangement* ba)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&gradient_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, dest_array);
    gradient_velocity.normalized = false;
    gradient_velocity.filterMode = hipFilterModePoint;
    gradient_velocity.addressMode[0] = hipAddressModeClamp;
    gradient_velocity.addressMode[1] = hipAddressModeClamp;
    gradient_velocity.addressMode[2] = hipAddressModeClamp;
    gradient_velocity.channelDesc = desc;

    // Reading as texture would be more efficient. Hardware half-float
    // conversion?
    result = hipBindTextureToArray(&gradient_velocity, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, packed_array);
    gradient_packed.normalized = false;
    gradient_packed.filterMode = hipFilterModePoint;
    gradient_packed.addressMode[0] = hipAddressModeClamp;
    gradient_packed.addressMode[1] = hipAddressModeClamp;
    gradient_packed.addressMode[2] = hipAddressModeClamp;
    gradient_packed.channelDesc = desc;

    result = hipBindTextureToArray(&gradient_packed, packed_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    SubtractGradientKernel<<<grid, block>>>(gradient_scale, volume_size);

    hipUnbindTexture(&gradient_packed);
    hipUnbindTexture(&gradient_velocity);
}
