#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/advection_method.h"
#include "cuda/block_arrangement.h"
#include "cuda/field_offset.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_aux;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vx;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vy;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vz;

__device__ float3 GetVelocityStaggeredOffset(float3 pos)
{
    float v_x = tex3D(tex_vx, pos.x + 0.5f, pos.y,        pos.z       );
    float v_y = tex3D(tex_vy, pos.x,        pos.y + 0.5f, pos.z       );
    float v_z = tex3D(tex_vz, pos.x,        pos.y,        pos.z + 0.5f);
    return make_float3(v_x, v_y, v_z);
}

// =============================================================================

template <bool MidPoint>
__device__ inline float3 AdvectImpl(float3 vel, float3 pos, float time_step_over_cell_size)
{
    return pos - vel * time_step_over_cell_size;
}

template <>
__device__ inline float3 AdvectImpl<true>(float3 vel, float3 pos, float time_step_over_cell_size)
{
    float3 mid_point = pos - vel * 0.5f * time_step_over_cell_size;
    vel = GetVelocityStaggeredOffset(mid_point);
    return pos - vel * time_step_over_cell_size;
}

template <bool MidPoint>
__global__ void AdvectFieldBfeccStaggeredOffsetKernel(float3 offset, float time_step_over_cell_size, float dissipation, uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step_over_cell_size);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_new = tex3D(tex_aux, back_traced.x, back_traced.y, back_traced.z);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);

    auto r = __float2half_rn(dissipation * ��_new);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldMacCormackStaggeredOffsetKernel(float3 offset, float time_step_over_cell_size, float dissipation, uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step_over_cell_size);

    float ��_n = tex3D(tex, coord.x, coord.y, coord.z);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_np1_hat = tex3D(tex_aux, coord.x, coord.y, coord.z);

    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord, -time_step_over_cell_size);
    float ��_n_hat = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);

    float ��_new = ��_np1_hat + 0.5f * (��_n - ��_n_hat);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new)
        ��_new = ��_np1_hat;

    auto r = __float2half_rn(��_new * dissipation);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldSemiLagrangianStaggeredOffsetKernel(
    float3 offset, float time_step_over_cell_size, float dissipation,
    uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord,
                                              time_step_over_cell_size);
    float �� = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);
    auto r = __float2half_rn(�� * dissipation);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void BfeccRemoveErrorStaggeredOffsetKernel(
    float3 offset, float time_step_over_cell_size, uint3 volume_size)
{
    int x = VolumeX();
    int y = VolumeY();
    int z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord,
                                                -time_step_over_cell_size);

    float �� = tex3D(tex, coord.x, coord.y, coord.z);
    float r = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);
    r = 0.5f * (3.0f * �� - r);
    surf3Dwrite(__float2half_rn(r), surf, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

void AdvectFieldsBfeccStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                      float3* offset, int num_of_fields,
                                      hipArray* vel_x, hipArray* vel_y,
                                      hipArray* vel_z, hipArray* aux,
                                      float cell_size, float time_step,
                                      float dissipation, uint3 volume_size,
                                      bool mid_point, BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        // Pass 1: Calculate ��_n_plus_1_hat, and store in |fnp1[i]|.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step / cell_size, 1.0f, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step / cell_size, 1.0f, volume_size);

        // Pass 2: Calculate ��_n_hat, and store in |aux|.
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        {
            auto bound_a = BindHelper::Bind(&tex_aux, fnp1[i], false,
                                            hipFilterModeLinear,
                                            hipAddressModeClamp);
            if (bound_a.error() != hipSuccess)
                return;

            if (mid_point)
                BfeccRemoveErrorStaggeredOffsetKernel<true><<<grid, block>>>(
                    offset[i], time_step / cell_size, volume_size);
            else
                BfeccRemoveErrorStaggeredOffsetKernel<false><<<grid, block>>>(
                    offset[i], time_step / cell_size, volume_size);
        }

        // Pass 3: Calculate the final result.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound_a = BindHelper::Bind(&tex_aux, aux, false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp);
        if (bound_a.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldBfeccStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
        else
            AdvectFieldBfeccStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
    }
}

void AdvectFieldsMacCormackStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                           float3* offset, int num_of_fields,
                                           hipArray* vel_x, hipArray* vel_y,
                                           hipArray* vel_z, hipArray* aux,
                                           float cell_size, float time_step,
                                           float dissipation, uint3 volume_size,
                                           bool mid_point, BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    auto bound_a = BindHelper::Bind(&tex_aux, aux, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_a.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step / cell_size, 1.0f, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step / cell_size, 1.0f, volume_size);

        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldMacCormackStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
        else
            AdvectFieldMacCormackStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
    }
}

void AdvectFieldsSemiLagrangianStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                               float3* offset,
                                               int num_of_fields,
                                               hipArray* vel_x,
                                               hipArray* vel_y,
                                               hipArray* vel_z,
                                               float cell_size, float time_step,
                                               float dissipation,
                                               uint3 volume_size,
                                               bool mid_point,
                                               BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step / cell_size,
                1.0f - dissipation * time_step, volume_size);
    }
}

namespace kern_launcher
{
void AdvectScalarFieldStaggered(hipArray* fnp1, hipArray* fn,
                                hipArray* vel_x, hipArray* vel_y,
                                hipArray* vel_z, hipArray* aux,
                                float cell_size, float time_step,
                                float dissipation, AdvectionMethod method,
                                uint3 volume_size, bool mid_point,
                                BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1};
    hipArray* fns[] = {fn};
    float3 offsets[] = {make_float3(0.0f)};
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, cell_size, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, cell_size,
                                         time_step, dissipation, volume_size,
                                         mid_point, ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, cell_size, time_step,
                                                  dissipation, volume_size,
                                                  mid_point, ba);
    }
    DCHECK_KERNEL();
}

void AdvectVelocityStaggered(hipArray* fnp1_x, hipArray* fnp1_y,
                             hipArray* fnp1_z, hipArray* fn_x,
                             hipArray* fn_y, hipArray* fn_z, hipArray* vel_x,
                             hipArray* vel_y, hipArray* vel_z, hipArray* aux,
                             float cell_size, float time_step,
                             float dissipation, AdvectionMethod method,
                             uint3 volume_size, bool mid_point,
                             BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1_x, fnp1_y, fnp1_z};
    hipArray* fns[] = {fn_x, fn_y, fn_z};
    float3 offsets[] = {
        -GetOffsetVelocityField(0),
        -GetOffsetVelocityField(1),
        -GetOffsetVelocityField(2)
    };
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, cell_size, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, cell_size,
                                         time_step, dissipation, volume_size,
                                         mid_point, ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, cell_size, time_step,
                                                  dissipation, volume_size,
                                                  mid_point, ba);
    }
    DCHECK_KERNEL();
}

void AdvectVorticityStaggered(hipArray* fnp1_x, hipArray* fnp1_y,
                              hipArray* fnp1_z, hipArray* fn_x,
                              hipArray* fn_y, hipArray* fn_z,
                              hipArray* vel_x, hipArray* vel_y,
                              hipArray* vel_z, hipArray* aux, float cell_size,
                              float time_step,  float dissipation,
                              AdvectionMethod method, uint3 volume_size,
                              bool mid_point, BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1_x, fnp1_y, fnp1_z};
    hipArray* fns[] = {fn_x, fn_y, fn_z};
    float3 offsets[] = {
        -GetOffsetVorticityField(0),
        -GetOffsetVorticityField(1),
        -GetOffsetVorticityField(2)
    };
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, cell_size, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, cell_size,
                                         time_step, dissipation, volume_size,
                                         mid_point, ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, cell_size, time_step,
                                                  dissipation, volume_size,
                                                  mid_point, ba);
    }

    DCHECK_KERNEL();
}
}