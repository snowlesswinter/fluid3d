#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "advection_method.h"
#include "block_arrangement.h"
#include "field_offset.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> surf;
surface<void, cudaSurfaceType3D> surf_x;
surface<void, cudaSurfaceType3D> surf_y;
surface<void, cudaSurfaceType3D> surf_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_x;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_y;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_z;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_aux;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vx;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vy;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_vz;

__device__ float3 GetVelocityStaggeredOffset(float3 pos)
{
    float v_x = tex3D(tex_vx, pos.x + 0.5f, pos.y,        pos.z       );
    float v_y = tex3D(tex_vy, pos.x,        pos.y + 0.5f, pos.z       );
    float v_z = tex3D(tex_vz, pos.x,        pos.y,        pos.z + 0.5f);
    return make_float3(v_x, v_y, v_z);
}

// =============================================================================

template <bool MidPoint>
__device__ inline float3 AdvectImpl(float3 vel, float3 pos, float time_step)
{
    return pos - vel * time_step;
}

template <>
__device__ inline float3 AdvectImpl<true>(float3 vel, float3 pos, float time_step)
{
    float3 mid_point = pos - vel * 0.5f * time_step;
    vel = GetVelocityStaggeredOffset(mid_point);
    return pos - vel * time_step;
}

template <bool MidPoint>
__global__ void AdvectFieldBfeccStaggeredOffsetKernel(float3 offset,
                                                      float time_step,
                                                      float dissipation,
                                                      uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_new = tex3D(tex_aux, back_traced.x, back_traced.y, back_traced.z);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);

    auto r = __float2half_rn((1.0f - dissipation * time_step) * ��_new);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldMacCormackStaggeredOffsetKernel(float3 offset,
                                                           float time_step,
                                                           float dissipation,
                                                           uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step);

    float ��_n = tex3D(tex, coord.x, coord.y, coord.z);

    float ��0 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(tex, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(tex, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(tex, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(tex, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_np1_hat = tex3D(tex_aux, coord.x, coord.y, coord.z);

    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord, -time_step);
    float ��_n_hat = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);

    float ��_new = ��_np1_hat + 0.5f * (��_n - ��_n_hat);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new)
        ��_new = ��_np1_hat;

    ��_new *= (1.0f - dissipation * time_step);
    auto r = __float2half_rn(��_new);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void AdvectFieldSemiLagrangianStaggeredOffsetKernel(
    float3 offset, float time_step, float dissipation, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;

    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 back_traced = AdvectImpl<MidPoint>(vel, coord, time_step);
    float �� = tex3D(tex, back_traced.x, back_traced.y, back_traced.z);
    �� *= (1.0f - dissipation * time_step);
    auto r = __float2half_rn(��);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <bool MidPoint>
__global__ void BfeccRemoveErrorStaggeredOffsetKernel(float3 offset,
                                                      float time_step,
                                                      uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 vel = GetVelocityStaggeredOffset(coord + offset);
    float3 forward_trace = AdvectImpl<MidPoint>(vel, coord, -time_step);

    float �� = tex3D(tex, coord.x, coord.y, coord.z);
    float r = tex3D(tex_aux, forward_trace.x, forward_trace.y, forward_trace.z);
    r = 0.5f * (3.0f * �� - r);
    surf3Dwrite(__float2half_rn(r), surf, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

void AdvectFieldsBfeccStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                      float3* offset, int num_of_fields,
                                      hipArray* vel_x, hipArray* vel_y,
                                      hipArray* vel_z, hipArray* aux,
                                      float time_step, float dissipation,
                                      uint3 volume_size, bool mid_point,
                                      BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        // Pass 1: Calculate ��_n_plus_1_hat, and store in |fnp1[i]|.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);

        // Pass 2: Calculate ��_n_hat, and store in |aux|.
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        {
            auto bound_a = BindHelper::Bind(&tex_aux, fnp1[i], false,
                                            hipFilterModeLinear,
                                            hipAddressModeClamp);
            if (bound_a.error() != hipSuccess)
                return;

            if (mid_point)
                BfeccRemoveErrorStaggeredOffsetKernel<true><<<grid, block>>>(
                    offset[i], time_step, volume_size);
            else
                BfeccRemoveErrorStaggeredOffsetKernel<false><<<grid, block>>>(
                    offset[i], time_step, volume_size);
        }

        // Pass 3: Calculate the final result.
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound_a = BindHelper::Bind(&tex_aux, aux, false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp);
        if (bound_a.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldBfeccStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step, dissipation, volume_size);
        else
            AdvectFieldBfeccStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step, dissipation, volume_size);
    }
}

void AdvectFieldsMacCormackStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                           float3* offset, int num_of_fields,
                                           hipArray* vel_x, hipArray* vel_y,
                                           hipArray* vel_z, hipArray* aux,
                                           float time_step, float dissipation,
                                           uint3 volume_size, bool mid_point,
                                           BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    auto bound_a = BindHelper::Bind(&tex_aux, aux, false, hipFilterModeLinear,
                                    hipAddressModeClamp);
    if (bound_a.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);

        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldMacCormackStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step, dissipation, volume_size);
        else
            AdvectFieldMacCormackStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step, dissipation, volume_size);
    }
}

void AdvectFieldsSemiLagrangianStaggeredOffset(hipArray** fnp1, hipArray** fn,
                                               float3* offset,
                                               int num_of_fields,
                                               hipArray* vel_x,
                                               hipArray* vel_y,
                                               hipArray* vel_z,
                                               float time_step,
                                               float dissipation,
                                               uint3 volume_size,
                                               bool mid_point,
                                               BlockArrangement* ba)
{
    auto bound_vx = BindHelper::Bind(&tex_vx, vel_x, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vx.error() != hipSuccess)
        return;

    auto bound_vy = BindHelper::Bind(&tex_vy, vel_y, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vy.error() != hipSuccess)
        return;

    auto bound_vz = BindHelper::Bind(&tex_vz, vel_z, false,
                                     hipFilterModeLinear,
                                     hipAddressModeClamp);
    if (bound_vz.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangePrefer3dLocality(&block, &grid, volume_size);
    for (int i = 0; i < num_of_fields; i++) {
        if (BindCudaSurfaceToArray(&surf, fnp1[i]) != hipSuccess)
            return;

        auto bound = BindHelper::Bind(&tex, fn[i], false, hipFilterModeLinear,
                                      hipAddressModeClamp);
        if (bound.error() != hipSuccess)
            return;

        if (mid_point)
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<true><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);
        else
            AdvectFieldSemiLagrangianStaggeredOffsetKernel<false><<<grid, block>>>(
                offset[i], time_step, 0.0f, volume_size);
    }
}

void LaunchAdvectScalarFieldStaggered(hipArray* fnp1, hipArray* fn,
                                      hipArray* vel_x, hipArray* vel_y,
                                      hipArray* vel_z, hipArray* aux,
                                      float time_step, float dissipation,
                                      AdvectionMethod method,
                                      uint3 volume_size, bool mid_point,
                                      BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1};
    hipArray* fns[] = {fn};
    float3 offsets[] = {make_float3(0.0f)};
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, time_step,
                                         dissipation, volume_size, mid_point,
                                         ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, time_step, dissipation,
                                                  volume_size, mid_point, ba);
    }
}

void LaunchAdvectVelocityStaggered(hipArray* fnp1_x, hipArray* fnp1_y,
                                   hipArray* fnp1_z, hipArray* fn_x,
                                   hipArray* fn_y, hipArray* fn_z,
                                   hipArray* vel_x, hipArray* vel_y,
                                   hipArray* vel_z, hipArray* aux,
                                   float time_step, float dissipation,
                                   AdvectionMethod method,
                                   uint3 volume_size, bool mid_point,
                                   BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1_x, fnp1_y, fnp1_z};
    hipArray* fns[] = {fn_x, fn_y, fn_z};
    float3 offsets[] = {
        -GetOffsetVelocityField(0),
        -GetOffsetVelocityField(1),
        -GetOffsetVelocityField(2)
    };
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, time_step,
                                         dissipation, volume_size, mid_point,
                                         ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, time_step, dissipation,
                                                  volume_size, mid_point, ba);
    }
}

void LaunchAdvectVorticityStaggered(hipArray* fnp1_x, hipArray* fnp1_y,
                                    hipArray* fnp1_z, hipArray* fn_x,
                                    hipArray* fn_y, hipArray* fn_z,
                                    hipArray* vel_x, hipArray* vel_y,
                                    hipArray* vel_z, hipArray* aux,
                                    float time_step, float dissipation,
                                    AdvectionMethod method,
                                    uint3 volume_size, bool mid_point,
                                    BlockArrangement* ba)
{
    hipArray* fnp1s[] = {fnp1_x, fnp1_y, fnp1_z};
    hipArray* fns[] = {fn_x, fn_y, fn_z};
    float3 offsets[] = {
        -GetOffsetVorticityField(0),
        -GetOffsetVorticityField(1),
        -GetOffsetVorticityField(2)
    };
    int num_of_fields = sizeof(fnp1s) / sizeof(fnp1s[0]);
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        AdvectFieldsMacCormackStaggeredOffset(fnp1s, fns, offsets,
                                              num_of_fields, vel_x, vel_y,
                                              vel_z, aux, time_step,
                                              dissipation, volume_size,
                                              mid_point, ba);
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        AdvectFieldsBfeccStaggeredOffset(fnp1s, fns, offsets, num_of_fields,
                                         vel_x, vel_y, vel_z, aux, time_step,
                                         dissipation, volume_size, mid_point,
                                         ba);
    } else {
        AdvectFieldsSemiLagrangianStaggeredOffset(fnp1s, fns, offsets,
                                                  num_of_fields, vel_x, vel_y,
                                                  vel_z, time_step, dissipation,
                                                  volume_size, mid_point, ba);
    }
}
