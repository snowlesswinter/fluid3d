#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "advection_method.h"
#include "block_arrangement.h"
#include "cuda_common.h"

surface<void, cudaSurfaceType3D> advect_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_velocity;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_intermediate;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_intermediate1;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_source;

__global__ void AdvectScalarBfeccStaggeredKernel(float time_step,
                                                 float dissipation,
                                                 bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float ��0 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_new = tex3D(advect_intermediate1, back_traced.x, back_traced.y, back_traced.z);
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = tex3D(advect_source, back_traced.x, back_traced.y,
                      back_traced.z);

    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��_new)) * ��_new :
        (1.0f - dissipation * time_step) * ��_new;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectScalarBfeccRemoveErrorStaggeredKernel(float time_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced = coord - time_step * make_float3(velocity);

    float �� = tex3D(advect_source, coord.x, coord.y, coord.z);
    float result = tex3D(advect_intermediate1, back_traced.x, back_traced.y,
                         back_traced.z);
    result = 0.5f * (3.0f * �� - result);
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectScalarMacCormackStaggeredKernel(float time_step, float dissipation,
                                       bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;
    float �� = tex3D(advect_source, coord.x, coord.y, coord.z);

    float ��0 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��1 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��2 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��3 = tex3D(advect_source, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);
    float ��4 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f);
    float ��5 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f);
    float ��6 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f);
    float ��7 = tex3D(advect_source, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f);

    float ��_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);
    float ��_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(��0, ��1), ��2), ��3), ��4), ��5), ��6), ��7);

    float ��_n_plus_1_hat = tex3D(advect_intermediate1, coord.x, coord.y, coord.z);
    float3 forward_traced = coord + time_step * velocity;
    float ��_n_hat = tex3D(advect_intermediate1, forward_traced.x, forward_traced.y, forward_traced.z);

    float ��_new = (��_n_plus_1_hat + 0.5f * (�� - ��_n_hat));
    float clamped = fmaxf(fminf(��_new, ��_max), ��_min);
    if (clamped != ��_new) // New extrema found, revert to the first order
                          // accurate semi-Lagrangian method.
        ��_new = ��_n_plus_1_hat;

    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��_new)) * ��_new :
        (1.0f - dissipation * time_step) * ��_new;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectScalarSemiLagrangianStaggeredKernel(float time_step, float dissipation,
                                           bool quadratic_dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced = coord - time_step * make_float3(velocity);

    float �� = tex3D(advect_source, back_traced.x, back_traced.y, back_traced.z);
    float result = quadratic_dissipation ?
        (1.0f - dissipation * time_step * (1.0f - ��)) * �� :
        (1.0f - dissipation * time_step) * ��;
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectVelocityBfeccStaggeredKernel(float time_step, float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 v_n = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * v_n;

    float3 v0 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v1 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v2 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v3 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));
    float3 v4 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v5 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v6 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v7 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));

    float3 v_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0, v1), v2), v3), v4), v5), v6), v7);
    float3 v_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0, v1), v2), v3), v4), v5), v6), v7);

    float3 v_new = make_float3(tex3D(advect_intermediate, back_traced.x, back_traced.y, back_traced.z));
    float3 clamped = fmaxf(fminf(v_new, v_max), v_min);
    if (clamped.x != v_new.x || clamped.y != v_new.y || clamped.z != v_new.z)
        v_new = make_float3(tex3D(advect_velocity, back_traced.x, back_traced.y, back_traced.z));

    v_new = (1.0f - dissipation * time_step) * v_new;
    ushort4 result = make_ushort4(__float2half_rn(v_new.x), __float2half_rn(v_new.y), __float2half_rn(v_new.z), 0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectVelocityBfeccRemoveErrorStaggeredKernel(float time_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity =
        make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity =
        make_float3(
            tex3D(advect_intermediate, back_traced.x, back_traced.y,
                  back_traced.z));
    new_velocity = 0.5f * (3.0f * velocity - new_velocity);
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void AdvectVelocityMacCormackStaggeredKernel(float time_step,
                                               float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 v_n = make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * v_n;

    float3 v0 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v1 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v2 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v3 = make_float3(tex3D(advect_velocity, back_traced.x - 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));
    float3 v4 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z - 0.5f));
    float3 v5 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y - 0.5f, back_traced.z + 0.5f));
    float3 v6 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z - 0.5f));
    float3 v7 = make_float3(tex3D(advect_velocity, back_traced.x + 0.5f, back_traced.y + 0.5f, back_traced.z + 0.5f));

    float3 v_min = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0, v1), v2), v3), v4), v5), v6), v7);
    float3 v_max = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0, v1), v2), v3), v4), v5), v6), v7);

    float3 v_n_plus_1_hat = make_float3(tex3D(advect_intermediate, coord.x, coord.y, coord.z));
    float3 forward_trace = coord + time_step * v_n;
    float3 v_n_hat = make_float3(tex3D(advect_intermediate, forward_trace.x, forward_trace.y, forward_trace.z));

    float3 v_new = (v_n_plus_1_hat + 0.5f * (v_n - v_n_hat));
    float3 clamped = fmaxf(fminf(v_new, v_max), v_min);
    if (clamped.x != v_new.x || clamped.y != v_new.y || clamped.z != v_new.z)
        v_new = v_n_plus_1_hat;

    v_new = (1.0f - dissipation * time_step) * v_new;
    ushort4 result = make_ushort4(__float2half_rn(v_new.x), __float2half_rn(v_new.y), __float2half_rn(v_new.z), 0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z, hipBoundaryModeTrap);
}

__global__ void AdvectVelocitySemiLagrangianStaggeredKernel(float time_step,
                                                   float dissipation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z) + 0.5f;
    float3 velocity =
        make_float3(tex3D(advect_velocity, coord.x, coord.y, coord.z));
    float3 back_traced = coord - time_step * velocity;

    float3 new_velocity =
        (1.0f - dissipation * time_step) * 
            make_float3(
                tex3D(advect_velocity, back_traced.x, back_traced.y,
                      back_traced.z));
    ushort4 result = make_ushort4(__float2half_rn(new_velocity.x),
                                  __float2half_rn(new_velocity.y),
                                  __float2half_rn(new_velocity.z),
                                  0);
    surf3Dwrite(result, advect_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchAdvectScalarBfeccStaggered(hipArray_t dest_array, hipArray_t velocity_array,
                       hipArray_t source_array, hipArray_t intermediate_array,
                       float time_step, float dissipation,
                       bool quadratic_dissipation, uint3 volume_size)
{
    // Pass 1: Calculate ��_n_plus_1_hat, and store in |dest_array|.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array,
                                      false, hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array,
                                         false, hipFilterModeLinear);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianStaggeredKernel << <grid, block >> >(time_step, 0.0f,
                                                quadratic_dissipation);

    // Pass 2: Calculate ��_n_hat, and store in |intermediate_array|.
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_intermediate1 = BindHelper::Bind(&advect_intermediate1,
                                                dest_array, false,
                                                hipFilterModeLinear);
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarBfeccRemoveErrorStaggeredKernel << <grid, block >> >(-time_step);

    // Pass 3: Calculate the final result.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    bound_intermediate1.Take(
        BindHelper::Bind(&advect_intermediate1, intermediate_array, false,
                         hipFilterModeLinear));
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarBfeccStaggeredKernel << <grid, block >> >(time_step, dissipation,
                                       quadratic_dissipation);
}

void LaunchAdvectScalarMacCormackStaggered(hipArray_t dest_array, hipArray_t velocity_array,
                            hipArray_t source_array,
                            hipArray_t intermediate_array, float time_step,
                            float dissipation, bool quadratic_dissipation,
                            uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array, false,
                                      hipFilterModeLinear);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianStaggeredKernel << <grid, block >> >(time_step, 0.0f,
                                                quadratic_dissipation);

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_intermediate1 = BindHelper::Bind(&advect_intermediate1,
                                                intermediate_array, false,
                                                hipFilterModeLinear);
    if (bound_intermediate1.error() != hipSuccess)
        return;

    AdvectScalarMacCormackStaggeredKernel << <grid, block >> >(time_step, dissipation,
                                            quadratic_dissipation);
}

void LaunchAdvectScalarStaggered(hipArray_t dest_array, hipArray_t velocity_array,
                  hipArray_t source_array, hipArray_t intermediate_array,
                  float time_step, float dissipation,
                  bool quadratic_dissipation, uint3 volume_size,
                  AdvectionMethod method)
{
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        LaunchAdvectScalarMacCormackStaggered(dest_array, velocity_array, source_array,
                               intermediate_array, time_step, dissipation,
                               false, volume_size);
        return;
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        LaunchAdvectScalarBfeccStaggered(dest_array, velocity_array, source_array,
                          intermediate_array, time_step, dissipation, false,
                          volume_size);
        return;
    }

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    auto bound_source = BindHelper::Bind(&advect_source, source_array, false,
                                         hipFilterModeLinear);
    if (bound_source.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectScalarSemiLagrangianStaggeredKernel << <grid, block >> >(time_step, dissipation,
                                                quadratic_dissipation);
}

void LaunchAdvectVelocityBfeccStaggered(hipArray_t dest_array,
                               hipArray_t velocity_array,
                               hipArray_t intermediate_array, float time_step,
                               float time_step_prev, float dissipation,
                               uint3 volume_size)
{
    // Pass 1: Calculate ��_n_plus_1_hat, and store in |dest_array|.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianStaggeredKernel << <grid, block >> >(time_step, 0.0f);

    // Pass 2: Calculate ��_n_hat, and store in |intermediate_array|.
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_intermediate = BindHelper::Bind(&advect_intermediate, dest_array,
                                               false, hipFilterModeLinear);
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityBfeccRemoveErrorStaggeredKernel << <grid, block >> >(-time_step);

    // Pass 3: Calculate the final result.
    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    bound_intermediate.Take(
        BindHelper::Bind(&advect_intermediate, intermediate_array, false,
                         hipFilterModeLinear));
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityBfeccStaggeredKernel << <grid, block >> >(time_step, dissipation);
}

void LaunchAdvectVelocityMacCormackStaggered(hipArray_t dest_array,
                                    hipArray_t velocity_array,
                                    hipArray_t intermediate_array,
                                    float time_step, float time_step_prev,
                                    float dissipation, uint3 volume_size)
{
    if (BindCudaSurfaceToArray(&advect_dest, intermediate_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianStaggeredKernel << <grid, block >> >(time_step, 0.0f);

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_intermediate = BindHelper::Bind(&advect_intermediate,
                                               intermediate_array, false,
                                               hipFilterModeLinear);
    if (bound_intermediate.error() != hipSuccess)
        return;

    AdvectVelocityMacCormackStaggeredKernel << <grid, block >> >(time_step, dissipation);
}

void LaunchAdvectVelocityStaggered(hipArray_t dest_array, hipArray_t velocity_array,
                          hipArray_t intermediate_array, float time_step,
                          float time_step_prev, float dissipation,
                          uint3 volume_size, AdvectionMethod method)
{
    if (method == MACCORMACK_SEMI_LAGRANGIAN) {
        LaunchAdvectVelocityMacCormackStaggered(dest_array, velocity_array,
                                       intermediate_array, time_step,
                                       time_step_prev, dissipation,
                                       volume_size);
        return;
    } else if (method == BFECC_SEMI_LAGRANGIAN) {
        LaunchAdvectVelocityBfeccStaggered(dest_array, velocity_array,
                                  intermediate_array, time_step, time_step_prev,
                                  dissipation, volume_size);
        return;
    }

    if (BindCudaSurfaceToArray(&advect_dest, dest_array) != hipSuccess)
        return;

    auto bound_vel = BindHelper::Bind(&advect_velocity, velocity_array, false,
                                      hipFilterModeLinear);
    if (bound_vel.error() != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    AdvectVelocitySemiLagrangianStaggeredKernel << <grid, block >> >(time_step,
                                                        dissipation);
}
