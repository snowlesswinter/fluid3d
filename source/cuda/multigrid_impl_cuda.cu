#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

surface<void, hipTextureType3D> residual_dest;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> residual_source;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_coarse;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_fine;
surface<void, hipTextureType3D> prolongate_pure_dest;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> prolongate_pure_coarse;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> prolongate_pure_fine;
surface<void, hipTextureType3D> guess_dest;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> guess_source;
surface<void, hipTextureType3D> restrict_residual_dest;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> restrict_residual_source;
surface<void, hipTextureType3D> restrict_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> restrict_source;

__global__ void ComputeResidualPackedPureKernel(float inverse_h_square,
                                                int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float  near =   tex3D(residual_source, coord.x, coord.y, coord.z - 1.0f).x;
    float  south =  tex3D(residual_source, coord.x, coord.y - 1.0f, coord.z).x;
    float  west =   tex3D(residual_source, coord.x - 1.0f, coord.y, coord.z).x;
    float2 center = tex3D(residual_source, coord.x, coord.y, coord.z);
    float  east =   tex3D(residual_source, coord.x + 1.0f, coord.y, coord.z).x;
    float  north =  tex3D(residual_source, coord.x, coord.y + 1.0f, coord.z).x;
    float  far =    tex3D(residual_source, coord.x, coord.y, coord.z + 1.0f).x;
    float  b_center = center.y;

    if (coord.y == volume_size.y - 1)
        north = center.x;

    if (coord.y == 0)
        south = center.x;

    if (coord.x == volume_size.x - 1)
        east = center.x;

    if (coord.x == 0)
        west = center.x;

    if (coord.z == volume_size.z - 1)
        far = center.x;

    if (coord.z == 0)
        near = center.x;

    float v = b_center -
        (north + south + east + west + far + near - 6.0 * center.x) *
        inverse_h_square;
    ushort raw = __float2half_rn(v);
    surf3Dwrite(raw, residual_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

__global__ void ProlongatePackedKernel(float4* out_data,
                                       int num_of_blocks_per_slice,
                                       int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 c = make_float3(x, y, z);
    c *= 0.5f;

    int odd_x = x - ((x >> 1) << 1);
    int odd_y = y - ((y >> 1) << 1);
    int odd_z = z - ((z >> 1) << 1);

    float t_x = -1.0f * (1 - odd_x) * 0.08333333f;
    float t_y = -1.0f * (1 - odd_y) * 0.08333333f;
    float t_z = -1.0f * (1 - odd_z) * 0.08333333f;

    float3 t_c = make_float3(c.x + t_x, c.y + t_y, c.z + t_z);
    float4 result_float = tex3D(prolongate_coarse, t_c.x, t_c.y, t_c.z);

    float3 f_coord = make_float3(float(x) + 0.5f, float(y) + 0.5f,
                                 float(z) + 0.5f);

    float4 original = tex3D(prolongate_fine, f_coord.x, f_coord.y, f_coord.z);
    float4 result = make_float4(original.x + result_float.x, original.y, 0, 0);

    out_data[index] = result;
}

__global__ void ProlongatePackedPureKernel(int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 c = make_float3(x, y, z);
    c *= 0.5f;

    int odd_x = x - ((x >> 1) << 1);
    int odd_y = y - ((y >> 1) << 1);
    int odd_z = z - ((z >> 1) << 1);

    float t_x = -1.0f * (1 - odd_x) * 0.08333333f;
    float t_y = -1.0f * (1 - odd_y) * 0.08333333f;
    float t_z = -1.0f * (1 - odd_z) * 0.08333333f;

    float3 t_c = make_float3(c.x + t_x, c.y + t_y, c.z + t_z);
    float2 result_float = tex3D(prolongate_pure_coarse, t_c.x, t_c.y, t_c.z);

    float3 f_coord = make_float3(x, y, z) + 0.5f;

    float2 original = tex3D(prolongate_pure_fine, f_coord.x, f_coord.y, f_coord.z);
    float2 result = make_float2(original.x + result_float.x, original.y);

    ushort2 raw = make_ushort2(__float2half_rn(result.x),
                               __float2half_rn(result.y));
    surf3Dwrite(raw, prolongate_pure_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void RelaxWithZeroGuessPackedPureKernel(
    float alpha_omega_over_beta, float one_minus_omega, float minus_h_square,
    float omega_times_inverse_beta, int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);

    float  near =    tex3D(guess_source, coord.x, coord.y, coord.z - 1.0f).y;
    float  south =   tex3D(guess_source, coord.x, coord.y - 1.0f, coord.z).y;
    float  west =    tex3D(guess_source, coord.x - 1.0f, coord.y, coord.z).y;
    float2 center =  tex3D(guess_source, coord.x, coord.y, coord.z);
    float  east =    tex3D(guess_source, coord.x + 1.0f, coord.y, coord.z).y;
    float  north =   tex3D(guess_source, coord.x, coord.y + 1.0f, coord.z).y;
    float  far =     tex3D(guess_source, coord.x, coord.y, coord.z + 1.0f).y;
    float  b_center = center.y;

    if (coord.y == volume_size.y - 1)
        north = b_center;

    if (coord.y == 0)
        south = b_center;

    if (coord.x == volume_size.x - 1)
        east = b_center;

    if (coord.x == 0)
        west = b_center;

    if (coord.z == volume_size.z - 1)
        far = b_center;

    if (coord.z == 0)
        near = b_center;

    float v = one_minus_omega * (alpha_omega_over_beta * b_center) +
        (alpha_omega_over_beta * (north + south + east + west + far + near) +
        minus_h_square * b_center) * omega_times_inverse_beta;

    ushort2 raw = make_ushort2(__float2half_rn(v), __float2half_rn(b_center));
    surf3Dwrite(raw, guess_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap); 
}

__global__ void RestrictPackedPureKernel(int3 volume_size_fine)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = (make_float3(x, y, z) + 0.5f) * 2.0f;

    float c1 = 0.015625f;
    float c2 = 0.03125f;
    float c4 = 0.0625f;
    float c8 = 0.125f;

    // Changing the order of the following voxel-fetching code will NOT affect
    // the performance of this kernel.
    float4 north_east_near =      c1 * tex3D(restrict_source, coord.x + 1.0f, coord.y + 1.0f, coord.z - 1.0f);
    float4 north_center_near =    c2 * tex3D(restrict_source, coord.x,        coord.y + 1.0f, coord.z - 1.0f);
    float4 north_west_near =      c1 * tex3D(restrict_source, coord.x - 1.0f, coord.y + 1.0f, coord.z - 1.0f);
    float4 center_east_near =     c2 * tex3D(restrict_source, coord.x + 1.0f, coord.y,        coord.z - 1.0f);
    float4 center_center_near =   c4 * tex3D(restrict_source, coord.x,        coord.y,        coord.z - 1.0f);
    float4 center_west_near =     c2 * tex3D(restrict_source, coord.x - 1.0f, coord.y,        coord.z - 1.0f);
    float4 south_east_near =      c1 * tex3D(restrict_source, coord.x + 1.0f, coord.y - 1.0f, coord.z - 1.0f);
    float4 south_center_near =    c2 * tex3D(restrict_source, coord.x,        coord.y - 1.0f, coord.z - 1.0f);
    float4 south_west_near =      c1 * tex3D(restrict_source, coord.x - 1.0f, coord.y - 1.0f, coord.z - 1.0f);

    float4 north_east_center =    c2 * tex3D(restrict_source, coord.x + 1.0f, coord.y + 1.0f, coord.z);
    float4 north_center_center =  c4 * tex3D(restrict_source, coord.x,        coord.y + 1.0f, coord.z);
    float4 north_west_center =    c2 * tex3D(restrict_source, coord.x - 1.0f, coord.y + 1.0f, coord.z);
    float4 center_east_center =   c4 * tex3D(restrict_source, coord.x + 1.0f, coord.y,        coord.z);
    float4 center_center_center = c8 * tex3D(restrict_source, coord.x,        coord.y,        coord.z);
    float4 center_west_center =   c4 * tex3D(restrict_source, coord.x - 1.0f, coord.y,        coord.z);
    float4 south_east_center =    c2 * tex3D(restrict_source, coord.x + 1.0f, coord.y - 1.0f, coord.z);
    float4 south_center_center =  c4 * tex3D(restrict_source, coord.x,        coord.y - 1.0f, coord.z);
    float4 south_west_center =    c2 * tex3D(restrict_source, coord.x - 1.0f, coord.y - 1.0f, coord.z);

    float4 north_east_far =       c1 * tex3D(restrict_source, coord.x + 1.0f, coord.y + 1.0f, coord.z + 1.0f);
    float4 north_center_far =     c2 * tex3D(restrict_source, coord.x,        coord.y + 1.0f, coord.z + 1.0f);
    float4 north_west_far =       c1 * tex3D(restrict_source, coord.x - 1.0f, coord.y + 1.0f, coord.z + 1.0f);
    float4 center_east_far =      c2 * tex3D(restrict_source, coord.x + 1.0f, coord.y,        coord.z + 1.0f);
    float4 center_center_far =    c4 * tex3D(restrict_source, coord.x,        coord.y,        coord.z + 1.0f);
    float4 center_west_far =      c2 * tex3D(restrict_source, coord.x - 1.0f, coord.y,        coord.z + 1.0f);
    float4 south_east_far =       c1 * tex3D(restrict_source, coord.x + 1.0f, coord.y - 1.0f, coord.z + 1.0f);
    float4 south_center_far =     c2 * tex3D(restrict_source, coord.x,        coord.y - 1.0f, coord.z + 1.0f);
    float4 south_west_far =       c1 * tex3D(restrict_source, coord.x - 1.0f, coord.y - 1.0f, coord.z + 1.0f);

    float3 tex_size = make_float3(volume_size_fine) - 1.001f;
    float scale = 0.5f;

    if (coord.x > tex_size.x) {
        center_east_center = center_center_center;
    }

    if (coord.x < 1.0001f) { 
        center_west_center = scale * center_center_center;
    }

    if (coord.z > tex_size.z) {
        center_center_far = scale * center_center_center;
    }

    if (coord.z < 1.0001f) {
        center_center_near = scale * center_center_center;
    }

    if (coord.y > tex_size.y) {
        north_center_center = scale * center_center_center;
    }

    if (coord.y < 1.0001f) {
        south_center_center = scale * center_center_center;
    }

    // Pass 2: 1-center cells.
    if (coord.x > tex_size.x) {
        center_east_near = scale * center_center_near;
        north_east_center = scale * north_center_center;
        south_east_center = scale * south_center_center;
        center_east_far = scale * center_center_far;
    }

    if (coord.x < 1.0001f) {
        center_west_near = scale * center_center_near;
        north_west_center = scale * north_center_center;
        south_west_center = scale * south_center_center;
        center_west_far = scale * center_center_far;
    }

    if (coord.z > tex_size.z) {
        north_center_far = scale * north_center_center;
        center_east_far = scale * center_east_center;
        center_west_far = scale * center_west_center;
        south_center_far = scale * south_center_center;
    }

    if (coord.z < 1.0001f) {
        north_center_near = scale * north_center_center;
        center_east_near = scale * center_east_center;
        center_west_near = scale * center_west_center;
        south_center_near = scale * south_center_center;
    }

    if (coord.y > tex_size.y) {
        north_center_near = scale * center_center_near;
        north_east_center = scale * center_east_center;
        north_west_center = scale * center_west_center;
        north_center_far = scale * center_center_far;
    }

    if (coord.y < 1.0001f) {
        south_center_near = scale * center_center_near;
        south_east_center = scale * center_east_center;
        south_west_center = scale * center_west_center;
        south_center_far = scale * center_center_far;
    }

    // Pass 3: corner cells.
    if (coord.x > tex_size.x) {
        north_east_near = scale * north_center_near;
        south_east_near = scale * south_center_near;
        north_east_far = scale * north_center_far;
        south_east_far = scale * south_center_far;
    }

    if (coord.x < 1.0001f) {
        north_west_near = scale * north_center_near;
        south_west_near = scale * south_center_near;
        north_west_far = scale * north_center_far;
        south_west_far = scale * south_center_far;
    }

    if (coord.z > tex_size.z) {
        north_east_far = scale * north_east_center;
        north_west_far = scale * north_west_center;
        south_east_far = scale * south_east_center;
        south_west_far = scale * south_west_center;
    }

    if (coord.z < 1.0001f) {
        north_east_near = scale * north_east_center;
        north_west_near = scale * north_west_center;
        south_east_near = scale * south_east_center;
        south_west_near = scale * south_west_center;
    }

    if (coord.y > tex_size.y) {
        north_east_near = scale * center_east_near;
        north_west_near = scale * center_west_near;
        north_east_far = scale * center_east_far;
        north_west_far = scale * center_west_far;
    }

    if (coord.y < 1.0001f) {
        south_east_near = scale * center_east_near;
        south_west_near = scale * center_west_near;
        south_east_far = scale * center_east_far;
        south_west_far = scale * center_west_far;
    }

    float4 result =
        north_east_near +
        north_center_near +
        north_west_near +
        center_east_near +
        center_center_near +
        center_west_near +
        south_east_near +
        south_center_near +
        south_west_near +

        north_east_center +
        north_center_center +
        north_west_center +
        center_west_center +
        center_center_center +
        center_west_center +
        south_east_center +
        south_center_center +
        south_west_center +

        north_east_far +
        north_center_far +
        north_west_far +
        center_east_far +
        center_center_far +
        center_west_far +
        south_east_far +
        south_center_far +
        south_west_far;

    ushort4 raw = make_ushort4(__float2half_rn(result.x),
                               __float2half_rn(result.y),
                               0, 0);
    surf3Dwrite(raw, restrict_dest, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void RestrictResidualPackedPureKernel(int3 volume_size_fine)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = (make_float3(x, y, z) + 0.5f) * 2.0f;

    float c1 = 0.015625f;
    float c2 = 0.03125f;
    float c4 = 0.0625f;
    float c8 = 0.125f;

    float north_east_near =      c1 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y + 1.0f, coord.z - 1.0f);
    float north_center_near =    c2 * tex3D(restrict_residual_source, coord.x,        coord.y + 1.0f, coord.z - 1.0f);
    float north_west_near =      c1 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y + 1.0f, coord.z - 1.0f);
    float center_east_near =     c2 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y,        coord.z - 1.0f);
    float center_center_near =   c4 * tex3D(restrict_residual_source, coord.x,        coord.y,        coord.z - 1.0f);
    float center_west_near =     c2 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y,        coord.z - 1.0f);
    float south_east_near =      c1 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y - 1.0f, coord.z - 1.0f);
    float south_center_near =    c2 * tex3D(restrict_residual_source, coord.x,        coord.y - 1.0f, coord.z - 1.0f);
    float south_west_near =      c1 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y - 1.0f, coord.z - 1.0f);

    float north_east_center =    c2 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y + 1.0f, coord.z);
    float north_center_center =  c4 * tex3D(restrict_residual_source, coord.x,        coord.y + 1.0f, coord.z);
    float north_west_center =    c2 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y + 1.0f, coord.z);
    float center_east_center =   c4 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y,        coord.z);
    float center_center_center = c8 * tex3D(restrict_residual_source, coord.x,        coord.y,        coord.z);
    float center_west_center =   c4 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y,        coord.z);
    float south_east_center =    c2 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y - 1.0f, coord.z);
    float south_center_center =  c4 * tex3D(restrict_residual_source, coord.x,        coord.y - 1.0f, coord.z);
    float south_west_center =    c2 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y - 1.0f, coord.z);

    float north_east_far =       c1 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y + 1.0f, coord.z + 1.0f);
    float north_center_far =     c2 * tex3D(restrict_residual_source, coord.x,        coord.y + 1.0f, coord.z + 1.0f);
    float north_west_far =       c1 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y + 1.0f, coord.z + 1.0f);
    float center_east_far =      c2 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y,        coord.z + 1.0f);
    float center_center_far =    c4 * tex3D(restrict_residual_source, coord.x,        coord.y,        coord.z + 1.0f);
    float center_west_far =      c2 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y,        coord.z + 1.0f);
    float south_east_far =       c1 * tex3D(restrict_residual_source, coord.x + 1.0f, coord.y - 1.0f, coord.z + 1.0f);
    float south_center_far =     c2 * tex3D(restrict_residual_source, coord.x,        coord.y - 1.0f, coord.z + 1.0f);
    float south_west_far =       c1 * tex3D(restrict_residual_source, coord.x - 1.0f, coord.y - 1.0f, coord.z + 1.0f);

    float3 tex_size = make_float3(volume_size_fine) - 1.001f;
    float scale = 0.5f;

    if (coord.x > tex_size.x) {
        center_east_center = center_center_center;
    }

    if (coord.x < 1.0001f) { 
        center_west_center = scale * center_center_center;
    }

    if (coord.z > tex_size.z) {
        center_center_far = scale * center_center_center;
    }

    if (coord.z < 1.0001f) {
        center_center_near = scale * center_center_center;
    }

    if (coord.y > tex_size.y) {
        north_center_center = scale * center_center_center;
    }

    if (coord.y < 1.0001f) {
        south_center_center = scale * center_center_center;
    }

    // Pass 2: 1-center cells.
    if (coord.x > tex_size.x) {
        center_east_near = scale * center_center_near;
        north_east_center = scale * north_center_center;
        south_east_center = scale * south_center_center;
        center_east_far = scale * center_center_far;
    }

    if (coord.x < 1.0001f) {
        center_west_near = scale * center_center_near;
        north_west_center = scale * north_center_center;
        south_west_center = scale * south_center_center;
        center_west_far = scale * center_center_far;
    }

    if (coord.z > tex_size.z) {
        north_center_far = scale * north_center_center;
        center_east_far = scale * center_east_center;
        center_west_far = scale * center_west_center;
        south_center_far = scale * south_center_center;
    }

    if (coord.z < 1.0001f) {
        north_center_near = scale * north_center_center;
        center_east_near = scale * center_east_center;
        center_west_near = scale * center_west_center;
        south_center_near = scale * south_center_center;
    }

    if (coord.y > tex_size.y) {
        north_center_near = scale * center_center_near;
        north_east_center = scale * center_east_center;
        north_west_center = scale * center_west_center;
        north_center_far = scale * center_center_far;
    }

    if (coord.y < 1.0001f) {
        south_center_near = scale * center_center_near;
        south_east_center = scale * center_east_center;
        south_west_center = scale * center_west_center;
        south_center_far = scale * center_center_far;
    }

    // Pass 3: corner cells.
    if (coord.x > tex_size.x) {
        north_east_near = scale * north_center_near;
        south_east_near = scale * south_center_near;
        north_east_far = scale * north_center_far;
        south_east_far = scale * south_center_far;
    }

    if (coord.x < 1.0001f) {
        north_west_near = scale * north_center_near;
        south_west_near = scale * south_center_near;
        north_west_far = scale * north_center_far;
        south_west_far = scale * south_center_far;
    }

    if (coord.z > tex_size.z) {
        north_east_far = scale * north_east_center;
        north_west_far = scale * north_west_center;
        south_east_far = scale * south_east_center;
        south_west_far = scale * south_west_center;
    }

    if (coord.z < 1.0001f) {
        north_east_near = scale * north_east_center;
        north_west_near = scale * north_west_center;
        south_east_near = scale * south_east_center;
        south_west_near = scale * south_west_center;
    }

    if (coord.y > tex_size.y) {
        north_east_near = scale * center_east_near;
        north_west_near = scale * center_west_near;
        north_east_far = scale * center_east_far;
        north_west_far = scale * center_west_far;
    }

    if (coord.y < 1.0001f) {
        south_east_near = scale * center_east_near;
        south_west_near = scale * center_west_near;
        south_east_far = scale * center_east_far;
        south_west_far = scale * center_west_far;
    }

    float result =
        north_east_near +
        north_center_near +
        north_west_near +
        center_east_near +
        center_center_near +
        center_west_near +
        south_east_near +
        south_center_near +
        south_west_near +

        north_east_center +
        north_center_center +
        north_west_center +
        center_west_center +
        center_center_center +
        center_west_center +
        south_east_center +
        south_center_center +
        south_west_center +

        north_east_far +
        north_center_far +
        north_west_far +
        center_east_far +
        center_center_far +
        center_west_far +
        south_east_far +
        south_center_far +
        south_west_far;

    ushort2 raw = make_ushort2(0, __float2half_rn(result));
    surf3Dwrite(raw, restrict_residual_dest, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchComputeResidualPackedPure(hipArray* dest_array,
                                     hipArray* source_array,
                                     float inverse_h_square,
                                     int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&residual_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    residual_source.normalized = false;
    residual_source.filterMode = hipFilterModePoint;
    residual_source.addressMode[0] = hipAddressModeClamp;
    residual_source.addressMode[1] = hipAddressModeClamp;
    residual_source.addressMode[2] = hipAddressModeClamp;
    residual_source.channelDesc = desc;

    result = hipBindTextureToArray(&residual_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    ComputeResidualPackedPureKernel<<<grid, block>>>(inverse_h_square,
                                                     volume_size);

    hipUnbindTexture(&residual_source);
}

void LaunchProlongatePacked(float4* dest_array, hipArray* coarse_array,
                            hipArray* fine_array, int3 volume_size_fine)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, coarse_array);
    prolongate_coarse.normalized = false;
    prolongate_coarse.filterMode = hipFilterModeLinear;
    prolongate_coarse.addressMode[0] = hipAddressModeClamp;
    prolongate_coarse.addressMode[1] = hipAddressModeClamp;
    prolongate_coarse.addressMode[2] = hipAddressModeClamp;
    prolongate_coarse.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&prolongate_coarse,
                                                coarse_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, fine_array);
    prolongate_fine.normalized = false;

    // TODO: Disabling the linear filter mode may slightly speed up the kernel.
    prolongate_fine.filterMode = hipFilterModeLinear;
    prolongate_fine.addressMode[0] = hipAddressModeClamp;
    prolongate_fine.addressMode[1] = hipAddressModeClamp;
    prolongate_fine.addressMode[2] = hipAddressModeClamp;
    prolongate_fine.channelDesc = desc;

    result = hipBindTextureToArray(&prolongate_fine, fine_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int3 volume_size = volume_size_fine;
    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ProlongatePackedKernel<<<grid, block>>>(dest_array, num_of_blocks_per_slice,
                                            slice_stride, volume_size);

    hipUnbindTexture(&prolongate_fine);
    hipUnbindTexture(&prolongate_coarse);
}

void LaunchProlongatePackedPure(hipArray* dest_array, hipArray* coarse_array,
                                hipArray* fine_array, int3 volume_size_fine)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&prolongate_pure_dest,
                                                dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, coarse_array);
    prolongate_pure_coarse.normalized = false;
    prolongate_pure_coarse.filterMode = hipFilterModeLinear;
    prolongate_pure_coarse.addressMode[0] = hipAddressModeClamp;
    prolongate_pure_coarse.addressMode[1] = hipAddressModeClamp;
    prolongate_pure_coarse.addressMode[2] = hipAddressModeClamp;
    prolongate_pure_coarse.channelDesc = desc;

    result = hipBindTextureToArray(&prolongate_pure_coarse, coarse_array,
                                    &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, fine_array);
    prolongate_pure_fine.normalized = false;
    prolongate_pure_fine.filterMode = hipFilterModeLinear;
    prolongate_pure_fine.addressMode[0] = hipAddressModeClamp;
    prolongate_pure_fine.addressMode[1] = hipAddressModeClamp;
    prolongate_pure_fine.addressMode[2] = hipAddressModeClamp;
    prolongate_pure_fine.channelDesc = desc;

    result = hipBindTextureToArray(&prolongate_pure_fine, fine_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size_fine.x / 8);
    dim3 grid(volume_size_fine.x / block.x, volume_size_fine.y / block.y,
              volume_size_fine.z / block.z);

    ProlongatePackedPureKernel<<<grid, block>>>(volume_size_fine);

    hipUnbindTexture(&prolongate_pure_fine);
    hipUnbindTexture(&prolongate_pure_coarse);
}

void LaunchRelaxWithZeroGuessPackedPure(hipArray* dest_array,
                                        hipArray* source_array,
                                        float alpha_omega_over_beta,
                                        float one_minus_omega,
                                        float minus_h_square,
                                        float omega_times_inverse_beta,
                                        int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&guess_dest, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    guess_source.normalized = false;
    guess_source.filterMode = hipFilterModePoint;
    guess_source.addressMode[0] = hipAddressModeClamp;
    guess_source.addressMode[1] = hipAddressModeClamp;
    guess_source.addressMode[2] = hipAddressModeClamp;
    guess_source.channelDesc = desc;

    result = hipBindTextureToArray(&guess_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    RelaxWithZeroGuessPackedPureKernel<<<grid, block>>>(
        alpha_omega_over_beta, one_minus_omega, minus_h_square,
        omega_times_inverse_beta, volume_size);

    hipUnbindTexture(&guess_source);
}

void LaunchRestrictPackedPure(hipArray* dest_array, hipArray* source_array,
                              int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&restrict_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    restrict_source.normalized = false;
    restrict_source.filterMode = hipFilterModeLinear;
    restrict_source.addressMode[0] = hipAddressModeClamp;
    restrict_source.addressMode[1] = hipAddressModeClamp;
    restrict_source.addressMode[2] = hipAddressModeClamp;
    restrict_source.channelDesc = desc;

    result = hipBindTextureToArray(&restrict_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int3 volume_size_fine = volume_size * 2;
    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    RestrictPackedPureKernel<<<grid, block>>>(volume_size_fine);

    hipUnbindTexture(&restrict_source);
}

void LaunchRestrictResidualPackedPure(hipArray* dest_array,
                                      hipArray* source_array, int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&restrict_residual_dest,
                                                dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    restrict_residual_source.normalized = false;
    restrict_residual_source.filterMode = hipFilterModeLinear;
    restrict_residual_source.addressMode[0] = hipAddressModeClamp;
    restrict_residual_source.addressMode[1] = hipAddressModeClamp;
    restrict_residual_source.addressMode[2] = hipAddressModeClamp;
    restrict_residual_source.channelDesc = desc;

    result = hipBindTextureToArray(&restrict_residual_source, source_array,
                                    &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int3 volume_size_fine = volume_size * 2;
    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    RestrictResidualPackedPureKernel<<<grid, block>>>(volume_size_fine);

    hipUnbindTexture(&restrict_residual_source);
}
