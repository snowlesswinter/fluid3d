#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

texture<float1, hipTextureType3D, hipReadModeElementType> in_tex;
surface<void, hipTextureType3D> clear_volume;

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float1 cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc.x;
}

__global__ void ClearVolume4Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(value, clear_volume, x * sizeof(float4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolume2Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(make_float2(value.x, value.y), clear_volume, x * sizeof(float2),
                y, z, hipBoundaryModeTrap);
}

__global__ void ClearVolume1Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(value.x, clear_volume, x * sizeof(float1), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf4Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort4 raw = make_ushort4(__float2half_rn(value.x),
                               __float2half_rn(value.y),
                               __float2half_rn(value.z),
                               __float2half_rn(value.w));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf2Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort2 raw = make_ushort2(__float2half_rn(value.x),
                               __float2half_rn(value.y));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf1Kernel(float4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort1 raw = make_ushort1(__float2half_rn(value.x));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort1), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

bool IsHalf1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 16 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 16 && desc.z == 0 && desc.w == 0)||
            (desc.y == 16 && desc.z == 16 && desc.w == 16));
}

bool IsFloat1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 32 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 32 && desc.z == 0 && desc.w == 0)||
            (desc.y == 32 && desc.z == 32 && desc.w == 32));
}

bool IsCompliant(const hipChannelFormatDesc& desc)
{
    return IsHalf1Or2Or4(desc) || IsFloat1Or2Or4(desc);
}

void LaunchClearVolumeKernel(hipArray* dest_array, float4 value,
                             int3 volume_size)
{
    hipChannelFormatDesc desc;
    hipError_t result = hipGetChannelDesc(&desc, dest_array);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    result = cudaBindSurfaceToArray(&clear_volume, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 16);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);

    assert(IsCompliant(desc));
    if (desc.x == 16 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf1Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf2Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 16 && desc.w == 16 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf4Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolume1Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume2Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 32 && desc.w == 32 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume4Kernel<<<grid, block>>>(value);
}