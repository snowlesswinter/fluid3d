#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "third_party/glm/common.hpp"
#include "third_party/glm/glm.hpp"
#include "third_party/glm/mat3x3.hpp"
#include "third_party/glm/mat4x4.hpp"
#include "third_party/glm/vec2.hpp"
#include "third_party/glm/vec3.hpp"

texture<float1, hipTextureType3D, hipReadModeElementType> in_tex;
surface<void, cudaSurfaceType3D> clear_volume;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> raycast_density;
surface<void, cudaSurfaceType2D> raycast_dest;

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float1 cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc.x;
}

__global__ void ClearVolume4Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(make_float4(value.x, value.y, value.z, value.w), clear_volume,
                x * sizeof(float4), y, z, hipBoundaryModeTrap);
}

__global__ void ClearVolume2Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(make_float2(value.x, value.y), clear_volume, x * sizeof(float2),
                y, z, hipBoundaryModeTrap);
}

__global__ void ClearVolume1Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(value.x, clear_volume, x * sizeof(float), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf4Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort4 raw = make_ushort4(__float2half_rn(value.x),
                               __float2half_rn(value.y),
                               __float2half_rn(value.z),
                               __float2half_rn(value.w));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf2Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort2 raw = make_ushort2(__float2half_rn(value.x),
                               __float2half_rn(value.y));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf1Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort1 raw = make_ushort1(__float2half_rn(value.x));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort1), y, z,
                hipBoundaryModeTrap);
}

__device__ bool IntersectAABB(glm::vec3 ray_dir, glm::vec3 eye_pos,
                              glm::vec3 min_pos, glm::vec3 max_pos, float* near,
                              float* far)
{
    glm::vec3 inverse_ray_dir = 1.0f / ray_dir;
    glm::vec3 bottom = inverse_ray_dir * (min_pos - eye_pos);
    glm::vec3 top = inverse_ray_dir * (max_pos - eye_pos);
    glm::vec3 near_corner_dist = glm::min(top, bottom);
    glm::vec3 far_corner_dist = glm::max(top, bottom);
    glm::vec2 t = glm::max(glm::vec2(near_corner_dist.x),
                           glm::vec2(near_corner_dist.y, near_corner_dist.z));
    *near = glm::max(t.x, t.y);
    t = glm::min(glm::vec2(far_corner_dist.x),
                 glm::vec2(far_corner_dist.y, far_corner_dist.z));
    *far = glm::min(t.x, t.y);
    return near <= far;
}

__global__ void RaycastKernel(glm::mat3 model_view, glm::vec2 viewport_size,
                              glm::vec3 eye_pos, float focal_length,
                              glm::vec2 offset, glm::vec3 light_intensity)
{
    const float kMaxDistance = sqrt(2.0f);
    const int kNumSamples = 224;
    const float kStepSize = kMaxDistance / static_cast<float>(kNumSamples);
    const int kNumLightSamples = 64;
    const float kLightScale =
        kMaxDistance / static_cast<float>(kNumLightSamples);
    const float kAbsorptionTimesStepSize = 10.0f * kStepSize;
    const float kDensityFactor = 30.0f;
    const float kOcclusionFactor = 15.0f;
    const glm::vec3 light_pos(1.5f, 0.7f, 0.0f);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= static_cast<int>(viewport_size.x) ||
            y >= static_cast<int>(viewport_size.y))
        return;

    glm::vec3 ray_dir;

    // Normalize ray direction vector and transfrom to world space.
    ray_dir.x = 2.0f * x / viewport_size.x - 1.0f;
    ray_dir.y = 2.0f * y / viewport_size.y - 1.0f;
    ray_dir.z = -focal_length;

    // Transform the ray direction vector to model-view space.
    ray_dir = glm::normalize(ray_dir * model_view);

    // Ray origin is already in model space.
    float near;
    float far;
    IntersectAABB(ray_dir, eye_pos, glm::vec3(-1.0f), glm::vec3(1.0f),
                  &near, &far);
    if (near < 0.0f)
        near = 0.0f;

    glm::vec3 ray_start = eye_pos + ray_dir * near;
    glm::vec3 ray_stop = eye_pos + ray_dir * far;

    // Transfrom to [0, 1) model space.
    ray_start = 0.5f * (ray_start + 1.0f);
    ray_stop = 0.5f * (ray_stop + 1.0f);

    glm::vec3 pos = ray_start;
    glm::vec3 step = glm::normalize(ray_stop - ray_start) * kStepSize;
    float travel = glm::distance(ray_stop, ray_start);
    float transparency = 1.0f;
    glm::vec3 accumulated(0.0f);

    for (int i = 0; i < kNumSamples && travel > 0.0f;
             i++, pos += step, travel -= kStepSize) {
        float density =
            tex3D(raycast_density, pos.x, pos.y, pos.z) * kDensityFactor;
        if (density < 0.0001f)
            continue;

        glm::vec3 light_dir = glm::normalize(light_pos - pos) * kLightScale;
        float light_weight = 1.0f;
        glm::vec3 l_pos = pos + light_dir;

        for (int j = 0; j < kNumLightSamples; j++) {
            float occlusion = tex3D(raycast_density, l_pos.x, l_pos.y, l_pos.z);
            light_weight *=
                1.0f - kAbsorptionTimesStepSize * occlusion * kOcclusionFactor;
            if (light_weight <= 0.01f)
                break;

            l_pos += light_dir;
        }

        transparency *= 1.0f - density * kAbsorptionTimesStepSize;
        accumulated +=
            light_intensity * light_weight * transparency * density * kStepSize;

        if (transparency <= 0.01f)
            break;
    }

    ushort4 raw = make_ushort4(__float2half_rn(accumulated.x),
                               __float2half_rn(accumulated.y),
                               __float2half_rn(accumulated.z),
                               __float2half_rn(1.0f - transparency));
    surf2Dwrite(raw, raycast_dest, (x + offset.x) * sizeof(ushort4),
                (y + offset.y), hipBoundaryModeTrap);
}

// =============================================================================

bool IsHalf1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 16 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 16 && desc.z == 0 && desc.w == 0)||
            (desc.y == 16 && desc.z == 16 && desc.w == 16));
}

bool IsFloat1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 32 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 32 && desc.z == 0 && desc.w == 0)||
            (desc.y == 32 && desc.z == 32 && desc.w == 32));
}

bool IsCompliant(const hipChannelFormatDesc& desc)
{
    return IsHalf1Or2Or4(desc) || IsFloat1Or2Or4(desc);
}

void LaunchClearVolumeKernel(hipArray* dest_array, const glm::vec4& value,
                             const glm::ivec3& volume_size)
{
    hipChannelFormatDesc desc;
    hipError_t result = hipGetChannelDesc(&desc, dest_array);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    result = cudaBindSurfaceToArray(&clear_volume, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 16);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);

    assert(IsCompliant(desc));
    if (desc.x == 16 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf1Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf2Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 16 && desc.w == 16 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf4Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolume1Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume2Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 32 && desc.w == 32 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume4Kernel<<<grid, block>>>(value);
}

void LaunchRaycastKernel(hipArray* dest_array, hipArray* density_array,
                         const glm::mat4& model_view,
                         const glm::ivec2& surface_size,
                         const glm::vec3& eye_pos, float focal_length)
{
    hipChannelFormatDesc desc;
    hipError_t result = hipGetChannelDesc(&desc, dest_array);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    result = cudaBindSurfaceToArray(&raycast_dest, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, density_array);
    raycast_density.normalized = true;
    raycast_density.filterMode = hipFilterModeLinear;
    raycast_density.addressMode[0] = hipAddressModeClamp;
    raycast_density.addressMode[1] = hipAddressModeClamp;
    raycast_density.addressMode[2] = hipAddressModeClamp;
    raycast_density.channelDesc = desc;

    result = hipBindTextureToArray(&raycast_density, density_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int t = min(surface_size.x, surface_size.y);
    glm::vec2 viewport_size(static_cast<float>(t));
    glm::ivec2 offset((surface_size.x - t) / 2, (surface_size.y - t) / 2);
    glm::mat3 m(model_view);

    dim3 block(32, 8, 1);
    dim3 grid((t + block.x - 1) / block.x, (t + block.y - 1) / block.y, 1);

    //glm::vec3 light_intensity(6.2109375f, 7.2265625f, 8.0078125f);
    glm::vec3 light_intensity = glm::normalize(glm::vec3(171, 160, 139));
    light_intensity *= 22.0f;
    RaycastKernel<<<grid, block>>>(m, viewport_size, eye_pos, focal_length,
                                   offset, light_intensity);
}