#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "third_party/glm/common.hpp"
#include "third_party/glm/glm.hpp"
#include "third_party/glm/mat3x3.hpp"
#include "third_party/glm/mat4x4.hpp"
#include "third_party/glm/vec2.hpp"
#include "third_party/glm/vec3.hpp"

texture<float1, hipTextureType3D, hipReadModeElementType> in_tex;
surface<void, cudaSurfaceType3D> clear_volume;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> raycast_density;
surface<void, cudaSurfaceType2D> raycast_dest;

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float1 cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc.x;
}

__global__ void ClearVolume4Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(make_float4(value.x, value.y, value.z, value.w), clear_volume,
                x * sizeof(float4), y, z, hipBoundaryModeTrap);
}

__global__ void ClearVolume2Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(make_float2(value.x, value.y), clear_volume, x * sizeof(float2),
                y, z, hipBoundaryModeTrap);
}

__global__ void ClearVolume1Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    surf3Dwrite(value.x, clear_volume, x * sizeof(float), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf4Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort4 raw = make_ushort4(__float2half_rn(value.x),
                               __float2half_rn(value.y),
                               __float2half_rn(value.z),
                               __float2half_rn(value.w));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort4), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf2Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort2 raw = make_ushort2(__float2half_rn(value.x),
                               __float2half_rn(value.y));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort2), y, z,
                hipBoundaryModeTrap);
}

__global__ void ClearVolumeHalf1Kernel(glm::vec4 value)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    ushort1 raw = make_ushort1(__float2half_rn(value.x));
    surf3Dwrite(raw, clear_volume, x * sizeof(ushort1), y, z,
                hipBoundaryModeTrap);
}

__device__ bool IntersectAABB(glm::vec3 ray_dir, glm::vec3 eye_pos,
                              glm::vec3 min_pos, glm::vec3 max_pos, float* near,
                              float* far)
{
    glm::vec3 inverse_ray_dir = 1.0f / ray_dir;
    glm::vec3 bottom = inverse_ray_dir * (min_pos - eye_pos);
    glm::vec3 top = inverse_ray_dir * (max_pos - eye_pos);
    glm::vec3 near_corner_dist = glm::min(top, bottom);
    glm::vec3 far_corner_dist = glm::max(top, bottom);
    glm::vec2 t = glm::max(glm::vec2(near_corner_dist.x),
                           glm::vec2(near_corner_dist.y, near_corner_dist.z));
    *near = glm::max(t.x, t.y);
    t = glm::min(glm::vec2(far_corner_dist.x),
                 glm::vec2(far_corner_dist.y, far_corner_dist.z));
    *far = glm::min(t.x, t.y);
    return near <= far;
}

__global__ void RaycastKernel(glm::mat3 model_view, glm::vec2 viewport_size,
                              glm::vec3 eye_pos, float focal_length,
                              glm::vec2 offset, glm::vec3 light_intensity,
                              int num_samples, float step_size,
                              int num_light_samples, float light_scale,
                              float step_absorption, float density_factor,
                              float occlusion_factor)
{
    const glm::vec3 light_pos(1.5f, 0.7f, 0.0f);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= static_cast<int>(viewport_size.x) ||
            y >= static_cast<int>(viewport_size.y))
        return;

    glm::vec3 ray_dir;

    // Normalize ray direction vector and transfrom to world space.
    ray_dir.x = 2.0f * x / viewport_size.x - 1.0f;
    ray_dir.y = 2.0f * y / viewport_size.y - 1.0f;
    ray_dir.z = -focal_length;

    // Transform the ray direction vector to model-view space.
    ray_dir = glm::normalize(ray_dir * model_view);

    // Ray origin is already in model space.
    float near;
    float far;
    IntersectAABB(ray_dir, eye_pos, glm::vec3(-1.0f), glm::vec3(1.0f),
                  &near, &far);
    if (near < 0.0f)
        near = 0.0f;

    glm::vec3 ray_start = eye_pos + ray_dir * near;
    glm::vec3 ray_stop = eye_pos + ray_dir * far;

    // Transfrom to [0, 1) model space.
    ray_start = 0.5f * (ray_start + 1.0f);
    ray_stop = 0.5f * (ray_stop + 1.0f);

    glm::vec3 pos = ray_start;
    glm::vec3 step = glm::normalize(ray_stop - ray_start) * step_size;
    float travel = glm::distance(ray_stop, ray_start);
    float transmittance = 1.0f;
    float luminance = 0.0f;

    for (int i = 0; i < num_samples && travel > 0.0f;
            i++, pos += step, travel -= step_size) {
        float density =
            tex3D(raycast_density, pos.x, pos.y, pos.z) * density_factor;
        if (density < 0.01f)
            continue;

        glm::vec3 light_dir = glm::normalize(light_pos - pos) * light_scale;
        float light_weight = 1.0f;
        glm::vec3 l_pos = pos + light_dir;

        for (int j = 0; j < num_light_samples; j++) {
            float d = tex3D(raycast_density, l_pos.x, l_pos.y, l_pos.z);
            light_weight *= 1.0f - step_absorption * d * occlusion_factor;
            if (light_weight <= 0.01f)
                break;

            // Early termination. Great performance gain.
            if (l_pos.x < 0.0f || l_pos.y < 0.0f || l_pos.z < 0.0f ||
                    l_pos.x > 1.0f || l_pos.y > 1.0f || l_pos.z > 1.0f)
                break;

            l_pos += light_dir;
        }

        transmittance *= 1.0f - density * step_absorption;
        luminance += light_weight * transmittance * density;

        if (transmittance <= 0.01f)
            break;
    }

    ushort4 raw = make_ushort4(
        __float2half_rn(light_intensity.x * luminance * step_size),
        __float2half_rn(light_intensity.y * luminance * step_size),
        __float2half_rn(light_intensity.z * luminance * step_size),
        __float2half_rn(1.0f - transmittance));
    surf2Dwrite(raw, raycast_dest, (x + offset.x) * sizeof(ushort4),
                (y + offset.y), hipBoundaryModeTrap);
}

__global__ void RaycastFastKernel(glm::mat3 model_view, glm::vec2 viewport_size,
                                  glm::vec3 eye_pos, float focal_length,
                                  glm::vec2 offset, glm::vec3 light_intensity,
                                  int num_samples, float step_size,
                                  int num_light_samples, float light_scale,
                                  float step_absorption, float density_factor,
                                  float occlusion_factor)
{
    const glm::vec3 light_pos(1.5f, 0.7f, 0.0f);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= static_cast<int>(viewport_size.x) ||
            y >= static_cast<int>(viewport_size.y))
        return;

    glm::vec3 ray_dir;

    // Normalize ray direction vector and transfrom to world space.
    ray_dir.x = 2.0f * x / viewport_size.x - 1.0f;
    ray_dir.y = 2.0f * y / viewport_size.y - 1.0f;
    ray_dir.z = -focal_length;

    // Transform the ray direction vector to model-view space.
    ray_dir = glm::normalize(ray_dir * model_view);

    // Ray origin is already in model space.
    float near;
    float far;
    IntersectAABB(ray_dir, eye_pos, glm::vec3(-1.0f), glm::vec3(1.0f),
                  &near, &far);
    if (near < 0.0f)
        near = 0.0f;

    glm::vec3 ray_start = eye_pos + ray_dir * near;
    glm::vec3 ray_stop = eye_pos + ray_dir * far;

    // Transfrom to [0, 1) model space.
    ray_start = 0.5f * (ray_start + 1.0f);
    ray_stop = 0.5f * (ray_stop + 1.0f);

    glm::vec3 pos = ray_start;
    glm::vec3 step = glm::normalize(ray_stop - ray_start) * step_size;
    float travel = glm::distance(ray_stop, ray_start);
    float transmittance = 1.0f;
    float luminance = 0.0f;

    for (int i = 0; i < num_samples && travel > 0.0f;
            i++, pos += step, travel -= step_size) {
        float density =
            tex3D(raycast_density, pos.x, pos.y, pos.z) * density_factor;
        if (density < 0.01f)
            continue;

        transmittance *= 1.0f - density * step_absorption;
        if (transmittance <= 0.01f)
            break;
    }

    ushort4 raw = make_ushort4(__float2half_rn(light_intensity.x),
                               __float2half_rn(light_intensity.y),
                               __float2half_rn(light_intensity.z),
                               __float2half_rn(1.0f - transmittance));
    surf2Dwrite(raw, raycast_dest, (x + offset.x) * sizeof(ushort4),
                (y + offset.y), hipBoundaryModeTrap);
}

__global__ void RaycastKernel_color(glm::mat3 model_view,
                                    glm::vec2 viewport_size, glm::vec3 eye_pos,
                                    float focal_length, glm::vec2 offset,
                                    glm::vec3 light_intensity, int num_samples,
                                    float step_size, int num_light_samples,
                                    float light_scale, float step_absorption,
                                    float density_factor,
                                    float occlusion_factor)
{
    const glm::vec3 light_pos(1.5f, 0.7f, 0.0f);
    glm::vec3 smoke_color(57.0f, 88.0f, 78.0f);
    glm::vec3 dark = glm::normalize(smoke_color) * 0.2f;
    smoke_color = glm::normalize(glm::vec3(140.0f, 190.0f, 154.0f)) * 15.0f;
    smoke_color = glm::normalize(glm::vec3(128.0f, 190.0f, 234.0f)) * 15.0f;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= static_cast<int>(viewport_size.x) ||
        y >= static_cast<int>(viewport_size.y))
        return;

    glm::vec3 ray_dir;

    // Normalize ray direction vector and transfrom to world space.
    ray_dir.x = 2.0f * x / viewport_size.x - 1.0f;
    ray_dir.y = 2.0f * y / viewport_size.y - 1.0f;
    ray_dir.z = -focal_length;

    // Transform the ray direction vector to model-view space.
    ray_dir = glm::normalize(ray_dir * model_view);

    // Ray origin is already in model space.
    float near;
    float far;
    IntersectAABB(ray_dir, eye_pos, glm::vec3(-1.0f), glm::vec3(1.0f),
                  &near, &far);
    if (near < 0.0f)
        near = 0.0f;

    glm::vec3 ray_start = eye_pos + ray_dir * near;
    glm::vec3 ray_stop = eye_pos + ray_dir * far;

    // Transfrom to [0, 1) model space.
    ray_start = 0.5f * (ray_start + 1.0f);
    ray_stop = 0.5f * (ray_stop + 1.0f);

    glm::vec3 pos = ray_start;
    glm::vec3 step = glm::normalize(ray_stop - ray_start) * step_size;
    float travel = glm::distance(ray_stop, ray_start);
    float transmittance = 1.0f;
    float luminance = 0.0f;

    for (int i = 0; i < num_samples && travel > 0.0f;
         i++, pos += step, travel -= step_size)
    {
        float density =
            tex3D(raycast_density, pos.x, pos.y, pos.z) * density_factor;
        if (density < 0.01f)
            continue;

        glm::vec3 light_dir = glm::normalize(light_pos - pos) * light_scale;
        float light_weight = 1.0f;
        glm::vec3 l_pos = pos + light_dir;

        for (int j = 0; j < num_light_samples; j++)
        {
            float d = tex3D(raycast_density, l_pos.x, l_pos.y, l_pos.z);
            light_weight *= 1.0f - step_absorption * d * occlusion_factor;
            if (light_weight <= 0.01f)
                break;

            // Early termination. Great performance gain.
            if (l_pos.x < 0.0f || l_pos.y < 0.0f || l_pos.z < 0.0f ||
                l_pos.x > 1.0f || l_pos.y > 1.0f || l_pos.z > 1.0f)
                break;

            l_pos += light_dir;
        }

        transmittance *= 1.0f - density * step_absorption;
        luminance += light_weight * transmittance * density;

        if (transmittance <= 0.01f)
            break;
    }

    float alpha = glm::max(0.0f, glm::min(luminance / 20.0f, 1.0f));
    ushort4 raw = make_ushort4(
        __float2half_rn(dark.x + (light_intensity.x * alpha + (1.0f - alpha) * smoke_color.x) * luminance * step_size),
        __float2half_rn(dark.y + (light_intensity.y * alpha + (1.0f - alpha) * smoke_color.y) * luminance * step_size),
        __float2half_rn(dark.z + (light_intensity.z * alpha + (1.0f - alpha) * smoke_color.z) * luminance * step_size),
        __float2half_rn(1.0f - transmittance));
    surf2Dwrite(raw, raycast_dest, (x + offset.x) * sizeof(ushort4),
                (y + offset.y), hipBoundaryModeTrap);
}

// =============================================================================

bool IsHalf1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 16 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 16 && desc.z == 0 && desc.w == 0)||
            (desc.y == 16 && desc.z == 16 && desc.w == 16));
}

bool IsFloat1Or2Or4(const hipChannelFormatDesc& desc)
{
    if (desc.f != hipChannelFormatKindFloat)
        return false;

    return desc.x == 32 &&
        ((desc.y == 0 && desc.z == 0 && desc.w == 0) ||
            (desc.y == 32 && desc.z == 0 && desc.w == 0)||
            (desc.y == 32 && desc.z == 32 && desc.w == 32));
}

bool IsCompliant(const hipChannelFormatDesc& desc)
{
    return IsHalf1Or2Or4(desc) || IsFloat1Or2Or4(desc);
}

void LaunchClearVolumeKernel(hipArray* dest_array, const glm::vec4& value,
                             const glm::ivec3& volume_size)
{
    hipChannelFormatDesc desc;
    hipError_t result = hipGetChannelDesc(&desc, dest_array);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    result = cudaBindSurfaceToArray(&clear_volume, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 16);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);

    assert(IsCompliant(desc));
    if (desc.x == 16 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf1Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf2Kernel<<<grid, block>>>(value);
    else if (desc.x == 16 && desc.y == 16 && desc.z == 16 && desc.w == 16 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolumeHalf4Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 0 && desc.z == 0 && desc.w == 0 &&
            desc.f == hipChannelFormatKindFloat)
        ClearVolume1Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 0 && desc.w == 0 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume2Kernel<<<grid, block>>>(value);
    else if (desc.x == 32 && desc.y == 32 && desc.z == 32 && desc.w == 32 &&
             desc.f == hipChannelFormatKindFloat)
        ClearVolume4Kernel<<<grid, block>>>(value);
}

void LaunchRaycastKernel(hipArray* dest_array, hipArray* density_array,
                         const glm::mat4& model_view,
                         const glm::ivec2& surface_size,
                         const glm::vec3& eye_pos, const glm::vec3& light_color,
                         float light_intensity, float focal_length,
                         int num_samples, int num_light_samples,
                         float absorption, float density_factor,
                         float occlusion_factor)
{
    hipChannelFormatDesc desc;
    hipError_t result = hipGetChannelDesc(&desc, dest_array);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    result = cudaBindSurfaceToArray(&raycast_dest, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, density_array);
    raycast_density.normalized = true;
    raycast_density.filterMode = hipFilterModeLinear;
    raycast_density.addressMode[0] = hipAddressModeClamp;
    raycast_density.addressMode[1] = hipAddressModeClamp;
    raycast_density.addressMode[2] = hipAddressModeClamp;
    raycast_density.channelDesc = desc;

    result = hipBindTextureToArray(&raycast_density, density_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int t = min(surface_size.x, surface_size.y);
    glm::vec2 viewport_size(static_cast<float>(t));
    glm::ivec2 offset((surface_size.x - t) / 2, (surface_size.y - t) / 2);
    glm::mat3 m(model_view);

    dim3 block(32, 8, 1);
    dim3 grid((t + block.x - 1) / block.x, (t + block.y - 1) / block.y, 1);

    glm::vec3 intensity = glm::normalize(light_color);
    intensity *= light_intensity;
    const float kMaxDistance = sqrt(2.0f);
    const float kStepSize = kMaxDistance / static_cast<float>(num_samples);
    const float kLightScale =
        kMaxDistance / static_cast<float>(num_light_samples);
    const float kAbsorptionTimesStepSize = absorption * kStepSize;
    
    const bool fast = true;
    if (fast)
        RaycastFastKernel<<<grid, block>>>(m, viewport_size, eye_pos,
                                           focal_length, offset, intensity,
                                           num_samples, kStepSize,
                                           num_light_samples, kLightScale,
                                           kAbsorptionTimesStepSize,
                                           density_factor, occlusion_factor);
    else
        RaycastKernel<<<grid, block>>>(m, viewport_size, eye_pos, focal_length,
                                       offset, intensity, num_samples,
                                       kStepSize, num_light_samples,
                                       kLightScale, kAbsorptionTimesStepSize,
                                       density_factor, occlusion_factor);
}