#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "block_arrangement.h"

surface<void, cudaSurfaceType3D> jacobi;
texture<ushort2, hipTextureType3D, hipReadModeNormalizedFloat> jacobi_packed;

__global__ void DampedJacobiKernel(float minus_square_cell_size,
                                   float omega_over_beta, uint3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float near =              tex3D(jacobi_packed, x, y, z - 1.0f).x;
    float south =             tex3D(jacobi_packed, x, y - 1.0f, z).x;
    float west =              tex3D(jacobi_packed, x - 1.0f, y, z).x;
    float2 packed_center =    tex3D(jacobi_packed, x, y, z);
    float east =              tex3D(jacobi_packed, x + 1.0f, y, z).x;
    float north =             tex3D(jacobi_packed, x, y + 1.0f, z).x;
    float far =               tex3D(jacobi_packed, x, y, z + 1.0f).x;

    float u = omega_over_beta * 3.0f * packed_center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        packed_center.y) * omega_over_beta;

    ushort2 raw = make_ushort2(__float2half_rn(u),
                               __float2half_rn(packed_center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__device__ void ReadBlockAndHalo_32x6(int z, uint tx, uint ty, float2* smem)
{
    uint linear_index = ty * blockDim.x + tx;

    const uint smem_width = 48;

    uint sx =  linear_index % smem_width;
    uint sy1 = linear_index / smem_width;
    uint sy2 = sy1 + 4;

    int ix =  static_cast<int>(blockIdx.x * blockDim.x + sx) - 8;
    int iy1 = static_cast<int>(blockIdx.y * blockDim.y + sy1) - 1;
    int iy2 = static_cast<int>(blockIdx.y * blockDim.y + sy2) - 1;

    smem[sx + sy1 * smem_width] = tex3D(jacobi_packed, ix, iy1, z);
    smem[sx + sy2 * smem_width] = tex3D(jacobi_packed, ix, iy2, z);
}

__device__ void SaveToRegisters(float2* smem, uint si, uint bw, float* south,
                                float* west, float2* center, float* east,
                                float* north)
{
    __syncthreads();

    *south =  smem[si - bw].x;
    *west =   smem[si - 1].x;
    *center = smem[si];
    *east =   smem[si + 1].x;
    *north =  smem[si + bw].x;
}

__global__ void DampedJacobiKernel_smem_25d_32x6(float minus_square_cell_size,
                                                 float omega_over_beta,
                                                 uint3 volume_size)
{
    __shared__ float2 smem[384];

    const uint tx = threadIdx.x;
    const uint ty = threadIdx.y;

    const uint bw = blockDim.x + 16;
    const uint ox = blockIdx.x * blockDim.x + tx;
    const uint oy = blockIdx.y * blockDim.y + ty;

    const uint si = (ty + 1) * bw + tx + 8;

    float  south;
    float  west;
    float2 center;
    float  east;
    float  north;

    ReadBlockAndHalo_32x6(0, tx, ty, smem);
    SaveToRegisters(smem, si, bw, &south, &west, &center, &east, &north);

    ReadBlockAndHalo_32x6(1, tx, ty, smem);

    float t1 = omega_over_beta * 4.0f * center.x +
        (west + east + south + north + minus_square_cell_size * center.y) *
        omega_over_beta;
    float b = center.y;
    float near = t1;

    ushort2 raw;
    float far;

    for (uint i = 2; i < volume_size.z; i++) {
        SaveToRegisters(smem, si, bw, &south, &west, &center, &east, &north);
        ReadBlockAndHalo_32x6(i, tx, ty, smem);

        far = center.x * omega_over_beta;
        near = t1 + far;
        raw = make_ushort2(__float2half_rn(near), __float2half_rn(b));
        if (oy < volume_size.y)
            surf3Dwrite(raw, jacobi, ox * sizeof(ushort2), oy, i - 2,
                        hipBoundaryModeTrap);

        // t1 is now pointing to plane |i - 1|.
        t1 = omega_over_beta * 3.0f * center.x +
            (west + east + south + north + near + minus_square_cell_size *
            center.y) * omega_over_beta;
        b = center.y;
    }

    SaveToRegisters(smem, si, bw, &south, &west, &center, &east, &north);
    if (oy >= volume_size.y)
        return;

    near = center.x * omega_over_beta + t1;
    raw = make_ushort2(__float2half_rn(near),
                       __float2half_rn(b));
    surf3Dwrite(raw, jacobi, ox * sizeof(ushort2), oy, volume_size.z - 2,
                hipBoundaryModeTrap);

    t1 = omega_over_beta * 4.0f * center.x +
        (west + east + south + north + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    raw = make_ushort2(__float2half_rn(t1), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, ox * sizeof(ushort2), oy, volume_size.z - 1,
                hipBoundaryModeTrap);
}

__global__ void DampedJacobiKernel_smem_branch(float minus_square_cell_size,
                                               float omega_over_beta,
                                               uint3 volume_size)
{
    __shared__ float2 cached_block[1000];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int bw = blockDim.x + 2;
    int bh = blockDim.y + 2;

    int index = (threadIdx.z + 1) * bw * bh + (threadIdx.y + 1) * bw +
        threadIdx.x + 1;
    cached_block[index] = tex3D(jacobi_packed, x, y, z);

    if (threadIdx.x == 0)
        cached_block[index - 1] = x == 0 ?                       cached_block[index] : tex3D(jacobi_packed, x - 1, y, z);

    if (threadIdx.x == blockDim.x - 1)
        cached_block[index + 1] = x == volume_size.x - 1 ?       cached_block[index] : tex3D(jacobi_packed, x + 1, y, z);

    if (threadIdx.y == 0)
        cached_block[index - bw] = y == 0 ?                      cached_block[index] : tex3D(jacobi_packed, x, y - 1, z);

    if (threadIdx.y == blockDim.y - 1)
        cached_block[index + bw] = y == volume_size.y - 1 ?      cached_block[index] : tex3D(jacobi_packed, x, y + 1, z);

    if (threadIdx.z == 0)
        cached_block[index - bw * bh] = z == 0 ?                 cached_block[index] : tex3D(jacobi_packed, x, y, z - 1);

    if (threadIdx.z == blockDim.z - 1)
        cached_block[index + bw * bh] = z == volume_size.z - 1 ? cached_block[index] : tex3D(jacobi_packed, x, y, z + 1);

    __syncthreads();

    float  near =   cached_block[index - bw * bh].x;
    float  south =  cached_block[index - bw].x;
    float  west =   cached_block[index - 1].x;
    float2 center = cached_block[index];
    float  east =   cached_block[index + 1].x;
    float  north =  cached_block[index + bw].x;
    float  far =    cached_block[index + bw * bh].x;

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiKernel_smem_assist_thread(
    float minus_square_cell_size, float omega_over_beta, uint3 volume_size)
{
    // Shared memory solution with halo handled by assistant threads still
    // runs a bit slower than the texture-only way(less than 3ms on my GTX
    // 660Ti doing 40 times Jacobi).
    //
    // With the bank conflicts solved, I think the difference can be narrowed
    // down to around 1ms. But, it may say that the power of shared memory is
    // not as that great as expected, for Jacobi at least. Or maybe the texture
    // cache is truely really fast.

    const int cache_size = 1000;
    const int bd = 10;
    const int bh = 10;
    const int slice_stride = cache_size / bd;
    const int bw = slice_stride / bh;

    __shared__ float2 cached_block[cache_size];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = (threadIdx.z + 1) * slice_stride + (threadIdx.y + 1) * bw +
        threadIdx.x + 1;

    // Kernel runs faster if we place the normal fetch prior to the assistant
    // process.
    cached_block[index] = tex3D(jacobi_packed, x, y, z);

    int inner = 0;
    int inner_x = 0;
    int inner_y = 0;
    int inner_z = 0;
    switch (threadIdx.z) {
        case 0: {
            // near
            inner = (threadIdx.y + 1) * bw + threadIdx.x + 1;
            inner_x = x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z - 1;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
        }
        case 1: {
            // south
            inner = (threadIdx.y + 1) * slice_stride + threadIdx.x + 1;
            inner_x = x;
            inner_y = blockIdx.y * blockDim.y - 1;
            inner_z = blockIdx.z * blockDim.z + threadIdx.y;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
        }
        case 2: {
            // west
            inner = (threadIdx.x + 1) * slice_stride + (threadIdx.y + 1) * bw;

            // It's more efficient to put z in the inner-loop than y.
            inner_x = blockIdx.x * blockDim.x - 1;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.x;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
        }
        case 5:
            // east
            inner = (threadIdx.x + 1) * slice_stride + (threadIdx.y + 1) * bw +
                blockDim.x + 1;
            inner_x = blockIdx.x * blockDim.x + blockDim.x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.x;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
        case 6:
            // north
            inner = (threadIdx.y + 1) * slice_stride + (blockDim.y + 1) * bw +
                threadIdx.x + 1;
            inner_x = x;
            inner_y = blockIdx.y * blockDim.y + blockDim.y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.y;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
        case 7:
            // far
            inner = (blockDim.z + 1) * slice_stride + (threadIdx.y + 1) * bw +
                threadIdx.x + 1;
            inner_x = x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + blockDim.z;
            cached_block[inner] = tex3D(jacobi_packed, inner_x, inner_y,
                                        inner_z);
            break;
    }
    __syncthreads();

    float  near =   cached_block[index - slice_stride].x;
    float  south =  cached_block[index - bw].x;
    float  west =   cached_block[index - 1].x;
    float2 center = cached_block[index];
    float  east =   cached_block[index + 1].x;
    float  north =  cached_block[index + bw].x;
    float  far =    cached_block[index + slice_stride].x;

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiKernel_smem_faces_assist_thread(
    float minus_square_cell_size, float omega_over_beta, uint3 volume_size)
{
    const int cache_size = 512;
    const int bd = 8;
    const int bh = 8;
    const int slice_stride = cache_size / bd;
    const int bw = slice_stride / bh;

    __shared__ float2 cached_block[cache_size];
    __shared__ float cached_face_xyz0[bw * bh];
    __shared__ float cached_face_xyz1[bw * bh];
    __shared__ float cached_face_xzy0[bw * bd];
    __shared__ float cached_face_xzy1[bw * bd];
    __shared__ float cached_face_yzx0[bh * bd];
    __shared__ float cached_face_yzx1[bh * bd];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = threadIdx.z * slice_stride + threadIdx.y * bw + threadIdx.x;

    cached_block[index] = tex3D(jacobi_packed, x, y, z);

    int inner_x = 0;
    int inner_y = 0;
    int inner_z = 0;
    switch (threadIdx.z) {
        case 0: {
            // near
            inner_x = x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z - 1;
            cached_face_xyz0[blockDim.x * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
        }
        case 1: {
            // south
            inner_x = x;
            inner_y = blockIdx.y * blockDim.y - 1;
            inner_z = blockIdx.z * blockDim.z + threadIdx.y;
            cached_face_xzy0[blockDim.x * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
        }
        case 2: {
            // west
            inner_x = blockIdx.x * blockDim.x - 1;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.x;
            cached_face_yzx0[blockDim.y * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
        }
        case 5:
            // east
            inner_x = blockIdx.x * blockDim.x + blockDim.x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.x;
            cached_face_yzx1[blockDim.y * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
        case 6:
            // north
            inner_x = x;
            inner_y = blockIdx.y * blockDim.y + blockDim.y;
            inner_z = blockIdx.z * blockDim.z + threadIdx.y;
            cached_face_xzy1[blockDim.x * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
        case 7:
            // far
            inner_x = x;
            inner_y = y;
            inner_z = blockIdx.z * blockDim.z + blockDim.z;
            cached_face_xyz1[blockDim.x * threadIdx.y + threadIdx.x] =
                tex3D(jacobi_packed, inner_x, inner_y, inner_z).x;
            break;
    }
    __syncthreads();

    float2 center = cached_block[index];
    float near =  threadIdx.z == 0 ?              cached_face_xyz0[blockDim.x * threadIdx.y + threadIdx.x] : cached_block[index - slice_stride].x;
    float south = threadIdx.y == 0 ?              cached_face_xzy0[blockDim.x * threadIdx.z + threadIdx.x] : cached_block[index - bw].x;
    float west =  threadIdx.x == 0 ?              cached_face_yzx0[blockDim.y * threadIdx.y + threadIdx.z] : cached_block[index - 1].x;
    float east =  threadIdx.x == blockDim.x - 1 ? cached_face_yzx1[blockDim.y * threadIdx.y + threadIdx.z] : cached_block[index + 1].x;
    float north = threadIdx.y == blockDim.y - 1 ? cached_face_xzy1[blockDim.x * threadIdx.z + threadIdx.x] : cached_block[index + bw].x;
    float far =   threadIdx.z == blockDim.z - 1 ? cached_face_xyz1[blockDim.x * threadIdx.y + threadIdx.x] : cached_block[index + slice_stride].x;

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiKernel_smem_dedicated_assist_thread(
    float minus_square_cell_size, float omega_over_beta, uint3 volume_size)
{
    __shared__ float2 cached_block[1000];

    int x = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
    int y = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
    int z = blockIdx.z * (blockDim.z - 2) + threadIdx.z - 1;

    int index = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x;

    cached_block[index] = tex3D(jacobi_packed, x, y, z);

    __syncthreads();

    if (threadIdx.x < 1 || threadIdx.x > blockDim.x - 2 ||
            threadIdx.y < 1 || threadIdx.y > blockDim.y - 2 ||
            threadIdx.z < 1 || threadIdx.z > blockDim.z - 2)
        return;

    float2 center = cached_block[index];
    float near =    cached_block[index - blockDim.x * blockDim.y].x;
    float south =   cached_block[index - blockDim.x].x;
    float west =    cached_block[index - 1].x;
    float east =    cached_block[index + 1].x;
    float north =   cached_block[index + blockDim.x].x;
    float far =     cached_block[index + blockDim.x * blockDim.y].x;

    float u = omega_over_beta * 3.0f * center.x +
        (west + east + south + north + far + near + minus_square_cell_size *
        center.y) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(center.y));
    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

__global__ void DampedJacobiKernel_smem_no_halo_storage(
    float minus_square_cell_size, float omega_over_beta, uint3 volume_size)
{
    __shared__ float2 cached_block[512];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x;

    cached_block[index] = tex3D(jacobi_packed, x, y, z);
    __syncthreads();

    float center = cached_block[index].x;
    float near =  threadIdx.z == 0 ?              (z == 0 ?                 center : tex3D(jacobi_packed, x, y, z - 1.0f).x) : cached_block[index - blockDim.x * blockDim.y].x;
    float south = threadIdx.y == 0 ?              (y == 0 ?                 center : tex3D(jacobi_packed, x, y - 1.0f, z).x) : cached_block[index - blockDim.x].x;
    float west =  threadIdx.x == 0 ?              (x == 0 ?                 center : tex3D(jacobi_packed, x - 1.0f, y, z).x) : cached_block[index - 1].x;
    float east =  threadIdx.x == blockDim.x - 1 ? (x == volume_size.x - 1 ? center : tex3D(jacobi_packed, x + 1.0f, y, z).x) : cached_block[index + 1].x;
    float north = threadIdx.y == blockDim.y - 1 ? (y == volume_size.y - 1 ? center : tex3D(jacobi_packed, x, y + 1.0f, z).x) : cached_block[index + blockDim.x].x;
    float far =   threadIdx.z == blockDim.z - 1 ? (z == volume_size.z - 1 ? center : tex3D(jacobi_packed, x, y, z + 1.0f).x) : cached_block[index + blockDim.x * blockDim.y].x;

    float b_center = cached_block[index].y;

    float u = omega_over_beta * 3.0f * center +
        (west + east + south + north + far + near + minus_square_cell_size *
        b_center) * omega_over_beta;
    ushort2 raw = make_ushort2(__float2half_rn(u), __float2half_rn(b_center));

    surf3Dwrite(raw, jacobi, x * sizeof(ushort2), y, z, hipBoundaryModeTrap);
}

// =============================================================================

void LaunchDampedJacobi(hipArray* dest_array, hipArray* source_array,
                        float minus_square_cell_size, float omega_over_beta,
                        uint3 volume_size, BlockArrangement* ba)
{
    hipChannelFormatDesc desc;
    hipGetChannelDesc(&desc, dest_array);
    hipError_t result = cudaBindSurfaceToArray(&jacobi, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipGetChannelDesc(&desc, source_array);
    jacobi_packed.normalized = false;
    jacobi_packed.filterMode = hipFilterModePoint;
    jacobi_packed.addressMode[0] = hipAddressModeClamp;
    jacobi_packed.addressMode[1] = hipAddressModeClamp;
    jacobi_packed.addressMode[2] = hipAddressModeClamp;
    jacobi_packed.channelDesc = desc;

    result = hipBindTextureToArray(&jacobi_packed, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    bool smem = false;
    bool smem_25d = true;
    if (smem_25d) {
        dim3 block(32, 6, 1);
        dim3 grid((volume_size.x + block.x - 1) / block.x,
                  (volume_size.y + block.y - 1) / block.y,
                  1);
        DampedJacobiKernel_smem_25d_32x6<<<grid, block>>>(
            minus_square_cell_size, omega_over_beta, volume_size);
    } else if (smem) {
        dim3 block(8, 8, 8);
        dim3 grid((volume_size.x + block.x - 1) / block.x,
                  (volume_size.y + block.y - 1) / block.y,
                  (volume_size.z + block.z - 1) / block.z);
        DampedJacobiKernel_smem_assist_thread<<<grid, block>>>(
            minus_square_cell_size, omega_over_beta, volume_size);
    } else {
        dim3 block;
        dim3 grid;
        ba->Arrange(&block, &grid, volume_size);
        DampedJacobiKernel<<<grid, block>>>(minus_square_cell_size,
                                            omega_over_beta, volume_size);
    }
    

    hipUnbindTexture(&jacobi_packed);
}
