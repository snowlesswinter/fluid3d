#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/aux_buffer_manager.h"
#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/multi_precision_texture.cuh"

surface<void, cudaSurfaceType3D> surf;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_0;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_1;
texture<float, hipTextureType3D, hipReadModeElementType> texf;
texture<long2, hipTextureType3D, hipReadModeElementType> texd;

struct UpperBoundaryHandlerNeumann
{
    __device__ void HandleUpperBoundary(float* north, float center, int y,
                                        int height)
    {
    }
};

struct UpperBoundaryHandlerOutflow
{
    __device__ void HandleUpperBoundary(float* north, float center, int y,
                                        int height)
    {
        if (y == height - 1) {
            if (center > 0.0f)
                *north = -center;
            else
                *north = 0.0f;
        }
    }
};

// =============================================================================

template <typename FPType, typename UpperBoundaryHandler>
__global__ void ApplyStencilKernel(uint3 volume_size,
                                   UpperBoundaryHandler handler)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    typedef typename Tex3d<FPType>::ValType ValType;

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    Tex3d<FPType> t3d;
    ValType near   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z - 1.0f);
    ValType south  = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y - 1.0f, z);
    ValType west   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x - 1.0f, y,        z);
    ValType center = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z);
    ValType east   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x + 1.0f, y,        z);
    ValType north  = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y + 1.0f, z);
    ValType far    = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z + 1.0f);

    //handler.HandleUpperBoundary(&north, center, y, volume_size.y);

    // NOTE: The coefficient 'h^2' is premultiplied in the divergence kernel.
    float v = (north + south + east + west + far + near - 6.0f * center);
    auto r = __float2half_rn(v);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

__global__ void ScaleVectorKernel(double* coef, uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    double e1 = tex3D(tex_1, x, y, z);

    auto r = __float2half_rn(*coef * e1);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

__global__ void ScaledAddKernel(double* coef, double sign, uint3 volume_size)
{
    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    double e0 = tex3D(tex_0, x, y, z);
    double e1 = tex3D(tex_1, x, y, z);

    auto r = __float2half_rn(e0 + *coef * sign * e1);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <typename FPType>
struct SchemeDefault
{
    __device__ FPType Load(uint i, uint row_stride, uint slice_stride)
    {
        uint z = i / slice_stride;
        uint y = (i % slice_stride) / row_stride;
        uint x = i % row_stride;

        FPType ��0 = tex3D(tex_0, static_cast<float>(x), static_cast<float>(y),
                          static_cast<float>(z));
        FPType ��1 = tex3D(tex_1, static_cast<float>(x), static_cast<float>(y),
                          static_cast<float>(z));
        return ��0 * ��1;
    }
    __device__ void Save(FPType* dest, FPType result)
    {
        *dest = result;
    }
};

template <typename FPType>
struct SchemeAlpha : public SchemeDefault<FPType>
{
    __device__ void Save(FPType* dest, FPType result)
    {
        if (result > 0.00000001f || result < -0.00000001f)
            *dest = *rho_ / result;
        else
            *dest = 0.0f;
    }

    FPType* rho_;
};

template <typename FPType>
struct SchemeBeta : public SchemeDefault<FPType>
{
    __device__ void Save(FPType* dest, FPType result)
    {
        *dest = result;

        FPType t = *rho_;
        if (t > 0.00000001f || t < -0.00000001f)
            *beta_ = result / t;
        else
            *beta_ = 0;
    }

    FPType* rho_;
    FPType* beta_;
};

#include "volume_reduction.cuh"

// =============================================================================

void LaunchApplyStencil(hipArray* aux, hipArray* search, bool outflow,
                        uint3 volume_size, BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
        return;

    auto bound = SelectiveBind(search, false, hipFilterModePoint,
                               hipAddressModeClamp, &tex, &texf, &texd);
    if (!bound)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeRowScan(&block, &grid, volume_size);

    UpperBoundaryHandlerOutflow outflow_handler;
    UpperBoundaryHandlerNeumann neumann_handler;
    if (outflow)
        ApplyStencilKernel<ushort><<<grid, block>>>(volume_size, outflow_handler);
    else
        ApplyStencilKernel<ushort><<<grid, block>>>(volume_size, neumann_handler);

    DCHECK_KERNEL();
}

void LaunchComputeAlpha(double* alpha, double* rho, hipArray* vec0,
                        hipArray* vec1, uint3 volume_size,
                        BlockArrangement* ba, AuxBufferManager* bm)
{
    auto bound_0 = BindHelper::Bind(&tex_0, vec0, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_0.error() != hipSuccess)
        return;

    auto bound_1 = BindHelper::Bind(&tex_1, vec1, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_1.error() != hipSuccess)
        return;

    SchemeAlpha<double> scheme;
    scheme.rho_ = rho;
    ReduceVolume(alpha, scheme, volume_size, ba, bm);

    DCHECK_KERNEL();
}

void LaunchComputeRho(double* rho, hipArray* search, hipArray* residual,
                      uint3 volume_size, BlockArrangement* ba,
                      AuxBufferManager* bm)
{
    auto bound_0 = BindHelper::Bind(&tex_0, search, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_0.error() != hipSuccess)
        return;

    auto bound_1 = BindHelper::Bind(&tex_1, residual, false,
                                    hipFilterModePoint, hipAddressModeClamp);
    if (bound_1.error() != hipSuccess)
        return;

    SchemeDefault<double> scheme;
    ReduceVolume(rho, scheme, volume_size, ba, bm);

    DCHECK_KERNEL();
}

void LaunchComputeRhoAndBeta(double* beta, double* rho_new, double* rho,
                             hipArray* vec0, hipArray* vec1,
                             uint3 volume_size, BlockArrangement* ba,
                             AuxBufferManager* bm)
{
    
    auto bound_0 = BindHelper::Bind(&tex_0, vec0, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_0.error() != hipSuccess)
        return;

    auto bound_1 = BindHelper::Bind(&tex_1, vec1, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_1.error() != hipSuccess)
        return;

    SchemeBeta<double> scheme;
    scheme.beta_ = beta;
    scheme.rho_ = rho;
    ReduceVolume(rho_new, scheme, volume_size, ba, bm);

    DCHECK_KERNEL();
}

void LaunchScaledAdd(hipArray* dest, hipArray* v0, hipArray* v1,
                     double* coef, double sign, uint3 volume_size,
                     BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    auto bound_1 = BindHelper::Bind(&tex_1, v1, false, hipFilterModePoint,
                                    hipAddressModeClamp);
    if (bound_1.error() != hipSuccess)
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeRowScan(&block, &grid, volume_size);
    if (v0) {
        auto bound_0 = BindHelper::Bind(&tex_0, v0, false, hipFilterModePoint,
                                        hipAddressModeClamp);
        if (bound_0.error() != hipSuccess)
            return;

        ScaledAddKernel<<<grid, block>>>(coef, sign, volume_size);
    } else {
        ScaleVectorKernel<<<grid, block>>>(coef, volume_size);
    }

    DCHECK_KERNEL();
}
