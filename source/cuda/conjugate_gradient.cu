#include "hip/hip_runtime.h"
//
// Hypermorph - Fluid Simulator for interactive applications
// Copyright (C) 2016. JIANWEN TAN(jianwen.tan@gmail.com). All rights reserved.
//
// Hypermorph license (* see part 1 below)
// This software is provided 'as-is', without any express or implied
// warranty.  In no event will the authors be held liable for any damages
// arising from the use of this software.
//
// Permission is granted to anyone to use this software for any purpose,
// including commercial applications, and to alter it and redistribute it
// freely, subject to the following restrictions:
//
// 1. The origin of this software must not be misrepresented; you must not
//    claim that you wrote the original software. Acknowledgement of the
//    original author is required if you publish this in a paper, or use it
//    in a product.
// 2. Altered source versions must be plainly marked as such, and must not be
//    misrepresented as being the original software.
// 3. This notice may not be removed or altered from any source distribution.

#include <cassert>
#include <functional>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

#include "cuda/aux_buffer_manager.h"
#include "cuda/block_arrangement.h"
#include "cuda/cuda_common_host.h"
#include "cuda/cuda_common_kern.h"
#include "cuda/cuda_debug.h"
#include "cuda/mem_piece.h"
#include "cuda/multi_precision.cuh"

surface<void, cudaSurfaceType3D> surf;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_0;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> tex_1;
texture<float, hipTextureType3D, hipReadModeElementType> texf;
texture<float, hipTextureType3D, hipReadModeElementType> texf_0;
texture<float, hipTextureType3D, hipReadModeElementType> texf_1;
texture<long2, hipTextureType3D, hipReadModeElementType> texd;
texture<long2, hipTextureType3D, hipReadModeElementType> texd_0;
texture<long2, hipTextureType3D, hipReadModeElementType> texd_1;

struct UpperBoundaryHandlerNeumann
{
    __device__ void HandleUpperBoundary(float* north, float center, int y,
                                        int height)
    {
    }
};

struct UpperBoundaryHandlerOutflow
{
    __device__ void HandleUpperBoundary(float* north, float center, int y,
                                        int height)
    {
        if (y == height - 1) {
            if (center > 0.0f)
                *north = -center;
            else
                *north = 0.0f;
        }
    }
};

// =============================================================================

template <typename FPType, typename UpperBoundaryHandler>
__global__ void ApplyStencilKernel(uint3 volume_size,
                                   UpperBoundaryHandler handler)
{
    using ValType = typename Tex3d<FPType>::ValType;

    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    Tex3d<FPType> t3d;
    ValType near   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z - 1.0f);
    ValType south  = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y - 1.0f, z);
    ValType west   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x - 1.0f, y,        z);
    ValType center = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z);
    ValType east   = t3d(TexSel<FPType>::Tex(tex, texf, texd), x + 1.0f, y,        z);
    ValType north  = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y + 1.0f, z);
    ValType far    = t3d(TexSel<FPType>::Tex(tex, texf, texd), x,        y,        z + 1.0f);

    //handler.HandleUpperBoundary(&north, center, y, volume_size.y);

    // NOTE: The coefficient 'h^2' is premultiplied in the divergence kernel.
    float v = (north + south + east + west + far + near - 6.0f * center);
    auto r = __float2half_rn(v);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <typename FPType, typename T>
__global__ void ScaleVectorKernel(T* coef, uint3 volume_size)
{
    using ValType = typename Tex3d<FPType>::ValType;

    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    Tex3d<FPType> t3d;
    ValType e1 = t3d(TexSel<FPType>::Tex(tex_1, texf_1, texd_1), x, y, z);

    auto r = __float2half_rn(*coef * e1);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <typename FPType, typename T>
__global__ void ScaledAddKernel(T* coef, float sign, uint3 volume_size)
{
    using ValType = typename Tex3d<FPType>::ValType;

    uint x = VolumeX();
    uint y = VolumeY();
    uint z = VolumeZ();

    if (x >= volume_size.x || y >= volume_size.y || z >= volume_size.z)
        return;

    Tex3d<FPType> t3d;
    ValType e0 = t3d(TexSel<FPType>::Tex(tex_0, texf_0, texd_0), x, y, z);
    ValType e1 = t3d(TexSel<FPType>::Tex(tex_1, texf_1, texd_1), x, y, z);

    auto r = __float2half_rn(e0 + *coef * sign * e1);
    surf3Dwrite(r, surf, x * sizeof(r), y, z, hipBoundaryModeTrap);
}

template <typename FPType>
struct SchemeDefault
{
    __device__ FPType Load(uint i, uint row_stride, uint slice_stride)
    {
        uint z = i / slice_stride;
        uint y = (i % slice_stride) / row_stride;
        uint x = i % row_stride;

        float xf = static_cast<float>(x);
        float yf = static_cast<float>(y);
        float zf = static_cast<float>(z);

        using ValType = typename Tex3d<FPType>::ValType;

        Tex3d<FPType> t3d;
        FPType ��0 = t3d(TexSel<FPType>::Tex(tex_0, texf_0, texd_0), xf, yf, zf);
        FPType ��1 = t3d(TexSel<FPType>::Tex(tex_1, texf_1, texd_1), xf, yf, zf);
        return ��0 * ��1;
    }
    __device__ void Save(FPType* dest, FPType result)
    {
        *dest = result;
    }

    __host__ void Init() {}
};

template <typename FPType>
struct SchemeAlpha : public SchemeDefault<FPType>
{
    __device__ void Save(FPType* dest, FPType result)
    {
        if (result > 0.00000001f || result < -0.00000001f)
            *dest = *rho_ / result;
        else
            *dest = 0.0f;
    }

    template <typename ScalarPackType>
    __host__ void Init(const ScalarPackType& rho)
    {
        AssignScalar<FPType>(&rho_, rho);
    }

    FPType* rho_;
};

template <typename FPType>
struct SchemeBeta : public SchemeDefault<FPType>
{
    __device__ void Save(FPType* dest, FPType result)
    {
        *dest = result;

        FPType t = *rho_;
        if (t > 0.00000001f || t < -0.00000001f)
            *beta_ = result / t;
        else
            *beta_ = 0;
    }

    template <typename ScalarPackType>
    __host__ void Init(const ScalarPackType& beta, const ScalarPackType& rho)
    {
        AssignScalar<FPType>(&beta_, beta);
        AssignScalar<FPType>(&rho_, rho);
    }

    FPType* rho_;
    FPType* beta_;
};

#include "volume_reduction.cuh"

// =============================================================================

template <typename FPType, typename TupleType>
struct ApplyStencilWrapper
{
    static void Invoke(const TupleType& params)
    {
        dim3 grid = std::get<0>(params);
        dim3 block = std::get<1>(params);
        uint3 volume_size = std::get<2>(params);
        bool outflow = std::get<3>(params);

        UpperBoundaryHandlerOutflow outflow_handler;
        UpperBoundaryHandlerNeumann neumann_handler;
        if (outflow)
            ApplyStencilKernel<FPType><<<grid, block>>>(volume_size,
                                                        outflow_handler);
        else
            ApplyStencilKernel<FPType><<<grid, block>>>(volume_size,
                                                        neumann_handler);
    }
};

template <typename FPType, typename TupleType>
struct ScaledAddWrapper
{
    static void Invoke(const TupleType& params)
    {
        using SampleType = typename Tex3d<FPType>::ValType;

        dim3 grid = std::get<0>(params);
        dim3 block = std::get<1>(params);
        MemPiece coef = std::get<2>(params);
        float sign = std::get<3>(params);
        uint3 volume_size = std::get<4>(params);

        ScaledAddKernel<FPType><<<grid, block>>>(coef.AsType<SampleType>(),
                                                 sign, volume_size);
    }
};

template <typename FPType, typename TupleType>
struct ScaleVectorWrapper
{
    static void Invoke(const TupleType& params)
    {
        using SampleType = typename Tex3d<FPType>::ValType;

        dim3 grid = std::get<0>(params);
        dim3 block = std::get<1>(params);
        MemPiece coef = std::get<2>(params);
        uint3 volume_size = std::get<3>(params);

        ScaleVectorKernel<FPType><<<grid, block>>>(coef.AsType<SampleType>(),
                                                   volume_size);
    }
};

template <template <typename T, typename P> class Kern, typename BoundType,
    typename TupleType>
void InvokeKernel(const BoundType& bound, const TupleType& params)
{
    using FPType = typename TexTraits<typename BoundType::ThisTexType>::EleType;
    if (bound.Bound()) {
        Kern<FPType, TupleType>::Invoke(params);
        return;
    }

    using NextBoundType = typename BoundType::BaseType;
    InvokeKernel<Kern, NextBoundType>(static_cast<const NextBoundType&>(bound),
                                      params);
}

void LaunchApplyStencil(hipArray* aux, hipArray* search, bool outflow,
                        uint3 volume_size, BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, aux) != hipSuccess)
        return;

    auto bound = SelectiveBind(search, false, hipFilterModePoint,
                               hipAddressModeClamp, &tex, &texf, &texd);
    if (!bound.Succeeded())
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeRowScan(&block, &grid, volume_size);

    auto params = std::make_tuple(grid, block, volume_size, outflow);

    InvokeKernel<ApplyStencilWrapper>(bound, params);
    DCHECK_KERNEL();
}

void LaunchComputeAlpha(const MemPiece& alpha, const MemPiece& rho,
                        hipArray* vec0, hipArray* vec1, uint3 volume_size,
                        BlockArrangement* ba, AuxBufferManager* bm)
{
    auto bound_0 = SelectiveBind(vec0, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_0, &texf_0,
                                 &texd_0);
    if (!bound_0.Succeeded())
        return;

    auto bound_1 = SelectiveBind(vec1, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_1, &texf_1,
                                 &texd_1);
    if (!bound_1.Succeeded())
        return;

    ScalarPack alpha_typed = CreateScalarPack(alpha);
    ScalarPack rho_typed   = CreateScalarPack(rho);

    InvokeReduction<SchemeAlpha>(alpha_typed, volume_size, ba, bm, rho_typed);
    DCHECK_KERNEL();
}

void LaunchComputeRho(const MemPiece& rho, hipArray* search,
                      hipArray* residual, uint3 volume_size,
                      BlockArrangement* ba, AuxBufferManager* bm)
{
    auto bound_0 = SelectiveBind(search, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_0, &texf_0,
                                 &texd_0);
    if (!bound_0.Succeeded())
        return;

    auto bound_1 = SelectiveBind(residual, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_1, &texf_1,
                                 &texd_1);
    if (!bound_1.Succeeded())
        return;

    ScalarPack rho_typed = CreateScalarPack(rho);

    InvokeReduction<SchemeDefault>(rho_typed, volume_size, ba, bm);
    DCHECK_KERNEL();
}

void LaunchComputeRhoAndBeta(const MemPiece& beta, const MemPiece& rho_new,
                             const MemPiece& rho, hipArray* vec0,
                             hipArray* vec1, uint3 volume_size,
                             BlockArrangement* ba, AuxBufferManager* bm)
{
    auto bound_0 = SelectiveBind(vec0, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_0, &texf_0,
                                 &texd_0);
    if (!bound_0.Succeeded())
        return;

    auto bound_1 = SelectiveBind(vec1, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_1, &texf_1,
                                 &texd_1);
    if (!bound_1.Succeeded())
        return;

    ScalarPack rho_new_typed = CreateScalarPack(rho_new);
    ScalarPack beta_typed    = CreateScalarPack(beta);
    ScalarPack rho_typed     = CreateScalarPack(rho);

    InvokeReduction<SchemeBeta>(rho_new_typed, volume_size, ba, bm, beta_typed,
                                rho_typed);
    DCHECK_KERNEL();
}

void LaunchScaledAdd(hipArray* dest, hipArray* v0, hipArray* v1,
                     const MemPiece& coef, float sign, uint3 volume_size,
                     BlockArrangement* ba)
{
    if (BindCudaSurfaceToArray(&surf, dest) != hipSuccess)
        return;

    auto bound_1 = SelectiveBind(v1, false, hipFilterModePoint,
                                 hipAddressModeClamp, &tex_1, &texf_1,
                                 &texd_1);
    if (!bound_1.Succeeded())
        return;

    dim3 block;
    dim3 grid;
    ba->ArrangeRowScan(&block, &grid, volume_size);
    if (v0) {
        auto bound_0 = SelectiveBind(v0, false, hipFilterModePoint,
                                     hipAddressModeClamp, &tex_0, &texf_0,
                                     &texd_0);
        if (!bound_0.Succeeded())
            return;

        auto params = std::make_tuple(grid, block, coef, sign, volume_size);
        InvokeKernel<ScaledAddWrapper>(bound_0, params);
    } else {
        auto params = std::make_tuple(grid, block, coef, volume_size);
        InvokeKernel<ScaleVectorWrapper>(bound_1, params);
    }

    DCHECK_KERNEL();
}
