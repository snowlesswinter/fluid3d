#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "opengl/glew.h"

#include <hip/hip_vector_types.h>

texture<float1, hipTextureType3D, hipReadModeElementType> in_tex;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_coarse;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_fine;

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float1 cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc.x;
}

__global__ void ProlongatePackedKernel(float4* out_data,
                                       int num_of_blocks_per_slice,
                                       int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 c = make_float3(x, y, z);
    c *= 0.5f;

    int odd_x = x - ((x >> 1) << 1);
    int odd_y = y - ((y >> 1) << 1);
    int odd_z = z - ((z >> 1) << 1);

    float t_x = -1.0f * (1 - odd_x) * 0.08333333f;
    float t_y = -1.0f * (1 - odd_y) * 0.08333333f;
    float t_z = -1.0f * (1 - odd_z) * 0.08333333f;

    float3 t_c = make_float3(c.x + t_x, c.y + t_y, c.z + t_z);
    float4 result_float = tex3D(prolongate_coarse, t_c.x, t_c.y, t_c.z);

    float3 f_coord = make_float3(float(x) + 0.5f, float(y) + 0.5f,
                                 float(z) + 0.5f);

    float4 original = tex3D(prolongate_fine, f_coord.x, f_coord.y, f_coord.z);
    float4 result = make_float4(original.x + result_float.x, original.y, 0, 0);

    out_data[index] = result;
}

// =============================================================================

void LaunchProlongatePacked(float4* dest_array, hipArray* coarse_array,
                            hipArray* fine_array, int3 volume_size_fine)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    prolongate_coarse.normalized = false;
    prolongate_coarse.filterMode = hipFilterModeLinear;
    prolongate_coarse.addressMode[0] = hipAddressModeClamp;
    prolongate_coarse.addressMode[1] = hipAddressModeClamp;
    prolongate_coarse.addressMode[2] = hipAddressModeClamp;
    prolongate_coarse.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&prolongate_coarse,
                                                coarse_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    prolongate_fine.normalized = false;

    // TODO: Disabling the linear filter mode may slightly speed up the kernel.
    prolongate_fine.filterMode = hipFilterModeLinear;
    prolongate_fine.addressMode[0] = hipAddressModeClamp;
    prolongate_fine.addressMode[1] = hipAddressModeClamp;
    prolongate_fine.addressMode[2] = hipAddressModeClamp;
    prolongate_fine.channelDesc = desc;

    result = hipBindTextureToArray(&prolongate_fine, fine_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int3 volume_size = volume_size_fine;
    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ProlongatePackedKernel<<<grid, block>>>(dest_array, num_of_blocks_per_slice,
                                            slice_stride, volume_size);

    hipUnbindTexture(&prolongate_fine);
    hipUnbindTexture(&prolongate_coarse);
}
