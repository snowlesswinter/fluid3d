#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "glew.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cuda_gl_interop.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include "opengl/gl_texture.h"
#include "graphics_resource.h"

// hipReadModeNormalizedFloat
// hipReadModeElementType
texture<unsigned short, hipTextureType3D, hipReadModeNormalizedFloat> in_tex;

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc;
}

CudaCore::CudaCore()
{

}

CudaCore::~CudaCore()
{
    hipDeviceReset();
}

bool CudaCore::Init()
{
    int dev_id = findCudaGLDevice(0, nullptr);
    hipDeviceProp_t prop = {0};
    hipGetDeviceProperties(&prop, dev_id);
    return 0;
}

int CudaCore::RegisterGLImage(const GLTexture& texture,
                              GraphicsResource* graphics_res)
{
    hipError_t result = hipGraphicsGLRegisterImage(
        graphics_res->Receive(), texture.handle(), texture.target(),
        hipGraphicsRegisterFlagsNone);
    assert(result == hipSuccess);
    return result == hipSuccess ? 0 : -1;
}

void CudaCore::UnregisterGLImage(GraphicsResource* graphics_res)
{
    hipGraphicsUnregisterResource(graphics_res->resource());
}

void CudaCore::Absolute(GraphicsResource* graphics_res, unsigned int aa)
{
    assert(graphics_res);
    if (!graphics_res)
        return;

    float* out_data = nullptr;
    size_t out_size = 0;
    hipError_t result1 = hipMalloc((void**)&out_data, 128 * 128 * 128 * 4);
    assert(result1 == hipSuccess);
    if (result1 != hipSuccess)
        return;
    //hipGraphicsResource_t res1;
    //hipError_t result1 = hipGraphicsGLRegisterBuffer(
    //    &res1, aa, hipGraphicsRegisterFlagsNone);
    //
    //result1 = hipGraphicsMapResources(1, &res1);
    //assert(result1 == hipSuccess);
    //if (result1 != hipSuccess)
    //    return;

    //result1 = hipGraphicsResourceGetMappedPointer(
    //    reinterpret_cast<void**>(&out_data), &out_size, res1);
    //assert(result1 == hipSuccess);
    //if (result1 != hipSuccess)
    //    return;

    hipGraphicsResource_t res = graphics_res->resource();
    hipError_t result = hipGraphicsMapResources(1, &res);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipArray* dest_array = nullptr;
    result = hipGraphicsSubResourceGetMappedArray(&dest_array, res, 0, 0);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    hipChannelFormatDesc desc = cudaCreateChannelDescHalf();
    in_tex.normalized = true;
    in_tex.filterMode = hipFilterModeLinear;
    in_tex.addressMode[0] = hipAddressModeClamp;
    in_tex.addressMode[1] = hipAddressModeClamp;
    in_tex.addressMode[2] = hipAddressModeClamp;
    in_tex.channelDesc = desc;
    
    result = hipBindTextureToArray(&in_tex, dest_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, 8);
    dim3 grid(16, 16, 16);
    AbsoluteKernel<<<grid, block>>>(out_data, 128, 128, 128);

    result = hipGetLastError();
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    float* a = new float[128 * 128 * 128];
    result = hipMemcpy(a, out_data, 128 * 128 * 128 * 4, hipMemcpyDeviceToHost);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    double p = 0;
    double sum = 0;
    for (int i = 0; i < 128; i++)
    {
        for (int j = 0; j < 128; j++)
        {
            for (int k = 0; k < 128; k++)
            {
                p = a[i * 128 * 128 + j * 128 + k];
                sum += p;
            }
        }
    }

    hipUnbindTexture(&in_tex);
    hipGraphicsUnmapResources(1, &res);
}