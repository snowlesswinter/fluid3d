#include "hip/hip_runtime.h"
#include "cuda_core.h"

#include <cassert>

#include "opengl/glew.h"

#include <hip/hip_vector_types.h>

// hipReadModeNormalizedFloat
// hipReadModeElementType
texture<float1, hipTextureType3D, hipReadModeElementType> in_tex;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_coarse;
texture<float4, hipTextureType3D, hipReadModeElementType> prolongate_fine;
texture<float4, hipTextureType3D, hipReadModeElementType> advect_velocity;
texture<float, hipTextureType3D, hipReadModeElementType> advect_source;
texture<float4, hipTextureType3D, hipReadModeElementType> buoyancy_velocity;
texture<float, hipTextureType3D, hipReadModeElementType> buoyancy_temperature;
texture<float, hipTextureType3D, hipReadModeElementType> impulse_original;
texture<float4, hipTextureType3D, hipReadModeElementType> divergence_velocity;

__global__ void RoundPassedKernel(int* dest_array, int round, int x)
{
    dest_array[0] = x * x - round * round;
}

__global__ void AbsoluteKernel(float* out_data, int w, int h, int d)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;
    int index = block_offset * blockDim.x*blockDim.y*blockDim.z +
        blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
    float3 coord;
    coord.x = (float(blockIdx.x) * blockDim.x + threadIdx.x + 0.5f) / w;
    coord.y = (float(blockIdx.y) * blockDim.y + threadIdx.y + 0.5f) / h;
    coord.z = (float(blockIdx.z) * blockDim.z + threadIdx.x + 0.5f) / d;

    float1 cc = tex3D(in_tex, coord.x, coord.y, coord.z);
    out_data[index] = cc.x;
}

__global__ void ProlongatePackedKernel(float4* out_data,
                                       int num_of_blocks_per_slice,
                                       int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 c = make_float3(x, y, z);
    c *= 0.5f;

    int odd_x = x - ((x >> 1) << 1);
    int odd_y = y - ((y >> 1) << 1);
    int odd_z = z - ((z >> 1) << 1);

    float t_x = -1.0f * (1 - odd_x) * 0.08333333f;
    float t_y = -1.0f * (1 - odd_y) * 0.08333333f;
    float t_z = -1.0f * (1 - odd_z) * 0.08333333f;

    float3 t_c = make_float3(c.x + t_x, c.y + t_y, c.z + t_z);
    float4 result_float = tex3D(prolongate_coarse, t_c.x, t_c.y, t_c.z);

    float3 f_coord = make_float3(float(x) + 0.5f, float(y) + 0.5f,
                                 float(z) + 0.5f);

    float4 original = tex3D(prolongate_fine, f_coord.x, f_coord.y, f_coord.z);
    float4 result = make_float4(original.x + result_float.x, original.y, 0, 0);

    out_data[index] = result;
}

__global__ void AdvectVelocityKernel(float4* out_data, float time_step,
                                     float dissipation,
                                     int num_of_blocks_per_slice,
                                     int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced =
        coord - time_step * make_float3(velocity.x, velocity.y, velocity.z);

    out_data[index] = dissipation * tex3D(advect_velocity, back_traced.x,
                                          back_traced.y, back_traced.z);
}

__global__ void AdvectKernel(float* out_data, float time_step,
                             float dissipation, int num_of_blocks_per_slice,
                             int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced =
        coord - time_step * make_float3(velocity.x, velocity.y, velocity.z);

    out_data[index] = dissipation * tex3D(advect_source, back_traced.x,
                                          back_traced.y, back_traced.z);
}

__global__ void ApplyBuoyancyKernel(float4* out_data, float time_step,
                                    float ambient_temperature,
                                    float accel_factor, float gravity,
                                    int num_of_blocks_per_slice,
                                    int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(buoyancy_velocity, coord.x, coord.y, coord.z);
    float t = tex3D(buoyancy_temperature, coord.x, coord.y, coord.z);

    out_data[index] = velocity;
    if (t > ambient_temperature)
        out_data[index] += time_step * ((t - ambient_temperature) *
            accel_factor - gravity) * make_float4(0.0f, 1.0f, 0.0f, 0.0f);
}

__global__ void ApplyImpulseKernel(float* out_data, float3 center_point,
                                   float3 hotspot, float radius, float value,
                                   int num_of_blocks_per_slice,
                                   int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float original = tex3D(impulse_original, coord.x, coord.y, coord.z);

    if (coord.x > 1.0f && coord.y < 3.0f)
    {
        float2 diff = make_float2(coord.x, coord.z) -
            make_float2(center_point.x, center_point.z);
        float d = hypotf(diff.x, diff.y);
        if (d < radius)
        {
            diff = make_float2(coord.x, coord.z) -
                make_float2(hotspot.x, hotspot.z);
            float scale = (radius - hypotf(diff.x, diff.y)) / radius;
            scale = max(scale, 0.5f);
            out_data[index] = scale * value;
            return;
        }
    }

    out_data[index] = original;
}

__global__ void ComputeDivergenceKernel(float4* out_data,
                                        float half_inverse_cell_size,
                                        int num_of_blocks_per_slice,
                                        int slice_stride, int3 volume_size)
{
    int block_offset = gridDim.x * gridDim.y * blockIdx.z +
        gridDim.x * blockIdx.y + blockIdx.x;

    int x = threadIdx.z * blockDim.x + threadIdx.x;
    int z = block_offset / num_of_blocks_per_slice;
    int y = (block_offset - z * num_of_blocks_per_slice) * blockDim.y +
        threadIdx.y;

    int index = slice_stride * z + volume_size.x * y + x;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;

    float4 near = tex3D(divergence_velocity, coord.x, coord.y, coord.z - 1.0f);
    float4 south = tex3D(divergence_velocity, coord.x, coord.y - 1.0f, coord.z);
    float4 west = tex3D(divergence_velocity, coord.x - 1.0f, coord.y, coord.z);
    float4 center = tex3D(divergence_velocity, coord.x, coord.y, coord.z);
    float4 east = tex3D(divergence_velocity, coord.x + 1.0f, coord.y, coord.z);
    float4 north = tex3D(divergence_velocity, coord.x, coord.y + 1.0f, coord.z);
    float4 far = tex3D(divergence_velocity, coord.x, coord.y, coord.z + 1.0f);

    float diff_ew = east.x - west.x;
    float diff_ns = north.y - south.y;
    float diff_fn = far.z - near.z;

    // Handle boundary problem
    if (x >= volume_size.x - 1)
        diff_ew = -center.x - west.x;

    if (x <= 0)
        diff_ew = east.x + center.x;

    if (y >= volume_size.y - 1)
        diff_ns = -center.y - south.y;

    if (y <= 0)
        diff_ns = north.y + center.y;

    if (z >= volume_size.z - 1)
        diff_fn = -center.z - far.z;

    if (z <= 0)
        diff_fn = near.z + center.z;

    float alpha = 0;
    if (diff_ew != 0 || diff_ns != 0 || diff_fn != 0)
        alpha = 1;

    out_data[index] = make_float4(
        0.0f, half_inverse_cell_size * (diff_ew + diff_ns + diff_fn), 0.0f,
        alpha);// 0.0f);
}

// =============================================================================

void LaunchRoundPassed(int* dest_array, int round, int x)
{
    RoundPassedKernel<<<1, 1>>>(dest_array, round, x);
}

void LaunchProlongatePacked(float4* dest_array, hipArray* coarse_array,
                            hipArray* fine_array, int3 volume_size_fine)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    prolongate_coarse.normalized = false;
    prolongate_coarse.filterMode = hipFilterModeLinear;
    prolongate_coarse.addressMode[0] = hipAddressModeClamp;
    prolongate_coarse.addressMode[1] = hipAddressModeClamp;
    prolongate_coarse.addressMode[2] = hipAddressModeClamp;
    prolongate_coarse.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&prolongate_coarse,
                                                coarse_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    prolongate_fine.normalized = false;

    // TODO: Disabling the linear filter mode may slightly speed up the kernel.
    prolongate_fine.filterMode = hipFilterModeLinear;
    prolongate_fine.addressMode[0] = hipAddressModeClamp;
    prolongate_fine.addressMode[1] = hipAddressModeClamp;
    prolongate_fine.addressMode[2] = hipAddressModeClamp;
    prolongate_fine.channelDesc = desc;

    result = hipBindTextureToArray(&prolongate_fine, fine_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    int3 volume_size = volume_size_fine;
    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ProlongatePackedKernel<<<grid, block>>>(dest_array, num_of_blocks_per_slice,
                                            slice_stride, volume_size);

    hipUnbindTexture(&prolongate_fine);
    hipUnbindTexture(&prolongate_coarse);
}

void LaunchAdvectVelocity(float4* dest_array, hipArray* velocity_array,
                          float time_step, float dissipation, int3 volume_size)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    advect_velocity.normalized = false;
    advect_velocity.filterMode = hipFilterModeLinear;
    advect_velocity.addressMode[0] = hipAddressModeClamp;
    advect_velocity.addressMode[1] = hipAddressModeClamp;
    advect_velocity.addressMode[2] = hipAddressModeClamp;
    advect_velocity.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&advect_velocity,
                                                velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    AdvectVelocityKernel<<<grid, block>>>(dest_array, time_step, dissipation,
                                          num_of_blocks_per_slice, slice_stride,
                                          volume_size);

    hipUnbindTexture(&advect_velocity);
}

void LaunchAdvect(float* dest_array, hipArray* velocity_array,
                  hipArray* source_array, float time_step,
                  float dissipation, int3 volume_size)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    advect_velocity.normalized = false;
    advect_velocity.filterMode = hipFilterModeLinear;
    advect_velocity.addressMode[0] = hipAddressModeClamp;
    advect_velocity.addressMode[1] = hipAddressModeClamp;
    advect_velocity.addressMode[2] = hipAddressModeClamp;
    advect_velocity.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&advect_velocity,
                                                velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    desc = hipCreateChannelDesc<float>();
    advect_source.normalized = false;
    advect_source.filterMode = hipFilterModeLinear;
    advect_source.addressMode[0] = hipAddressModeClamp;
    advect_source.addressMode[1] = hipAddressModeClamp;
    advect_source.addressMode[2] = hipAddressModeClamp;
    advect_source.channelDesc = desc;

    result = hipBindTextureToArray(&advect_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    AdvectKernel<<<grid, block>>>(dest_array, time_step, dissipation,
                                  num_of_blocks_per_slice, slice_stride,
                                  volume_size);

    hipUnbindTexture(&advect_source);
    hipUnbindTexture(&advect_velocity);
}

void LaunchApplyBuoyancy(float4* dest_array, hipArray* velocity_array,
                         hipArray* temperature_array, float time_step,
                         float ambient_temperature, float accel_factor,
                         float gravity, int3 volume_size)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    buoyancy_velocity.normalized = false;
    buoyancy_velocity.filterMode = hipFilterModeLinear;
    buoyancy_velocity.addressMode[0] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[1] = hipAddressModeClamp;
    buoyancy_velocity.addressMode[2] = hipAddressModeClamp;
    buoyancy_velocity.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&buoyancy_velocity,
                                                velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    desc = hipCreateChannelDesc<float>();
    buoyancy_temperature.normalized = false;
    buoyancy_temperature.filterMode = hipFilterModeLinear;
    buoyancy_temperature.addressMode[0] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[1] = hipAddressModeClamp;
    buoyancy_temperature.addressMode[2] = hipAddressModeClamp;
    buoyancy_temperature.channelDesc = desc;

    result = hipBindTextureToArray(&buoyancy_temperature,
                                    temperature_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ApplyBuoyancyKernel<<<grid, block>>>(dest_array, time_step,
                                         ambient_temperature, accel_factor,
                                         gravity, num_of_blocks_per_slice,
                                         slice_stride, volume_size);

    hipUnbindTexture(&buoyancy_temperature);
    hipUnbindTexture(&buoyancy_velocity);
}

void LaunchApplyImpulse(float* dest_array, hipArray* original_array,
                        float3 center_point, float3 hotspot, float radius,
                        float value, int3 volume_size)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    impulse_original.normalized = false;
    impulse_original.filterMode = hipFilterModeLinear;
    impulse_original.addressMode[0] = hipAddressModeClamp;
    impulse_original.addressMode[1] = hipAddressModeClamp;
    impulse_original.addressMode[2] = hipAddressModeClamp;
    impulse_original.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&impulse_original,
                                                original_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ApplyImpulseKernel<<<grid, block>>>(dest_array, center_point, hotspot,
                                        radius, value, num_of_blocks_per_slice,
                                        slice_stride, volume_size);

    hipUnbindTexture(&impulse_original);
}

void LaunchComputeDivergence(float4* dest_array, hipArray* velocity_array,
                             float half_inverse_cell_size, int3 volume_size)
{
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    divergence_velocity.normalized = false;
    divergence_velocity.filterMode = hipFilterModeLinear;
    divergence_velocity.addressMode[0] = hipAddressModeClamp;
    divergence_velocity.addressMode[1] = hipAddressModeClamp;
    divergence_velocity.addressMode[2] = hipAddressModeClamp;
    divergence_velocity.channelDesc = desc;

    hipError_t result = hipBindTextureToArray(&divergence_velocity,
                                                velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int num_of_blocks_per_slice = volume_size.y / 8;
    int slice_stride = volume_size.x * volume_size.y;

    ComputeDivergenceKernel<<<grid, block>>>(dest_array, half_inverse_cell_size,
                                             num_of_blocks_per_slice,
                                             slice_stride, volume_size);

    hipUnbindTexture(&divergence_velocity);
}