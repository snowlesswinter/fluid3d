#include "cuda_core.h"

#include <cassert>

#include "glew.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include "opengl/gl_texture.h"
#include "graphics_resource.h"

CudaCore::CudaCore()
{

}

CudaCore::~CudaCore()
{
    hipDeviceReset();
}

bool CudaCore::Init()
{
    findCudaGLDevice(0, nullptr);
    return 0;
}

int CudaCore::RegisterGLImage(const GLTexture& texture,
                              GraphicsResource* graphics_res)
{
    hipError_t result = hipGraphicsGLRegisterImage(
        graphics_res->Receive(), texture.handle(), texture.target(),
        hipGraphicsRegisterFlagsNone);
    assert(result == hipSuccess);
    return result == hipSuccess ? 0 : -1;
}

void CudaCore::UnregisterGLImage(GraphicsResource* graphics_res)
{
    hipGraphicsUnregisterResource(graphics_res->resource());
}
