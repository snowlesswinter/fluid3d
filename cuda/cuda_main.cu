#include "cuda_main.h"

#include "glew.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include "opengl/gl_texture.h"

CudaMain* CudaMain::Instance()
{
    static CudaMain* instance = nullptr;
    if (!instance)
        instance = new CudaMain();

    return instance;
}

CudaMain::CudaMain()
    : graphics_res_(nullptr)
{

}

CudaMain::~CudaMain()
{

}

int CudaMain::RegisterGLImage(const GLTexture& texture)
{
    hipError_t result = hipGraphicsGLRegisterImage(
        &graphics_res_, texture.handle(), texture.target(),
        hipGraphicsRegisterFlagsWriteDiscard);
    return result == hipSuccess ? 0 : -1;
}
