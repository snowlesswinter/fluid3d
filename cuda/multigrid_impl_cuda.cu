#include "hip/hip_runtime.h"
#include <cassert>

#include "third_party/opengl/glew.h"

#include <hip/hip_vector_types.h>

surface<void, hipTextureType3D> advect_dest;
texture<ushort4, hipTextureType3D, hipReadModeNormalizedFloat> advect_velocity;
texture<ushort, hipTextureType3D, hipReadModeNormalizedFloat> advect_source;

__global__ void AdvectKernel(float time_step, float dissipation,
                                 int slice_stride, int3 volume_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float3 coord = make_float3(x, y, z);
    coord += 0.5f;
    float4 velocity = tex3D(advect_velocity, coord.x, coord.y, coord.z);
    float3 back_traced =
        coord - time_step * make_float3(velocity.x, velocity.y, velocity.z);

    float result = dissipation * tex3D(advect_source, back_traced.x,
                                       back_traced.y, back_traced.z);
    surf3Dwrite(__float2half_rn(result), advect_dest, x * sizeof(ushort), y, z,
                hipBoundaryModeTrap);
}

// =============================================================================

void LaunchAdvect(hipArray_t dest_array, hipArray_t velocity_array,
                      hipArray_t source_array, float time_step,
                      float dissipation, int3 volume_size)
{
    hipChannelFormatDesc desc = cudaCreateChannelDescHalf();
    advect_dest.channelDesc = desc;

    hipError_t result = cudaBindSurfaceToArray(&advect_dest, dest_array,
                                                &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    desc = cudaCreateChannelDescHalf4();
    advect_velocity.normalized = false;
    advect_velocity.filterMode = hipFilterModeLinear;
    advect_velocity.addressMode[0] = hipAddressModeClamp;
    advect_velocity.addressMode[1] = hipAddressModeClamp;
    advect_velocity.addressMode[2] = hipAddressModeClamp;
    advect_velocity.channelDesc = desc;

    result = hipBindTextureToArray(&advect_velocity, velocity_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    desc = cudaCreateChannelDescHalf();
    advect_source.normalized = false;
    advect_source.filterMode = hipFilterModeLinear;
    advect_source.addressMode[0] = hipAddressModeClamp;
    advect_source.addressMode[1] = hipAddressModeClamp;
    advect_source.addressMode[2] = hipAddressModeClamp;
    advect_source.channelDesc = desc;

    result = hipBindTextureToArray(&advect_source, source_array, &desc);
    assert(result == hipSuccess);
    if (result != hipSuccess)
        return;

    dim3 block(8, 8, volume_size.x / 8);
    dim3 grid(volume_size.x / block.x, volume_size.y / block.y,
              volume_size.z / block.z);
    int slice_stride = volume_size.x * volume_size.y;

    AdvectKernel<<<grid, block>>>(time_step, dissipation, slice_stride,
                                      volume_size);

    hipUnbindTexture(&advect_source);
    hipUnbindTexture(&advect_velocity);
}
